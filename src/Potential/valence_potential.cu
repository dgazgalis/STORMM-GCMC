#include "hip/hip_runtime.h"
// -*-c++-*-
#include "Constants/hpc_bounds.h"
#include "Constants/fixed_precision.h"
#include "DataTypes/common_types.h"
#include "Topology/atomgraph_abstracts.h"
#include "Synthesis/synthesis_abstracts.h"
#include "Synthesis/valence_workunit.h"

namespace omni {
namespace energy {

using numerics::max_int_accumulation_f;
using numerics::max_int_accumulation_ll;
using synthesis::maximum_valence_work_unit_atoms;

#include accumulation.i

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__ double3 crossProduct(const double3 a, const double3 b) {
  return { (a.y * b.z) - (a.z - b.y), (a.z * b.x) - (a.x - b.z), (a.x * b.y) - (a.y * b.x) };
}

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__ float3 crossProduct(const float3 a, const float3 b) {
  return { (a.y * b.z) - (a.z - b.y), (a.z * b.x) - (a.x - b.z), (a.x * b.y) - (a.y * b.x) };
}

// Single-precision floating point definitions
#define TCALC float
#  define TCALC3 float3
#  define CONV_FUNC __float2int_rn
#  define LLCONV_FUNC __float2ll_rn
#  define SQRT_FUNC sqrtf
#  define ACOS_FUNC acosf
#  define COS_FUNC  cosf
#  define SIN_FUNC  sinf
#  define ABS_FUNC  fabsf
#  define CHECK_COSARG

#  define COMPUTE_FORCE
#    define SPLIT_FORCE_ACCUMULATION
#      define KERNEL_NAME kfsValenceForceAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME  
#      define COMPUTE_ENERGY
#        define KERNEL_NAME kfsValenceForceEnergyAccumulation
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef COMPUTE_ENERGY
#    undef SPLIT_FORCE_ACCUMULATION
#    define KERNEL_NAME kfValenceForceAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME  
#    define UPDATE_ATOMS
#      define KERNEL_NAME kfValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef UPDATE_ATOMS
#    define COMPUTE_ENERGY
#      define KERNEL_NAME kfValenceForceEnergyAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#      define UPDATE_ATOMS
#        define KERNEL_NAME kfValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#    undef COMPUTE_ENERGY
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define KERNEL_NAME kfValenceEnergyAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME
#  undef  COMPUTE_ENERGY

// Clear single-precision floating point definitions
#  undef TCALC3
#  undef CONV_FUNC
#  undef LLCONV_FUNC
#  undef SQRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef CHECK_COSARG
#undef TCALC

// Double-precision floating point definitions
#define TCALC double
#  define TCALC3 double3
#  define CONV_FUNC __double2ll_rn
#  define LLCONV_FUNC __double2ll_rn
#  define SQRT_FUNC sqrt
#  define ACOS_FUNC acos
#  define COS_FUNC  cos
#  define SIN_FUNC  sin
#  define ABS_FUNC  fabs

#  define COMPUTE_FORCE
#    define KERNEL_NAME kdValenceForceAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME  
#    define UPDATE_ATOMS
#      define KERNEL_NAME kdValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef UPDATE_ATOMS
#    define COMPUTE_ENERGY
#      define KERNEL_NAME kdValenceForceEnergyAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#      define UPDATE_ATOMS
#        define KERNEL_NAME kdValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#    undef  COMPUTE_ENERGY
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define KERNEL_NAME kdValenceEnergyAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME
#  undef  COMPUTE_ENERGY

// Clear double-precision floating point definitions
#  undef TCALC3
#  undef CONV_FUNC
#  undef LLCONV_FUNC
#  undef SQRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#undef TCALC

} // namespace energy
} // namespace omni

#endif
