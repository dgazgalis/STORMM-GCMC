#include "hip/hip_runtime.h"
// -*-c++-*-
#include "copyright.h"
#include "Accelerator/ptx_macros.h"
#include "Accelerator/gpu_details.h"
#include "Constants/hpc_bounds.h"
#include "Constants/scaling.h"
#include "Constants/symbol_values.h"
#include "DataTypes/common_types.h"
#include "DataTypes/stormm_vector_types.h"
#include "Math/rounding.h"
#include "Numerics/split_fixed_precision.h"
#include "Potential/energy_enumerators.h"
#include "Synthesis/valence_workunit.h"
#include "Topology/atomgraph_abstracts.h"
#include "Topology/atomgraph_enumerators.h"
#include "hpc_valence_potential.h"

namespace stormm {
namespace energy {

using card::GpuDetails;
using card::KernelManager;
using constants::PrecisionModel;
using constants::large_block_size;
using constants::medium_block_size;
using constants::small_block_size;
using constants::twice_warp_bits_mask_int;
using constants::twice_warp_size_int;
using constants::warp_size_int;
using constants::warp_bits;
using constants::warp_bits_mask_int;
using math::roundUp;
using numerics::chooseAccumulationMethod;
using numerics::AccumulationMethod;
using numerics::max_int_accumulation;
using numerics::max_int_accumulation_f;
using numerics::max_int_accumulation_ll;
using numerics::max_llint_accumulation;
using numerics::max_llint_accumulation_f;
using symbols::asymptotic_to_one_f;
using symbols::asymptotic_to_one_lf;
using symbols::boltzmann_constant;
using symbols::inverse_one_minus_asymptote_f;
using symbols::inverse_one_minus_asymptote_lf;
using symbols::near_to_one_f;
using symbols::near_to_one_lf;
using symbols::pi;
using symbols::pi_f;
using symbols::twopi;
using symbols::twopi_f;
using symbols::inverse_twopi_f;
using synthesis::maximum_valence_work_unit_atoms;
using synthesis::VwuAbstractMap;
using synthesis::VwuGoal;
using synthesis::vwu_abstract_length;
using trajectory::ThermostatKind;
using topology::TorsionKind;
  
#include "Numerics/accumulation.cui"
#include "Math/rounding.cui"
#include "Math/vector_formulas.cui"
#include "Trajectory/thermostat_utilities.cui"

//-------------------------------------------------------------------------------------------------
// Compute an angle based on the value of its cosine, with the understanding that a fallback
// method is appropriate when the angle is too actue for acos to be numerically well-conditioned.
//
// Overloaded:
//   - Single-precision version
//   - Double-precision version
//
// Arguments:
//   costheta:   Cosine value of the angle of interest
//   crabbc:     The first of two vectors decsribing the displacements that determine the angle
//   crbccd:     The second of two vectors decsribing the displacements that determine the angle
//   bc:         Vector defining the directionality of the angle
//   scr:        Second vector defining the directionality of the angle
//-------------------------------------------------------------------------------------------------
__device__ __forceinline__ float devcAngleVerification(const float costheta, const float3 crabbc,
                                                       const float3 crbccd, const float3 bc,
                                                       const float3 scr) {
  if (fabsf(costheta) >= near_to_one_f) {

    // The floating-point representation of costheta is numerically ill-conditioned.  Compute the
    // distance from atom I to the plane of atoms J, K, and L to get the angle by the arcsin of an
    // extremely acute angle.
    const float mg_crabbc = 1.0f / sqrtf((crabbc.x * crabbc.x) + (crabbc.y * crabbc.y) +
                                         (crabbc.z * crabbc.z));
    const float mg_crbccd = 1.0f / sqrtf((crbccd.x * crbccd.x) + (crbccd.y * crbccd.y) +
                                         (crbccd.z * crbccd.z));
    const float nx_abbc = crabbc.x * mg_crabbc;
    const float ny_abbc = crabbc.y * mg_crabbc;
    const float nz_abbc = crabbc.z * mg_crabbc;
    const float nx_bccd = crbccd.x * mg_crbccd;
    const float ny_bccd = crbccd.y * mg_crbccd;
    const float nz_bccd = crbccd.z * mg_crbccd;
    float rdx = nx_bccd - nx_abbc;
    float rdy = ny_bccd - ny_abbc;
    float rdz = nz_bccd - nz_abbc;
    float rs = sqrtf((rdx * rdx) + (rdy * rdy) + (rdz * rdz));
    if (fabsf(rs) > 1.0f) {
      rdx = nx_bccd + nx_abbc;
      rdy = ny_bccd + ny_abbc;
      rdz = nz_bccd + nz_abbc;
      rs = pi_f - sqrtf((rdx * rdx) + (rdy * rdy) + (rdz * rdz));
    }
    return ((scr.x * bc.x) + (scr.y * bc.y) + (scr.z * bc.z) > 0.0f) ? rs : -rs;
  }
  else {
    return ((scr.x * bc.x) + (scr.y * bc.y) + (scr.z * bc.z) > 0.0f) ?
            acosf(costheta) : -acosf(costheta);
  }
}

__device__ __forceinline__ double devcAngleVerification(const double costheta,
                                                        const double3 crabbc, const double3 crbccd,
                                                        const double3 bc, const double3 scr) {
  if (fabs(costheta) >= near_to_one_lf) {

    // The double-precision arccosine function is also vulnerable to numerical instability near
    // zero, so planar dihedral angles can still generate divergent forces on the order of 3.0e-7
    // kcal/mol-A.  Correct this with a similar strategy to the single-precision case.
    const double mg_crabbc = 1.0 / sqrt((crabbc.x * crabbc.x) + (crabbc.y * crabbc.y) +
                                        (crabbc.z * crabbc.z));
    const double mg_crbccd = 1.0 / sqrt((crbccd.x * crbccd.x) + (crbccd.y * crbccd.y) +
                                        (crbccd.z * crbccd.z));
    const double nx_abbc = crabbc.x * mg_crabbc;
    const double ny_abbc = crabbc.y * mg_crabbc;
    const double nz_abbc = crabbc.z * mg_crabbc;
    const double nx_bccd = crbccd.x * mg_crbccd;
    const double ny_bccd = crbccd.y * mg_crbccd;
    const double nz_bccd = crbccd.z * mg_crbccd;
    double rdx = nx_bccd - nx_abbc;
    double rdy = ny_bccd - ny_abbc;
    double rdz = nz_bccd - nz_abbc;
    double rs = sqrt((rdx * rdx) + (rdy * rdy) + (rdz * rdz));
    if (fabs(rs) > 1.0) {
      rdx = nx_bccd + nx_abbc;
      rdy = ny_bccd + ny_abbc;
      rdz = nz_bccd + nz_abbc;
      rs = pi - sqrt((rdx * rdx) + (rdy * rdy) + (rdz * rdz));
    }
    return ((scr.x * bc.x) + (scr.y * bc.y) + (scr.z * bc.z) > 0.0) ? rs : -rs;
  }
  else {
    return ((scr.x * bc.x) + (scr.y * bc.y) + (scr.z * bc.z) > 0.0) ?
            acos(costheta) : -acos(costheta);
  }
}

//-------------------------------------------------------------------------------------------------
// Compute critical elements of the restraining potential: its difference from the target value
// that determines some harmonic stiffness penalty, the harmonic penalty stiffness, and the energy
// contribution.
//
// Overloaded:
//   - Single-precision version
//   - Double-precision version
//
// Arguments:
//   init_k   Initial stiffness parameters
//   final_k  Final stiffness parameters
//   init_r   Initial displacement parameters
//   final_r  Final displacement parameters
//   mixwt    Pre-calculated mixing factor for combining initial and final parameters
//   dr       The measured value of the restraint coordinate among its participating atoms
//-------------------------------------------------------------------------------------------------
__device__ __forceinline__
double3 restraintDelta(const double2 init_k, const double2 final_k, const double4 init_r,
                      const double4 final_r, const double2 mixwt, const double dr) {
  const double r1 = (mixwt.x * init_r.x) + (mixwt.y * final_r.x);
  const double r2 = (mixwt.x * init_r.y) + (mixwt.y * final_r.y);
  const double r3 = (mixwt.x * init_r.z) + (mixwt.y * final_r.z);
  const double r4 = (mixwt.x * init_r.w) + (mixwt.y * final_r.w);
  const double k2 = (mixwt.x * init_k.x) + (mixwt.y * final_k.x);
  const double k3 = (mixwt.x * init_k.y) + (mixwt.y * final_k.y);
  double dl, du, keq;
  if (dr < r1) {
    dl = r1 - r2;
    du = k2 * ((dl * dl) + (2.0 * dl * (dr - r1)));
    keq = k2;
  }
  else if (dr < r2) {
    dl = dr - r2;
    du = k2 * dl * dl;
    keq = k2;
  }
  else if (dr < r3) {
    dl = 0.0;
    du = 0.0;
    keq = 0.0;
  }
  else if (dr < r4) {
    dl = dr - r3;
    du = k3 * dl * dl;
    keq = k3;
  }
  else {
    dl = r4 - r3;
    du = k3 * ((dl * dl) + (2.0 * dl * (dr - r4)));
    keq = k3;
  }
  return { keq, dl, du };
}

__device__ __forceinline__
float3 restraintDelta(const float2 init_k, const float2 final_k, const float4 init_r,
                      const float4 final_r, const float2 mixwt, const float dr) {
  const float r1 = (mixwt.x * init_r.x) + (mixwt.y * final_r.x);
  const float r2 = (mixwt.x * init_r.y) + (mixwt.y * final_r.y);
  const float r3 = (mixwt.x * init_r.z) + (mixwt.y * final_r.z);
  const float r4 = (mixwt.x * init_r.w) + (mixwt.y * final_r.w);
  const float k2 = (mixwt.x * init_k.x) + (mixwt.y * final_k.x);
  const float k3 = (mixwt.x * init_k.y) + (mixwt.y * final_k.y);
  float dl, du, keq;
  if (dr < r1) {
    dl = r1 - r2;
    du = k2 * ((dl * dl) + (2.0 * dl * (dr - r1)));
    keq = k2;
  }
  else if (dr < r2) {
    dl = dr - r2;
    du = k2 * dl * dl;
    keq = k2;
  }
  else if (dr < r3) {
    dl = 0.0;
    du = 0.0;
    keq = 0.0;
  }
  else if (dr < r4) {
    dl = dr - r3;
    du = k3 * dl * dl;
    keq = k3;
  }
  else {
    dl = r4 - r3;
    du = k3 * ((dl * dl) + (2.0 * dl * (dr - r4)));
    keq = k3;
  }
  return { keq, dl, du };
}

//-------------------------------------------------------------------------------------------------
// Compute the mixture of end-point values that will determine the actual strength and displacement
// settings of a flat-bottom bimodal harmonic restraint.  The flag about a RestraintApparatus
// having time-dependent restraints is mostly for convenience, a way to tell whether there is any
// time-dependent restraint in the collection at all.  Initial and final settings of the steps for
// each restraint encode whether there is actual time dependence in the result.
//
// Overloaded:
//   - Single-precision version
//   - Double-precision version
//
// Arguments:
//   step_number  The current step number of the simulation (may include energy minimization step
//                counts)
//   init_step    The initial step at which the restraint engages
//   final_step   The final step at which the restraint becomes mature
//-------------------------------------------------------------------------------------------------
__device__ __forceinline__
double2 computeRestraintMixtureD(const int step_number, const int init_step,
                                 const int final_step) {
  if (step_number < init_step) {

    // If the restraint has not yet engaged, neither its initial or final values have any weight
    return { (double)(0.0), (double)(0.0) };
  }
  else if (init_step == final_step) {

    // The step count is far enough along that the restraint has been engaged, and it is constant.
    // Only the initial value matters.
    return { (double)(1.0), (double)(0.0) };
  }
  else if (step_number < final_step) {
    const double wslide = (double)(step_number - init_step) / (double)(final_step - init_step);

    // The difference between the initial and final steps is nonzero.  The mixture is a linear
    // combination of the two end points.
    return { (double)(1.0) - wslide, wslide };
  }

  // The step number has advanced beyond the point at which the restraint is mature.
  return { (double)(0.0), (double)(1.0) };
}

__device__ __forceinline__
float2 computeRestraintMixtureF(const int step_number, const int init_step, const int final_step) {
  if (step_number < init_step) {

    // If the restraint has not yet engaged, neither its initial or final values have any weight
    return { (float)(0.0), (float)(0.0) };
  }
  else if (init_step == final_step) {

    // The step count is far enough along that the restraint has been engaged, and it is constant.
    // Only the initial value matters.
    return { (float)(1.0), (float)(0.0) };
  }
  else if (step_number < final_step) {
    const float wslide = (float)(step_number - init_step) / (float)(final_step - init_step);

    // The difference between the initial and final steps is nonzero.  The mixture is a linear
    // combination of the two end points.
    return { (float)(1.0) - wslide, wslide };
  }

  // The step number has advanced beyond the point at which the restraint is mature.
  return { (float)(0.0), (float)(1.0) };
}

// Single-precision floating point definitions
#define TCALC float
#  define VALENCE_BLOCK_MULTIPLICITY 2
#  define TCALC2 float2
#  define TCALC3 float3
#  define TCALC4 float4
#  define LLCONV_FUNC __float2ll_rn
#  define SPLITCONV_FUNC floatToInt63
#  define SPLIT_TYPE int2
#  define SQRT_FUNC sqrtf
#  define ACOS_FUNC acosf
#  define COS_FUNC  cosf
#  define SIN_FUNC  sinf
#  define ABS_FUNC  fabsf
#  define MIX_FUNC  computeRestraintMixtureF
#  define TCALC_IS_SINGLE
  
#  define COMPUTE_FORCE
#    define SPLIT_FORCE_ACCUMULATION
#      define VALENCE_KERNEL_THREAD_COUNT medium_block_size
#      define KERNEL_NAME kfsValenceForceAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME  
#      define UPDATE_ATOMS
#        define KERNEL_NAME kfsValenceAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define COMPUTE_ENERGY
#        define VALENCE_KERNEL_THREAD_COUNT 448
#        define KERNEL_NAME kfsValenceForceEnergyAccumulation
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#        define VALENCE_KERNEL_THREAD_COUNT 384
#        define UPDATE_ATOMS
#          define KERNEL_NAME kfsValenceEnergyAtomUpdate
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef UPDATE_ATOMS
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef COMPUTE_ENERGY
#      undef VALENCE_KERNEL_THREAD_COUNT
#    undef SPLIT_FORCE_ACCUMULATION
#    define VALENCE_KERNEL_THREAD_COUNT medium_block_size
#    define KERNEL_NAME kfValenceForceAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME  
#    define UPDATE_ATOMS
#      define KERNEL_NAME kfValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef UPDATE_ATOMS
#    undef VALENCE_KERNEL_THREAD_COUNT
#    define COMPUTE_ENERGY
#      define VALENCE_KERNEL_THREAD_COUNT 448
#      define KERNEL_NAME kfValenceForceEnergyAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define VALENCE_KERNEL_THREAD_COUNT 384
#      define UPDATE_ATOMS
#        define KERNEL_NAME kfValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#      undef VALENCE_KERNEL_THREAD_COUNT
#    undef COMPUTE_ENERGY
#    undef VALENCE_KERNEL_THREAD_COUNT
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define VALENCE_KERNEL_THREAD_COUNT medium_block_size
#    define KERNEL_NAME kfValenceEnergyAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME
#    undef VALENCE_KERNEL_THREAD_COUNT
#  undef  COMPUTE_ENERGY

// Clear single-precision floating point definitions
#  undef VALENCE_BLOCK_MULTIPLICITY
#  undef TCALC2
#  undef TCALC3
#  undef TCALC4
#  undef LLCONV_FUNC
#  undef SPLITCONV_FUNC
#  undef SPLIT_TYPE
#  undef SQRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef MIX_FUNC
#  undef TCALC_IS_SINGLE
#undef TCALC

// Double-precision floating point definitions
#define TCALC double
#  define VALENCE_KERNEL_THREAD_COUNT small_block_size
#  define VALENCE_BLOCK_MULTIPLICITY  2
#  define TCALC2 double2
#  define TCALC3 double3
#  define TCALC4 double4
#  define LLCONV_FUNC __double2ll_rn
#  define SPLITCONV_FUNC doubleToInt95
#  define SPLIT_TYPE int95_t
#  define SQRT_FUNC sqrt
#  define ACOS_FUNC acos
#  define COS_FUNC  cos
#  define SIN_FUNC  sin
#  define ABS_FUNC  fabs
#  define MIX_FUNC  computeRestraintMixtureD
#  define SPLIT_FORCE_ACCUMULATION

#  define COMPUTE_FORCE
#    define KERNEL_NAME kdsValenceForceAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME  
#    define UPDATE_ATOMS
#      define KERNEL_NAME kdsValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef UPDATE_ATOMS
#    define COMPUTE_ENERGY
#      define KERNEL_NAME kdsValenceForceEnergyAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#      define UPDATE_ATOMS
#        define KERNEL_NAME kdsValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#    undef  COMPUTE_ENERGY
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define KERNEL_NAME kdsValenceEnergyAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME
#  undef  COMPUTE_ENERGY

// Clear double-precision floating point definitions
#  undef VALENCE_KERNEL_THREAD_COUNT
#  undef VALENCE_BLOCK_MULTIPLICITY
#  undef TCALC2
#  undef TCALC3
#  undef TCALC4
#  undef LLCONV_FUNC
#  undef SPLITCONV_FUNC
#  undef SPLIT_TYPE
#  undef SQRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef MIX_FUNC
#  undef SPLIT_FORCE_ACCUMULATION
#undef TCALC

//-------------------------------------------------------------------------------------------------
extern void valenceKernelSetup() {
  const hipSharedMemConfig sms_eight = hipSharedMemBankSizeEightByte;
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceAtomUpdate), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceAtomUpdate __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceEnergyAtomUpdate), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceEnergyAtomUpdate __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceForceAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceForceAccumulation __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceEnergyAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceEnergyAccumulation __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceForceEnergyAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceForceEnergyAccumulation __shared__ memory bank size to eight "
          "bytes.", "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceAtomUpdate), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceAtomUpdate __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceEnergyAtomUpdate), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceEnergyAtomUpdate __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceForceAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceForceAccumulation __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceEnergyAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceEnergyAccumulation __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceForceEnergyAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceForceEnergyAccumulation __shared__ memory bank size to eight "
          "bytes.", "valenceKernelSetup");
  }
}

//-------------------------------------------------------------------------------------------------
extern hipFuncAttributes queryValenceKernelRequirements(const PrecisionModel prec,
                                                         const EvaluateForce eval_frc,
                                                         const EvaluateEnergy eval_nrg,
                                                         const AccumulationMethod acc_meth,
                                                         const VwuGoal purpose) {

  // The kernel manager will have information about the GPU to use--look at the work units from
  // the perspective of overall occupancy on the GPU.
  hipFuncAttributes result;
  switch (prec) {
  case PrecisionModel::DOUBLE:
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (purpose) {
        case VwuGoal::ACCUMULATE:
          if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceForceEnergyAccumulation)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kdsValenceForceEnergyAccumulation.",
                  "queryValenceKernelRequirements");
          }
          break;
        case VwuGoal::MOVE_PARTICLES:
          if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceEnergyAtomUpdate)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kdsValenceEnergyAtomUpdate.",
                  "queryValenceKernelRequirements");
          }
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (purpose) {
        case VwuGoal::ACCUMULATE:
          if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceForceAccumulation)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kdsValenceForceAccumulation.",
                  "queryValenceKernelRequirements");
          }
          break;
        case VwuGoal::MOVE_PARTICLES:
          if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceAtomUpdate)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kdsValenceAtomUpdate.",
                  "queryValenceKernelRequirements");
          }
          break;
        }
        break;
      }
    case EvaluateForce::NO:
      if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceEnergyAccumulation)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel kdsValenceEnergyAccumulation.",
              "queryValenceKernelRequirements");
      }
      break;
    }
    break;
  case PrecisionModel::SINGLE:
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceEnergyAccumulation)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfsValenceForceEnergyAccumulation.",
                    "queryValenceKernelRequirements");
            }
            break;
          case VwuGoal::MOVE_PARTICLES:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceEnergyAtomUpdate)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfsValenceEnergyAtomUpdate.",
                    "queryValenceKernelRequirements");
            }
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceEnergyAccumulation)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfValenceForceEnergyAccumulation.",
                    "queryValenceKernelRequirements");
            }
            break;
          case VwuGoal::MOVE_PARTICLES:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAtomUpdate)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfValenceEnergyAtomUpdate.",
                    "queryValenceKernelRequirements");
            }
            break;
          }
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceAccumulation)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfsValenceForceAccumulation.",
                    "queryValenceKernelRequirements");
            }
            break;
          case VwuGoal::MOVE_PARTICLES:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceAtomUpdate)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfsValenceAtomUpdate.",
                    "queryValenceKernelRequirements");
            }
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceAccumulation)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfValenceForceAccumulation.",
                    "queryValenceKernelRequirements");
            }
            break;
          case VwuGoal::MOVE_PARTICLES:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceAtomUpdate)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfValenceAtomUpdate.",
                    "queryValenceKernelRequirements");
            }
            break;
          }
          break;
        }
        break;
      }
      break;
    case EvaluateForce::NO:
      if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAccumulation)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel kfValenceEnergyAccumulation.",
              "queryValenceKernelRequirements");
      }
      break;
    }
    break;
  }
  return result;
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const SyValenceKit<double> &poly_vk,
                          const SyRestraintKit<double, double2, double4> &poly_rk,
                          MMControlKit<double> *ctrl, PsSynthesisWriter *poly_psw,
                          const SyAtomUpdateKit<double, double2, double4> &poly_auk,
                          ThermostatWriter<double> *tstw, ScoreCardWriter *scw,
                          CacheResourceKit<double> *gmem_r, const EvaluateForce eval_force,
                          const EvaluateEnergy eval_energy, const VwuGoal purpose, const int2 bt) {
  switch (purpose) {
  case VwuGoal::ACCUMULATE:

    // When the goal is to accumulate energies, forces, or both, the force accumulation method
    // is set to use int64 data.  A 95-bit method that splits the accumulation with overflow into
    // a secondary 32-bit int may be added, and likewise become the sole option for
    // double-precision computations.
    switch (eval_force) {
    case EvaluateForce::YES:
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kdsValenceForceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                          *scw, *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kdsValenceForceAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
        break;
      }
      break;
    case EvaluateForce::NO:
      kdsValenceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                   *gmem_r);
      break;
    }
    break;
  case VwuGoal::MOVE_PARTICLES:

    // When the goal is to move particles, evaluating the force is obligatory, but the manner in
    // which forces are accumulated is still important.  Whether to accumulate energies while
    // evaluating forces and moving the particles remains a consideration in choosing the proper
    // kernel.
    switch (eval_energy) {
    case EvaluateEnergy::YES:
      kdsValenceEnergyAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk,
                                                 *tstw, *scw, *gmem_r);
      break;
    case EvaluateEnergy::NO:
      kdsValenceAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk, *tstw,
                                           *gmem_r);
      break;
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const SyValenceKit<float> &poly_vk,
                          const SyRestraintKit<float, float2, float4> &poly_rk,
                          MMControlKit<float> *ctrl, PsSynthesisWriter *poly_psw,
                          const SyAtomUpdateKit<float, float2, float4> &poly_auk,
                          ThermostatWriter<float> *tstw, ScoreCardWriter *scw,
                          CacheResourceKit<float> *gmem_r, const EvaluateForce eval_force,
                          const EvaluateEnergy eval_energy, const VwuGoal purpose,
                          const AccumulationMethod force_sum, const int2 bt) {
  AccumulationMethod refined_force_sum;
  switch (force_sum) {
  case AccumulationMethod::SPLIT:
  case AccumulationMethod::WHOLE:
    refined_force_sum = force_sum;
    break;
  case AccumulationMethod::AUTOMATIC:
    refined_force_sum = chooseAccumulationMethod(poly_psw->frc_bits);
    break;
  }
  switch (purpose) {
  case VwuGoal::ACCUMULATE:
    
    // When the goal is to accumulate energies, forces, or both, the force accumulation method
    // becomes a critical detail when choosing the kernel.
    switch (eval_force) {
    case EvaluateForce::YES:
      switch (refined_force_sum) {
      case AccumulationMethod::SPLIT:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfsValenceForceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                            *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfsValenceForceAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
          break;
        }
        break;
      case AccumulationMethod::WHOLE:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfValenceForceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                           *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfValenceForceAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
          break;
        }
        break;
      case AccumulationMethod::AUTOMATIC:
        break;
      }
      break;
    case EvaluateForce::NO:
      kfValenceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                  *gmem_r);
      break;
    }
    break;
  case VwuGoal::MOVE_PARTICLES:
    
    // When the goal is to move particles, evaluating the force is obligatory, but the manner in
    // which forces are accumulated is still important.  Whether to accumulate energies while
    // evaluating forces and moving the particles remains a consideration in choosing the proper
    // kernel.
    switch (refined_force_sum) {
    case AccumulationMethod::SPLIT:
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kfsValenceEnergyAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk,
                                                   *tstw, *scw, *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kfsValenceAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk, *tstw,
                                             *gmem_r);
        break;
      }
      break;
    case AccumulationMethod::WHOLE:
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kfValenceEnergyAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk,
                                                  *tstw, *scw, *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kfValenceAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk, *tstw,
                                            *gmem_r);
        break;
      }
      break;
    case AccumulationMethod::AUTOMATIC:
      break;
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const PrecisionModel prec, const AtomGraphSynthesis &poly_ag,
                          MolecularMechanicsControls *mmctrl, PhaseSpaceSynthesis *poly_ps,
                          Thermostat *heat_bath, ScoreCard *sc, CacheResource *tb_space,
                          const EvaluateForce eval_force, const EvaluateEnergy eval_energy,
                          const VwuGoal purpose, const AccumulationMethod force_sum,
                          const KernelManager &launcher) {
  const HybridTargetLevel tier = HybridTargetLevel::DEVICE;
  PsSynthesisWriter poly_psw = poly_ps->data(tier);
  ScoreCardWriter scw = sc->data(tier);
  const int2 bt = launcher.getValenceKernelDims(prec, eval_force, eval_energy,
                                                AccumulationMethod::SPLIT, purpose);
  switch (prec) {
  case PrecisionModel::DOUBLE:
    {
      const SyValenceKit<double> poly_vk = poly_ag.getDoublePrecisionValenceKit(tier);
      const SyRestraintKit<double, double2, double4> poly_rk =
        poly_ag.getDoublePrecisionRestraintKit(tier);
      const SyAtomUpdateKit<double, double2, double4> poly_auk =
        poly_ag.getDoublePrecisionAtomUpdateKit(tier);
      MMControlKit<double> ctrl = mmctrl->dpData(tier);
      ThermostatWriter tstw = heat_bath->dpData(tier);
      CacheResourceKit<double> gmem_r = tb_space->dpData(tier);
      launchValence(poly_vk, poly_rk, &ctrl, &poly_psw, poly_auk, &tstw, &scw, &gmem_r, eval_force,
                    eval_energy, purpose, bt);
    }
    break;
  case PrecisionModel::SINGLE:
    {
      const SyValenceKit<float> poly_vk = poly_ag.getSinglePrecisionValenceKit(tier);
      const SyRestraintKit<float, float2, float4> poly_rk =
        poly_ag.getSinglePrecisionRestraintKit(tier);
      const SyAtomUpdateKit<float, float2, float4> poly_auk =
        poly_ag.getSinglePrecisionAtomUpdateKit(tier);
      MMControlKit<float> ctrl = mmctrl->spData(tier);
      ThermostatWriter tstw = heat_bath->spData(tier);
      CacheResourceKit<float> gmem_r = tb_space->spData(tier);
      launchValence(poly_vk, poly_rk, &ctrl, &poly_psw, poly_auk, &tstw, &scw, &gmem_r, eval_force,
                    eval_energy, purpose, force_sum, bt);
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const PrecisionModel prec, const AtomGraphSynthesis &poly_ag,
                          MolecularMechanicsControls *mmctrl, PhaseSpaceSynthesis *poly_ps,
                          Thermostat *heat_bath, ScoreCard *sc, CacheResource *tb_space,
                          const EvaluateForce eval_force, const EvaluateEnergy eval_energy,
                          const VwuGoal purpose, const KernelManager &launcher) {
  if (prec == PrecisionModel::DOUBLE || poly_ps->getForceAccumulationBits() <= 24) {
    launchValence(prec, poly_ag, mmctrl, poly_ps, heat_bath, sc, tb_space, eval_force, eval_energy,
                  purpose, AccumulationMethod::SPLIT, launcher);
  }
  else {
    launchValence(prec, poly_ag, mmctrl, poly_ps, heat_bath, sc, tb_space, eval_force, eval_energy,
                  purpose, AccumulationMethod::WHOLE, launcher);
  }
}

} // namespace energy
} // namespace stormm
