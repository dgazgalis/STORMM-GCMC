#include "hip/hip_runtime.h"
// -*-c++-*-
#include "Constants/hpc_bounds.h"
#include "Constants/fixed_precision.h"
#include "Constants/scaling.h"
#include "Constants/symbol_values.h"
#include "DataTypes/common_types.h"
#include "Potential/scorecard.h"
#include "Synthesis/phasespace_synthesis.h"
#include "Synthesis/synthesis_abstracts.h"
#include "Synthesis/synthesis_enumerators.h"
#include "Synthesis/valence_workunit.h"
#include "Topology/atomgraph_abstracts.h"
#include "Topology/atomgraph_enumerators.h"

namespace omni {
namespace energy {

using constants::warp_size_int;
using constants::twice_warp_bits_mask_int;
using constants::twice_warp_size_int;
using math::roundUp;
using numerics::max_int_accumulation_f;
using numerics::max_int_accumulation_ll;
using symbols::asymptotic_to_one_f;
using symbols::asymptotic_to_one_lf;
using symbols::inverse_one_minus_asymptote_f;
using symbols::inverse_one_minus_asymptote_lf;
using symbols::near_to_one_f;
using symbols::pi_f;
using synthesis::maximum_valence_work_unit_atoms;
using synthesis::SyValenceKit;
using synthesis::PsSynthesisWriter;
using synthesis::VwuAbstractMap;
using synthesis::vwu_abstract_length;
using topology::TorsionKind;
  
#include "accumulation.cui"
#include "Math/rounding.cui"

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__ double3 crossProduct(const double3 a, const double3 b) {
  return { (a.y * b.z) - (a.z - b.y), (a.z * b.x) - (a.x - b.z), (a.x * b.y) - (a.y * b.x) };
}

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__ float3 crossProduct(const float3 a, const float3 b) {
  return { (a.y * b.z) - (a.z - b.y), (a.z * b.x) - (a.x - b.z), (a.x * b.y) - (a.y * b.x) };
}

// Single-precision floating point definitions
#define TCALC float
#  define TCALC3 float3
#  define CONV_FUNC __float2int_rn
#  define LLCONV_FUNC __float2ll_rn
#  define SQRT_FUNC sqrtf
#  define ACOS_FUNC acosf
#  define COS_FUNC  cosf
#  define SIN_FUNC  sinf
#  define ABS_FUNC  fabsf
#  define CHECK_COSARG

#  define COMPUTE_FORCE
#    define SPLIT_FORCE_ACCUMULATION
#      define KERNEL_NAME kfsValenceForceAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME  
#      define COMPUTE_ENERGY
#        define KERNEL_NAME kfsValenceForceEnergyAccumulation
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef COMPUTE_ENERGY
#    undef SPLIT_FORCE_ACCUMULATION
#    define KERNEL_NAME kfValenceForceAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME  
#    define UPDATE_ATOMS
#      define KERNEL_NAME kfValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef UPDATE_ATOMS
#    define COMPUTE_ENERGY
#      define KERNEL_NAME kfValenceForceEnergyAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#      define UPDATE_ATOMS
#        define KERNEL_NAME kfValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#    undef COMPUTE_ENERGY
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define KERNEL_NAME kfValenceEnergyAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME
#  undef  COMPUTE_ENERGY

// Clear single-precision floating point definitions
#  undef TCALC3
#  undef CONV_FUNC
#  undef LLCONV_FUNC
#  undef SQRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef CHECK_COSARG
#undef TCALC

// Double-precision floating point definitions
#define TCALC double
#  define TCALC3 double3
#  define CONV_FUNC __double2ll_rn
#  define LLCONV_FUNC __double2ll_rn
#  define SQRT_FUNC sqrt
#  define ACOS_FUNC acos
#  define COS_FUNC  cos
#  define SIN_FUNC  sin
#  define ABS_FUNC  fabs

#  define COMPUTE_FORCE
#    define KERNEL_NAME kdValenceForceAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME  
#    define UPDATE_ATOMS
#      define KERNEL_NAME kdValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef UPDATE_ATOMS
#    define COMPUTE_ENERGY
#      define KERNEL_NAME kdValenceForceEnergyAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#      define UPDATE_ATOMS
#        define KERNEL_NAME kdValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#    undef  COMPUTE_ENERGY
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define KERNEL_NAME kdValenceEnergyAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME
#  undef  COMPUTE_ENERGY

// Clear double-precision floating point definitions
#  undef TCALC3
#  undef CONV_FUNC
#  undef LLCONV_FUNC
#  undef SQRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#undef TCALC

} // namespace energy
} // namespace omni
