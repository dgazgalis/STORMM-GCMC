#include "hip/hip_runtime.h"
// -*-c++-*-
#include "copyright.h"
#include "Accelerator/ptx_macros.h"
#include "Accelerator/gpu_details.h"
#include "Constants/fixed_precision.h"
#include "Constants/hpc_bounds.h"
#include "Constants/scaling.h"
#include "Constants/symbol_values.h"
#include "DataTypes/common_types.h"
#include "DataTypes/stormm_vector_types.h"
#include "Math/rounding.h"
#include "Potential/energy_enumerators.h"
#include "Synthesis/valence_workunit.h"
#include "Topology/atomgraph_abstracts.h"
#include "Topology/atomgraph_enumerators.h"
#include "hpc_valence_potential.h"

namespace stormm {
namespace energy {

using card::GpuDetails;
using card::KernelManager;
using constants::large_block_size;
using constants::medium_block_size;
using constants::small_block_size;
using constants::twice_warp_bits_mask_int;
using constants::twice_warp_size_int;
using constants::warp_size_int;
using constants::warp_bits;
using constants::warp_bits_mask_int;
using data_types::int95_t;
using math::roundUp;
using numerics::chooseForceAccumulationMethod;
using numerics::ForceAccumulationMethod;
using numerics::max_int_accumulation;
using numerics::max_int_accumulation_f;
using numerics::max_int_accumulation_ll;
using numerics::max_llint_accumulation;
using numerics::max_llint_accumulation_f;
using numerics::PrecisionModel;
using symbols::asymptotic_to_one_f;
using symbols::asymptotic_to_one_lf;
using symbols::inverse_one_minus_asymptote_f;
using symbols::inverse_one_minus_asymptote_lf;
using symbols::near_to_one_f;
using symbols::near_to_one_lf;
using symbols::pi;
using symbols::pi_f;
using symbols::twopi;
using symbols::twopi_f;
using symbols::inverse_twopi_f;
using synthesis::maximum_valence_work_unit_atoms;
using synthesis::VwuAbstractMap;
using synthesis::VwuGoal;
using synthesis::vwu_abstract_length;
using topology::TorsionKind;
  
#include "accumulation.cui"
#include "Math/rounding.cui"
#include "Math/vector_formulas.cui"

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__ float angleVerification(const float costheta, const float3 crabbc,
                                                   const float3 crbccd, const float3 bc,
                                                   const float3 scr) {
  if (fabsf(costheta) >= near_to_one_f) {

    // The floating-point representation of costheta is numerically ill-conditioned.  Compute the
    // distance from atom I to the plane of atoms J, K, and L to get the angle by the arcsin of an
    // extremely acute angle.
    const float mg_crabbc = 1.0f / sqrtf((crabbc.x * crabbc.x) + (crabbc.y * crabbc.y) +
                                         (crabbc.z * crabbc.z));
    const float mg_crbccd = 1.0f / sqrtf((crbccd.x * crbccd.x) + (crbccd.y * crbccd.y) +
                                         (crbccd.z * crbccd.z));
    const float nx_abbc = crabbc.x * mg_crabbc;
    const float ny_abbc = crabbc.y * mg_crabbc;
    const float nz_abbc = crabbc.z * mg_crabbc;
    const float nx_bccd = crbccd.x * mg_crbccd;
    const float ny_bccd = crbccd.y * mg_crbccd;
    const float nz_bccd = crbccd.z * mg_crbccd;
    float rdx = nx_bccd - nx_abbc;
    float rdy = ny_bccd - ny_abbc;
    float rdz = nz_bccd - nz_abbc;
    float rs = sqrtf((rdx * rdx) + (rdy * rdy) + (rdz * rdz));
    if (fabsf(rs) > 1.0f) {
      rdx = nx_bccd + nx_abbc;
      rdy = ny_bccd + ny_abbc;
      rdz = nz_bccd + nz_abbc;
      rs = pi_f - sqrtf((rdx * rdx) + (rdy * rdy) + (rdz * rdz));
    }
    return ((scr.x * bc.x) + (scr.y * bc.y) + (scr.z * bc.z) > 0.0f) ? rs : -rs;
  }
  else {
    return ((scr.x * bc.x) + (scr.y * bc.y) + (scr.z * bc.z) > 0.0f) ?
            acosf(costheta) : -acosf(costheta);
  }
  __builtin_unreachable();
}

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__ double angleVerification(const double costheta, const double3 crabbc,
                                                    const double3 crbccd, const double3 bc,
                                                    const double3 scr) {
  if (fabs(costheta) >= near_to_one_lf) {

    // The double-precision arccosine function is also vulnerable to numerical instability near
    // zero, so planar dihedral angles can still generate divergent forces on the order of 3.0e-7
    // kcal/mol-A.  Correct this with a similar strategy to the single-precision case.
    const double mg_crabbc = 1.0 / sqrt((crabbc.x * crabbc.x) + (crabbc.y * crabbc.y) +
                                        (crabbc.z * crabbc.z));
    const double mg_crbccd = 1.0 / sqrt((crbccd.x * crbccd.x) + (crbccd.y * crbccd.y) +
                                        (crbccd.z * crbccd.z));
    const double nx_abbc = crabbc.x * mg_crabbc;
    const double ny_abbc = crabbc.y * mg_crabbc;
    const double nz_abbc = crabbc.z * mg_crabbc;
    const double nx_bccd = crbccd.x * mg_crbccd;
    const double ny_bccd = crbccd.y * mg_crbccd;
    const double nz_bccd = crbccd.z * mg_crbccd;
    double rdx = nx_bccd - nx_abbc;
    double rdy = ny_bccd - ny_abbc;
    double rdz = nz_bccd - nz_abbc;
    double rs = sqrt((rdx * rdx) + (rdy * rdy) + (rdz * rdz));
    if (fabs(rs) > 1.0) {
      rdx = nx_bccd + nx_abbc;
      rdy = ny_bccd + ny_abbc;
      rdz = nz_bccd + nz_abbc;
      rs = pi - sqrt((rdx * rdx) + (rdy * rdy) + (rdz * rdz));
    }
    return ((scr.x * bc.x) + (scr.y * bc.y) + (scr.z * bc.z) > 0.0) ? rs : -rs;
  }
  else {
    return ((scr.x * bc.x) + (scr.y * bc.y) + (scr.z * bc.z) > 0.0) ?
            acos(costheta) : -acos(costheta);
  }
  __builtin_unreachable();
}

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__
double3 restraintDelta(const double2 init_k, const double2 final_k, const double4 init_r,
                      const double4 final_r, const double2 mixwt, const double dr) {
  const double r1 = (mixwt.x * init_r.x) + (mixwt.y * final_r.x);
  const double r2 = (mixwt.x * init_r.y) + (mixwt.y * final_r.y);
  const double r3 = (mixwt.x * init_r.z) + (mixwt.y * final_r.z);
  const double r4 = (mixwt.x * init_r.w) + (mixwt.y * final_r.w);
  const double k2 = (mixwt.x * init_k.x) + (mixwt.y * final_k.x);
  const double k3 = (mixwt.x * init_k.y) + (mixwt.y * final_k.y);
  double dl, du, keq;
  if (dr < r1) {
    dl = r1 - r2;
    du = k2 * ((dl * dl) + (2.0 * dl * (dr - r1)));
    keq = k2;
  }
  else if (dr < r2) {
    dl = dr - r2;
    du = k2 * dl * dl;
    keq = k2;
  }
  else if (dr < r3) {
    dl = 0.0;
    du = 0.0;
    keq = 0.0;
  }
  else if (dr < r4) {
    dl = dr - r3;
    du = k3 * dl * dl;
    keq = k3;
  }
  else {
    dl = r4 - r3;
    du = k3 * ((dl * dl) + (2.0 * dl * (dr - r4)));
    keq = k3;
  }
  return { keq, dl, du };
}

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__
float3 restraintDelta(const float2 init_k, const float2 final_k, const float4 init_r,
                      const float4 final_r, const float2 mixwt, const float dr) {
  const float r1 = (mixwt.x * init_r.x) + (mixwt.y * final_r.x);
  const float r2 = (mixwt.x * init_r.y) + (mixwt.y * final_r.y);
  const float r3 = (mixwt.x * init_r.z) + (mixwt.y * final_r.z);
  const float r4 = (mixwt.x * init_r.w) + (mixwt.y * final_r.w);
  const float k2 = (mixwt.x * init_k.x) + (mixwt.y * final_k.x);
  const float k3 = (mixwt.x * init_k.y) + (mixwt.y * final_k.y);
  float dl, du, keq;
  if (dr < r1) {
    dl = r1 - r2;
    du = k2 * ((dl * dl) + (2.0 * dl * (dr - r1)));
    keq = k2;
  }
  else if (dr < r2) {
    dl = dr - r2;
    du = k2 * dl * dl;
    keq = k2;
  }
  else if (dr < r3) {
    dl = 0.0;
    du = 0.0;
    keq = 0.0;
  }
  else if (dr < r4) {
    dl = dr - r3;
    du = k3 * dl * dl;
    keq = k3;
  }
  else {
    dl = r4 - r3;
    du = k3 * ((dl * dl) + (2.0 * dl * (dr - r4)));
    keq = k3;
  }
  return { keq, dl, du };
}

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__
double2 computeRestraintMixtureD(const int step_number, const int init_step,
                                 const int final_step) {
  if (step_number < init_step) {

    // If the restraint has not yet engaged, neither its initial or final values have any weight
    return { (double)(0.0), (double)(0.0) };
  }
  else if (init_step == final_step) {

    // The step count is far enough along that the restraint has been engaged, and it is constant.
    // Only the initial value matters.
    return { (double)(1.0), (double)(0.0) };
  }
  else if (step_number < final_step) {
    const double wslide = (double)(step_number - init_step) / (double)(final_step - init_step);

    // The difference between the initial and final steps is nonzero.  The mixture is a linear
    // combination of the two end points.
    return { (double)(1.0) - wslide, wslide };
  }

  // The step number has advanced beyond the point at which the restraint is mature.
  return { (double)(0.0), (double)(1.0) };
}

//-------------------------------------------------------------------------------------------------
__device__ __forceinline__
float2 computeRestraintMixtureF(const int step_number, const int init_step, const int final_step) {
  if (step_number < init_step) {

    // If the restraint has not yet engaged, neither its initial or final values have any weight
    return { (float)(0.0), (float)(0.0) };
  }
  else if (init_step == final_step) {

    // The step count is far enough along that the restraint has been engaged, and it is constant.
    // Only the initial value matters.
    return { (float)(1.0), (float)(0.0) };
  }
  else if (step_number < final_step) {
    const float wslide = (float)(step_number - init_step) / (float)(final_step - init_step);

    // The difference between the initial and final steps is nonzero.  The mixture is a linear
    // combination of the two end points.
    return { (float)(1.0) - wslide, wslide };
  }

  // The step number has advanced beyond the point at which the restraint is mature.
  return { (float)(0.0), (float)(1.0) };
}

// Single-precision floating point definitions
#define TCALC float
#  define VALENCE_BLOCK_MULTIPLICITY 2
#  define TCALC2 float2
#  define TCALC3 float3
#  define TCALC4 float4
#  define LLCONV_FUNC __float2ll_rn
#  define SPLITCONV_FUNC floatToInt63
#  define SPLIT_TYPE int2
#  define SQRT_FUNC sqrtf
#  define ACOS_FUNC acosf
#  define COS_FUNC  cosf
#  define SIN_FUNC  sinf
#  define ABS_FUNC  fabsf
#  define MIX_FUNC  computeRestraintMixtureF
#  define TCALC_IS_SINGLE
  
#  define COMPUTE_FORCE
#    define SPLIT_FORCE_ACCUMULATION
#      define VALENCE_KERNEL_THREAD_COUNT medium_block_size
#      define KERNEL_NAME kfsValenceForceAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME  
#      define UPDATE_ATOMS
#        define KERNEL_NAME kfsValenceAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define COMPUTE_ENERGY
#        define VALENCE_KERNEL_THREAD_COUNT 448
#        define KERNEL_NAME kfsValenceForceEnergyAccumulation
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#        define UPDATE_ATOMS
#          define KERNEL_NAME kfsValenceEnergyAtomUpdate
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef UPDATE_ATOMS
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef COMPUTE_ENERGY
#      undef VALENCE_KERNEL_THREAD_COUNT
#    undef SPLIT_FORCE_ACCUMULATION
#    define VALENCE_KERNEL_THREAD_COUNT medium_block_size
#    define KERNEL_NAME kfValenceForceAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME  
#    define UPDATE_ATOMS
#      define KERNEL_NAME kfValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef UPDATE_ATOMS
#    undef VALENCE_KERNEL_THREAD_COUNT
#    define COMPUTE_ENERGY
#      define VALENCE_KERNEL_THREAD_COUNT 448
#      define KERNEL_NAME kfValenceForceEnergyAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#      define UPDATE_ATOMS
#        define KERNEL_NAME kfValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#      undef VALENCE_KERNEL_THREAD_COUNT
#    undef COMPUTE_ENERGY
#    undef VALENCE_KERNEL_THREAD_COUNT
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define VALENCE_KERNEL_THREAD_COUNT medium_block_size
#    define KERNEL_NAME kfValenceEnergyAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME
#    undef VALENCE_KERNEL_THREAD_COUNT
#  undef  COMPUTE_ENERGY

// Clear single-precision floating point definitions
#  undef VALENCE_BLOCK_MULTIPLICITY
#  undef TCALC2
#  undef TCALC3
#  undef TCALC4
#  undef LLCONV_FUNC
#  undef SPLITCONV_FUNC
#  undef SPLIT_TYPE
#  undef SQRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef MIX_FUNC
#  undef TCALC_IS_SINGLE
#undef TCALC

// Double-precision floating point definitions
#define TCALC double
#  define VALENCE_KERNEL_THREAD_COUNT small_block_size
#  define VALENCE_BLOCK_MULTIPLICITY  2
#  define TCALC2 double2
#  define TCALC3 double3
#  define TCALC4 double4
#  define LLCONV_FUNC __double2ll_rn
#  define SPLITCONV_FUNC doubleToInt95
#  define SPLIT_TYPE int95_t
#  define SQRT_FUNC sqrt
#  define ACOS_FUNC acos
#  define COS_FUNC  cos
#  define SIN_FUNC  sin
#  define ABS_FUNC  fabs
#  define MIX_FUNC  computeRestraintMixtureD
#  define SPLIT_FORCE_ACCUMULATION

#  define COMPUTE_FORCE
#    define KERNEL_NAME kdsValenceForceAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME  
#    define UPDATE_ATOMS
#      define KERNEL_NAME kdsValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef UPDATE_ATOMS
#    define COMPUTE_ENERGY
#      define KERNEL_NAME kdsValenceForceEnergyAccumulation
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#      define UPDATE_ATOMS
#        define KERNEL_NAME kdsValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef UPDATE_ATOMS
#    undef  COMPUTE_ENERGY
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define KERNEL_NAME kdsValenceEnergyAccumulation
#      include "valence_potential.cui"
#    undef KERNEL_NAME
#  undef  COMPUTE_ENERGY

// Clear double-precision floating point definitions
#  undef VALENCE_KERNEL_THREAD_COUNT
#  undef VALENCE_BLOCK_MULTIPLICITY
#  undef TCALC2
#  undef TCALC3
#  undef TCALC4
#  undef LLCONV_FUNC
#  undef SPLITCONV_FUNC
#  undef SPLIT_TYPE
#  undef SQRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef MIX_FUNC
#  undef SPLIT_FORCE_ACCUMULATION
#undef TCALC

//-------------------------------------------------------------------------------------------------
extern void valenceKernelSetup() {
  const hipSharedMemConfig sms_eight = hipSharedMemBankSizeEightByte;
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceAtomUpdate), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceAtomUpdate __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceEnergyAtomUpdate), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceEnergyAtomUpdate __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceForceAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceForceAccumulation __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceEnergyAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceEnergyAccumulation __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kfValenceForceEnergyAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kfValenceForceEnergyAccumulation __shared__ memory bank size to eight "
          "bytes.", "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceAtomUpdate), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceAtomUpdate __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceEnergyAtomUpdate), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceEnergyAtomUpdate __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceForceAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceForceAccumulation __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceEnergyAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceEnergyAccumulation __shared__ memory bank size to eight bytes.",
          "valenceKernelSetup");
  }
  if (hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kdsValenceForceEnergyAccumulation), sms_eight) != hipSuccess) {
    rtErr("Error setting kdsValenceForceEnergyAccumulation __shared__ memory bank size to eight "
          "bytes.", "valenceKernelSetup");
  }
}

//-------------------------------------------------------------------------------------------------
#if 0
int2 testValenceKernelSubdivision(const int max_threads, const int smp_count, const int vwu_size,
                                  const int vwu_count) {
  const std::vector<int> block_size_options = { 128, 160, 192, 224, 256, 288, 320, 384, 448, 512,
                                                768, 896, 1024 };
  const int n_options = block_size_options.size();
  const double dvwu_count   = static_cast<double>(vwu_count);
  const double dmax_threads = static_cast<double>(max_threads);
  double best_eff = 0.0;
  int best_block_size = max_threads;
  int best_block_mult = 1;
  for (size_t i = 0; i < n_options; i++) {
    const int tmp_bdim = block_size_options[i];

    // In some cases, the valence work unit size can be slightly greater than the thread count
    // per block, i.e. 1024 atoms and 896 threads in some of the kernels that compute forces and
    // energies.  However, if the number of atoms is approaching the number of threads per block,
    // efficiency will already be fairly high.
    if (vwu_size > tmp_bdim || tmp_bdim > max_threads) {
      continue;
    }
    const int tmp_mult    = (max_threads / tmp_bdim);
    const int tmp_blocks  = tmp_mult * smp_count;
    const double tmp_batches = static_cast<double>((vwu_count + tmp_blocks - 1) / tmp_blocks);
    const double sm_eff  = static_cast<double>(tmp_bdim * tmp_mult) / dmax_threads;

    // The efficiency within an individual valence work unit is estimated as the number of tasks
    // (five task estimates per atom) divided by the number of threads available to work on them,
    // cycling until each task can be done by one of the threads.  If there are more tasks than
    // threads, this implies threads do additional batches of work, and the idle threads in the
    // final batch are less an dless inefficiency as the workload grows.  The more batches a single
    // thread performs, the less memory reading ratio is also carries, so the threads' batch count
    // is given a minor bonus in the estimated score.
    const int vwu_batches = ((5 * vwu_size) + tmp_bdim - 1) / tmp_bdim;
    const double vwu_eff = (static_cast<double>(5 * vwu_size) /
                            static_cast<double>(vwu_batches * tmp_bdim)) *
                           static_cast<double>(vwu_batches + 5) /
                           static_cast<double>(vwu_batches + 6);
    const double net_eff = sm_eff * vwu_eff * dvwu_count / (tmp_batches *
                                                            static_cast<double>(tmp_blocks));
    if (net_eff >= best_eff) {
      best_eff = net_eff;
      best_block_size = tmp_bdim;
      best_block_mult = tmp_mult;
    }
  }
  return { best_block_mult, best_block_size };
}
#endif

//-------------------------------------------------------------------------------------------------
extern hipFuncAttributes queryValenceKernelRequirements(const PrecisionModel prec,
                                                         const EvaluateForce eval_frc,
                                                         const EvaluateEnergy eval_nrg,
                                                         const ForceAccumulationMethod acc_meth,
                                                         const VwuGoal purpose) {

  // The kernel manager will have information about the GPU to use--look at the work units from
  // the perspective of overall occupancy on the GPU.
  hipFuncAttributes result;
  switch (prec) {
  case PrecisionModel::DOUBLE:
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (purpose) {
        case VwuGoal::ACCUMULATE:
          if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceForceEnergyAccumulation)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kdsValenceForceEnergyAccumulation.",
                  "queryValenceKernelRequirements");
          }
          break;
        case VwuGoal::MOVE_PARTICLES:
          if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceEnergyAtomUpdate)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kdsValenceEnergyAtomUpdate.",
                  "queryValenceKernelRequirements");
          }
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (purpose) {
        case VwuGoal::ACCUMULATE:
          if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceForceAccumulation)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kdsValenceForceAccumulation.",
                  "queryValenceKernelRequirements");
          }
          break;
        case VwuGoal::MOVE_PARTICLES:
          if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceAtomUpdate)) != hipSuccess) {
            rtErr("Error obtaining attributes for kernel kdsValenceAtomUpdate.",
                  "queryValenceKernelRequirements");
          }
          break;
        }
        break;
      }
    case EvaluateForce::NO:
      if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsValenceEnergyAccumulation)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel kdsValenceEnergyAccumulation.",
              "queryValenceKernelRequirements");
      }
      break;
    }
    break;
  case PrecisionModel::SINGLE:
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (acc_meth) {
        case ForceAccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceEnergyAccumulation)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfsValenceForceEnergyAccumulation.",
                    "queryValenceKernelRequirements");
            }
            break;
          case VwuGoal::MOVE_PARTICLES:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceEnergyAtomUpdate)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfsValenceEnergyAtomUpdate.",
                    "queryValenceKernelRequirements");
            }
            break;
          }
          break;
        case ForceAccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceEnergyAccumulation)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfValenceForceEnergyAccumulation.",
                    "queryValenceKernelRequirements");
            }
            break;
          case VwuGoal::MOVE_PARTICLES:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAtomUpdate)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfValenceEnergyAtomUpdate.",
                    "queryValenceKernelRequirements");
            }
            break;
          }
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (acc_meth) {
        case ForceAccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceAccumulation)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfsValenceForceAccumulation.",
                    "queryValenceKernelRequirements");
            }
            break;
          case VwuGoal::MOVE_PARTICLES:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceAtomUpdate)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfsValenceAtomUpdate.",
                    "queryValenceKernelRequirements");
            }
            break;
          }
          break;
        case ForceAccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceAccumulation)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfValenceForceAccumulation.",
                    "queryValenceKernelRequirements");
            }
            break;
          case VwuGoal::MOVE_PARTICLES:
            if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceAtomUpdate)) != hipSuccess) {
              rtErr("Error obtaining attributes for kernel kfValenceAtomUpdate.",
                    "queryValenceKernelRequirements");
            }
            break;
          }
          break;
        }
        break;
      }
      break;
    case EvaluateForce::NO:
      if (hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAccumulation)) != hipSuccess) {
        rtErr("Error obtaining attributes for kernel kfValenceEnergyAccumulation.",
              "queryValenceKernelRequirements");
      }
      break;
    }
    break;
  }
  return result;
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const SyValenceKit<double> &poly_vk,
                          const SyRestraintKit<double, double2, double4> &poly_rk,
                          MMControlKit<double> *ctrl, PsSynthesisWriter *poly_psw,
                          ScoreCardWriter *scw, CacheResourceKit<double> *gmem_r,
                          const EvaluateForce eval_force, const EvaluateEnergy eval_energy,
                          const VwuGoal purpose, const int2 bt) {
  switch (purpose) {
  case VwuGoal::ACCUMULATE:

    // When the goal is to accumulate energies, forces, or both, the force accumulation method
    // is set to use int64 data.  A 95-bit method that splits the accumulation with overflow into
    // a secondary 32-bit int may be added, and likewise become the sole option for
    // double-precision computations.
    switch (eval_force) {
    case EvaluateForce::YES:
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kdsValenceForceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                          *scw, *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kdsValenceForceAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
        break;
      }
      break;
    case EvaluateForce::NO:
      kdsValenceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                   *gmem_r);
      break;
    }
    break;
  case VwuGoal::MOVE_PARTICLES:

    // When the goal is to move particles, evaluating the force is obligatory, but the manner in
    // which forces are accumulated is still important.  Whether to accumulate energies while
    // evaluating forces and moving the particles remains a consideration in choosing the proper
    // kernel.
    switch (eval_energy) {
    case EvaluateEnergy::YES:
      kdsValenceEnergyAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                 *gmem_r);
      break;
    case EvaluateEnergy::NO:
      kdsValenceAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
      break;
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const SyValenceKit<float> &poly_vk,
                          const SyRestraintKit<float, float2, float4> &poly_rk,
                          MMControlKit<float> *ctrl, PsSynthesisWriter *poly_psw,
                          ScoreCardWriter *scw, CacheResourceKit<float> *gmem_r,
                          const EvaluateForce eval_force, const EvaluateEnergy eval_energy,
                          const VwuGoal purpose, const ForceAccumulationMethod force_sum,
                          const int2 bt) {
  ForceAccumulationMethod refined_force_sum;
  switch (force_sum) {
  case ForceAccumulationMethod::SPLIT:
  case ForceAccumulationMethod::WHOLE:
    refined_force_sum = force_sum;
    break;
  case ForceAccumulationMethod::AUTOMATIC:
    refined_force_sum = chooseForceAccumulationMethod(poly_psw->frc_bits);
    break;
  }
  switch (purpose) {
  case VwuGoal::ACCUMULATE:
    
    // When the goal is to accumulate energies, forces, or both, the force accumulation method
    // becomes a critical detail when choosing the kernel.
    switch (eval_force) {
    case EvaluateForce::YES:
      switch (refined_force_sum) {
      case ForceAccumulationMethod::SPLIT:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfsValenceForceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                            *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfsValenceForceAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
          break;
        }
        break;
      case ForceAccumulationMethod::WHOLE:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfValenceForceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                           *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfValenceForceAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
          break;
        }
        break;
      case ForceAccumulationMethod::AUTOMATIC:
        break;
      }
      break;
    case EvaluateForce::NO:
      kfValenceEnergyAccumulation<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                  *gmem_r);
      break;
    }
    break;
  case VwuGoal::MOVE_PARTICLES:

    // When the goal is to move particles, evaluating the force is obligatory, but the manner in
    // which forces are accumulated is still important.  Whether to accumulate energies while
    // evaluating forces and moving the particles remains a consideration in choosing the proper
    // kernel.
    switch (refined_force_sum) {
    case ForceAccumulationMethod::SPLIT:
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kfsValenceEnergyAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                   *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kfsValenceAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
        break;
      }
      break;
    case ForceAccumulationMethod::WHOLE:
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kfValenceEnergyAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                  *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kfValenceAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *gmem_r);
        break;
      }
      break;
    case ForceAccumulationMethod::AUTOMATIC:
      break;
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const PrecisionModel prec, const AtomGraphSynthesis &poly_ag,
                          MolecularMechanicsControls *mmctrl, PhaseSpaceSynthesis *poly_ps,
                          ScoreCard *sc, CacheResource *tb_space, const EvaluateForce eval_force,
                          const EvaluateEnergy eval_energy, const VwuGoal purpose,
                          const ForceAccumulationMethod force_sum, const KernelManager &launcher) {
  const HybridTargetLevel tier = HybridTargetLevel::DEVICE;
  PsSynthesisWriter poly_psw = poly_ps->data(tier);
  ScoreCardWriter scw = sc->data(tier);
  const int2 bt = launcher.getValenceKernelDims(prec, eval_force, eval_energy,
                                                ForceAccumulationMethod::SPLIT, purpose);
  switch (prec) {
  case PrecisionModel::DOUBLE:
    {
      const SyValenceKit<double> poly_vk = poly_ag.getDoublePrecisionValenceKit(tier);
      const SyRestraintKit<double,
                           double2,
                           double4> poly_rk = poly_ag.getDoublePrecisionRestraintKit(tier);
      MMControlKit<double> ctrl = mmctrl->dpData(tier);
      CacheResourceKit<double> gmem_r = tb_space->dpData(tier);
      launchValence(poly_vk, poly_rk, &ctrl, &poly_psw, &scw, &gmem_r, eval_force, eval_energy,
                    purpose, bt);
    }
    break;
  case PrecisionModel::SINGLE:
    {
      const SyValenceKit<float> poly_vk = poly_ag.getSinglePrecisionValenceKit(tier);
      const SyRestraintKit<float,
                           float2, float4> poly_rk = poly_ag.getSinglePrecisionRestraintKit(tier);
      MMControlKit<float> ctrl = mmctrl->spData(tier);
      CacheResourceKit<float> gmem_r = tb_space->spData(tier);
      launchValence(poly_vk, poly_rk, &ctrl, &poly_psw, &scw, &gmem_r, eval_force, eval_energy,
                    purpose, force_sum, bt);
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const PrecisionModel prec, const AtomGraphSynthesis &poly_ag,
                          MolecularMechanicsControls *mmctrl, PhaseSpaceSynthesis *poly_ps,
                          ScoreCard *sc, CacheResource *tb_space, const EvaluateForce eval_force,
                          const EvaluateEnergy eval_energy, const VwuGoal purpose,
                          const KernelManager &launcher) {
  if (prec == PrecisionModel::DOUBLE || poly_ps->getForceAccumulationBits() <= 24) {
    launchValence(prec, poly_ag, mmctrl, poly_ps, sc, tb_space, eval_force, eval_energy, purpose,
                  ForceAccumulationMethod::SPLIT, launcher);
  }
  else {
    launchValence(prec, poly_ag, mmctrl, poly_ps, sc, tb_space, eval_force, eval_energy, purpose,
                  ForceAccumulationMethod::WHOLE, launcher);
  }
}

} // namespace energy
} // namespace stormm
