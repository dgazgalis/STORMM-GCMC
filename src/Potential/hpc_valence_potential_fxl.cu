#include "hip/hip_runtime.h"
// -*-c++-*-
#include "copyright.h"
#include "Accelerator/ptx_macros.h"
#include "Accelerator/gpu_details.h"
#include "Constants/hpc_bounds.h"
#include "Constants/scaling.h"
#include "Constants/symbol_values.h"
#include "DataTypes/common_types.h"
#include "DataTypes/stormm_vector_types.h"
#include "Math/rounding.h"
#include "Numerics/numeric_enumerators.h"
#include "Numerics/split_fixed_precision.h"
#include "Potential/cellgrid.h"
#include "Potential/energy_enumerators.h"
#include "Synthesis/valence_workunit.h"
#include "Topology/atomgraph_abstracts.h"
#include "Topology/atomgraph_enumerators.h"
#include "Trajectory/trajectory_enumerators.h"
#include "hpc_valence_potential.h"

namespace stormm {
namespace energy {

using constants::large_block_size;
using constants::medium_block_size;
using constants::small_block_size;
using constants::twice_warp_bits_mask_int;
using constants::twice_warp_size_int;
using constants::warp_size_int;
using constants::warp_bits;
using constants::warp_bits_mask_int;
using numerics::chooseAccumulationMethod;
using numerics::getEnumerationName;
using stmath::roundUp;
using symbols::asymptotic_to_one_f;
using symbols::asymptotic_to_one_lf;
using symbols::boltzmann_constant_f;
using symbols::gafs_to_kcal_f;
using symbols::inverse_one_minus_asymptote_f;
using symbols::inverse_one_minus_asymptote_lf;
using symbols::inverse_twopi_f;
using symbols::kcal_to_gafs_f;
using symbols::near_to_one_f;
using symbols::near_to_one_lf;
using symbols::pi;
using symbols::pi_f;
using symbols::twopi;
using symbols::twopi_f;
using synthesis::maximum_valence_work_unit_atoms;
using synthesis::half_valence_work_unit_atoms;
using synthesis::quarter_valence_work_unit_atoms;
using synthesis::eighth_valence_work_unit_atoms;
using synthesis::VwuAbstractMap;
using synthesis::vwu_abstract_length;
using trajectory::ThermostatKind;
using trajectory::ThermostatPartition;
using topology::TorsionKind;
using topology::VirtualSiteKind;

#include "Accelerator/syncwarp.cui"
#include "Math/rounding.cui"
#include "Math/vector_formulas.cui"
#include "Numerics/accumulation.cui"
#include "Trajectory/thermostat_utilities.cui"
#include "valence_util.cui"

// Single-precision floating point definitions
#define TCALC float
#  define TCALC2 float2
#  define TCALC3 float3
#  define TCALC4 float4
#  define LLCONV_FUNC __float2ll_rn
#  define SPLITCONV_FUNC floatToInt63
#  define SPLIT_TYPE int2
#  define SQRT_FUNC sqrtf
#  define CBRT_FUNC cbrtf
#  define ACOS_FUNC acosf
#  define COS_FUNC  cosf
#  define SIN_FUNC  sinf
#  define ABS_FUNC  fabsf
#  define MIX_FUNC  computeRestraintMixtureF
#  define TCALC_IS_SINGLE

#  define VALENCE_BLOCK_MULTIPLICITY 2

// Compile the standard kernels with all combinations of energy and force accumulation methods.
#  define COMPUTE_FORCE
#    define SPLIT_FORCE_ACCUMULATION
#      define VALENCE_KERNEL_THREAD_COUNT 512
#        define KERNEL_NAME kfsValenceForceAccumulationXL
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define UPDATE_ATOMS
#        define VALENCE_KERNEL_THREAD_COUNT 384
#          define KERNEL_NAME kfsValenceAtomUpdateXL
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef UPDATE_ATOMS
#      define COMPUTE_ENERGY
#        define VALENCE_KERNEL_THREAD_COUNT 448
#          define KERNEL_NAME kfsValenceForceEnergyAccumulationXL
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#        define UPDATE_ATOMS
#          define VALENCE_KERNEL_THREAD_COUNT 320
#            define KERNEL_NAME kfsValenceEnergyAtomUpdateXL
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_KERNEL_THREAD_COUNT
#        undef UPDATE_ATOMS
#      undef COMPUTE_ENERGY
#    undef SPLIT_FORCE_ACCUMULATION
#    define VALENCE_KERNEL_THREAD_COUNT 512
#      define KERNEL_NAME kfValenceForceAccumulationXL
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef VALENCE_KERNEL_THREAD_COUNT
#    define UPDATE_ATOMS
#      define VALENCE_KERNEL_THREAD_COUNT 384
#        define KERNEL_NAME kfValenceAtomUpdateXL
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef VALENCE_KERNEL_THREAD_COUNT
#    undef UPDATE_ATOMS
#    define COMPUTE_ENERGY
#      define VALENCE_KERNEL_THREAD_COUNT 448
#        define KERNEL_NAME kfValenceForceEnergyAccumulationXL
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define UPDATE_ATOMS
#        define VALENCE_KERNEL_THREAD_COUNT 320
#          define KERNEL_NAME kfValenceEnergyAtomUpdateXL
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef UPDATE_ATOMS
#    undef COMPUTE_ENERGY
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define VALENCE_KERNEL_THREAD_COUNT 512
#      define KERNEL_NAME kfValenceEnergyAccumulationXL
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef VALENCE_KERNEL_THREAD_COUNT
#  undef COMPUTE_ENERGY

// Make new kernels with a clash forgiveness check.
#  define CLASH_FORGIVENESS
#    define COMPUTE_FORCE
#      define SPLIT_FORCE_ACCUMULATION
#        define VALENCE_KERNEL_THREAD_COUNT 512
#          define KERNEL_NAME kfsValenceForceAccumulationNonClashXL
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#        define UPDATE_ATOMS
#          define VALENCE_KERNEL_THREAD_COUNT 384
#            define KERNEL_NAME kfsValenceAtomUpdateNonClashXL
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_KERNEL_THREAD_COUNT
#        undef UPDATE_ATOMS
#        define COMPUTE_ENERGY
#          define VALENCE_KERNEL_THREAD_COUNT 384
#            define KERNEL_NAME kfsValenceForceEnergyAccumulationNonClashXL
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_KERNEL_THREAD_COUNT
#          define UPDATE_ATOMS
#            define VALENCE_KERNEL_THREAD_COUNT 320
#              define KERNEL_NAME kfsValenceEnergyAtomUpdateNonClashXL
#                include "valence_potential.cui"
#              undef KERNEL_NAME
#            undef VALENCE_KERNEL_THREAD_COUNT
#          undef UPDATE_ATOMS
#        undef COMPUTE_ENERGY
#      undef SPLIT_FORCE_ACCUMULATION
#      define VALENCE_KERNEL_THREAD_COUNT 512
#        define KERNEL_NAME kfValenceForceAccumulationNonClashXL
#          include "valence_potential.cui"
#        undef KERNEL_NAME  
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define UPDATE_ATOMS
#        define VALENCE_KERNEL_THREAD_COUNT 384
#          define KERNEL_NAME kfValenceAtomUpdateNonClashXL
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef UPDATE_ATOMS
#      define COMPUTE_ENERGY
#        define VALENCE_KERNEL_THREAD_COUNT 384
#          define KERNEL_NAME kfValenceForceEnergyAccumulationNonClashXL
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#        define UPDATE_ATOMS
#          define VALENCE_KERNEL_THREAD_COUNT 320
#            define KERNEL_NAME kfValenceEnergyAtomUpdateNonClashXL
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_KERNEL_THREAD_COUNT
#        undef UPDATE_ATOMS
#      undef COMPUTE_ENERGY
#    undef COMPUTE_FORCE
#    define COMPUTE_ENERGY
#      define VALENCE_KERNEL_THREAD_COUNT 512
#        define KERNEL_NAME kfValenceEnergyAccumulationNonClashXL
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef VALENCE_KERNEL_THREAD_COUNT
#    undef COMPUTE_ENERGY
#  undef CLASH_FORGIVENESS

#  undef VALENCE_BLOCK_MULTIPLICITY

// Clear single-precision floating point definitions
#  undef TCALC2
#  undef TCALC3
#  undef TCALC4
#  undef LLCONV_FUNC
#  undef SPLITCONV_FUNC
#  undef SPLIT_TYPE
#  undef SQRT_FUNC
#  undef CBRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef MIX_FUNC
#  undef TCALC_IS_SINGLE
#undef TCALC

//-------------------------------------------------------------------------------------------------
#ifdef STORMM_USE_CUDA
extern hipFuncAttributes
queryValenceKernelRequirementsXL(const EvaluateForce eval_frc, const EvaluateEnergy eval_nrg,
                                 const AccumulationMethod acc_meth, const VwuGoal purpose,
                                 const ClashResponse collision_handling) {

  
  // The kernel manager will have information about the GPU to use--look at the work units from
  // the perspective of overall occupancy on the GPU.
  hipFuncAttributes result;
  hipError_t cfa = hipErrorInvalidValue;
  switch (collision_handling) {
  case ClashResponse::NONE:
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceEnergyAccumulationXL));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceEnergyAtomUpdateXL));
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceEnergyAccumulationXL));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAtomUpdateXL));
            break;
          }
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceAccumulationXL));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceAtomUpdateXL));
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceAccumulationXL));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceAtomUpdateXL));
            break;
          }
          break;
        }
        break;
      }
      break;
    case EvaluateForce::NO:
      cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAccumulationXL));
      break;
    }
    break;
  case ClashResponse::FORGIVE:
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceEnergyAccumulationNonClashXL));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceEnergyAtomUpdateNonClashXL));
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceEnergyAccumulationNonClashXL));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAtomUpdateNonClashXL));
            break;
          }
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceAccumulationNonClashXL));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceAtomUpdateNonClashXL));
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceAccumulationNonClashXL));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceAtomUpdateNonClashXL));
            break;
          }
          break;
        }
        break;
      }
      break;
    case EvaluateForce::NO:
      cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAccumulationNonClashXL));
      break;
    }
    break;
  }
  
  // Check for errors
  if (cfa != hipSuccess) {

    // Construct the appropriate error message
    std::string error_message("Error obtaining attributes for kernel k");
    switch (acc_meth) {
    case AccumulationMethod::SPLIT:
      error_message += "fs";
      break;
    case AccumulationMethod::WHOLE:
      error_message += "f";
      break;
    case AccumulationMethod::AUTOMATIC:
      rtErr("Kernels do not accept " + getEnumerationName(acc_meth) + " accumulation.",
            "queryValenceKernelRequirements");
      break;
    }
    error_message += "Valence";
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (purpose) {
        case VwuGoal::ACCUMULATE:
          error_message += "ForceEnergyAccumulation";
          break;
        case VwuGoal::MOVE_PARTICLES:
          error_message += "EnergyAtomUpdate";
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (purpose) {
        case VwuGoal::ACCUMULATE:
          error_message += "ForceAccumulation";
          break;
        case VwuGoal::MOVE_PARTICLES:
          error_message += "AtomUpdate";
          break;
        }
        break;
      }
      break;
    case EvaluateForce::NO:
      error_message += "EnergyAccumulation";
      break;
    }
    error_message += "XL.";

    // Report the error
    rtErr(error_message, "queryValenceKernelRequirementsXL");
  }
  
  return result;
}
#endif // STORMM_USE_CUDA

//-------------------------------------------------------------------------------------------------
extern void launchValenceXL(const SyValenceKit<float> &poly_vk,
                            const SyRestraintKit<float, float2, float4> &poly_rk,
                            MMControlKit<float> *ctrl, PsSynthesisWriter *poly_psw,
                            const SyAtomUpdateKit<float, float2, float4> &poly_auk,
                            ThermostatWriter<float> *tstw, ScoreCardWriter *scw,
                            CacheResourceKit<float> *gmem_r, const EvaluateForce eval_force,
                            const EvaluateEnergy eval_energy, const VwuGoal purpose,
                            const AccumulationMethod refined_force_sum, const int2 bt,
                            const float clash_distance, const float clash_ratio) {
  
  // Rather than a switch over cases of the ClashResponse enumerator, just use the nonzero values
  // of either parameter to indicate that clash damping has been requested.
  if (clash_distance >= 1.0e-6 || clash_ratio >= 1.0e-6) {
    switch (purpose) {
    case VwuGoal::ACCUMULATE:
    
      // When the goal is to accumulate energies, forces, or both, the force accumulation method
      // becomes a critical detail when choosing the kernel.
      switch (eval_force) {
      case EvaluateForce::YES:
        switch (refined_force_sum) {
        case AccumulationMethod::SPLIT:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            kfsValenceForceEnergyAccumulationNonClashXL<<<bt.x,
                                                          bt.y>>>(poly_vk, poly_rk, *ctrl,
                                                                  *poly_psw, clash_distance,
                                                                  clash_ratio, *scw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            kfsValenceForceAccumulationNonClashXL<<<bt.x,
                                                    bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                            clash_distance, clash_ratio,
                                                            *gmem_r);
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            kfValenceForceEnergyAccumulationNonClashXL<<<bt.x,
                                                         bt.y>>>(poly_vk, poly_rk, *ctrl,
                                                                 *poly_psw, clash_distance,
                                                                 clash_ratio, *scw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            kfValenceForceAccumulationNonClashXL<<<bt.x,
                                                   bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                           clash_distance, clash_ratio, *gmem_r);
            break;
          }
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        break;
      case EvaluateForce::NO:
        kfValenceEnergyAccumulationNonClashXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                              clash_distance, clash_ratio, *scw,
                                                              *gmem_r);
        break;
      }
      break;
    case VwuGoal::MOVE_PARTICLES:
    
      // When the goal is to move particles, evaluating the force is obligatory, but the manner in
      // which forces are accumulated is still important.  Whether to accumulate energies while
      // evaluating forces and moving the particles remains a consideration in choosing the proper
      // kernel.
      switch (refined_force_sum) {
      case AccumulationMethod::SPLIT:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfsValenceEnergyAtomUpdateNonClashXL<<<bt.x,
                                                 bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                         clash_distance, clash_ratio, poly_auk,
                                                         *tstw, *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfsValenceAtomUpdateNonClashXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                         clash_distance, clash_ratio, poly_auk,
                                                         *tstw, *gmem_r);
          break;
        }
        break;
      case AccumulationMethod::WHOLE:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfValenceEnergyAtomUpdateNonClashXL<<<bt.x,
                                                bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                        clash_distance, clash_ratio, poly_auk,
                                                        *tstw, *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfValenceAtomUpdateNonClashXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                        clash_distance, clash_ratio, poly_auk,
                                                        *tstw,*gmem_r);
          break;
        }
        break;
      case AccumulationMethod::AUTOMATIC:
        break;
      }
      break;
    }
  }
  else {
    switch (purpose) {
    case VwuGoal::ACCUMULATE:

      // See above for the rationale on whether forces or energies are evaluated in each context.
      switch (eval_force) {
      case EvaluateForce::YES:
        switch (refined_force_sum) {
        case AccumulationMethod::SPLIT:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            kfsValenceForceEnergyAccumulationXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl,
                                                                *poly_psw, *scw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            kfsValenceForceAccumulationXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                          *gmem_r);
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            kfValenceForceEnergyAccumulationXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl,
                                                               *poly_psw, *scw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            kfValenceForceAccumulationXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                         *gmem_r);
            break;
          }
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        break;
      case EvaluateForce::NO:
        kfValenceEnergyAccumulationXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                      *gmem_r);
        break;
      }
      break;
    case VwuGoal::MOVE_PARTICLES:
    
      // See above for the rationale on the choice of each kernel.
      switch (refined_force_sum) {
      case AccumulationMethod::SPLIT:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfsValenceEnergyAtomUpdateXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                       poly_auk, *tstw, *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfsValenceAtomUpdateXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk,
                                                 *tstw, *gmem_r);
          break;
        }
        break;
      case AccumulationMethod::WHOLE:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfValenceEnergyAtomUpdateXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                      poly_auk, *tstw, *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfValenceAtomUpdateXL<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk,
                                                *tstw, *gmem_r);
          break;
        }
        break;
      case AccumulationMethod::AUTOMATIC:
        break;
      }
      break;
    }
  }
}

} // namespace energy
} // namespace stormm
