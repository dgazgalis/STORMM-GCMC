#include "hip/hip_runtime.h"
// -*-c++-*-
#include "copyright.h"
#include "Accelerator/ptx_macros.h"
#include "Accelerator/gpu_details.h"
#include "Constants/hpc_bounds.h"
#include "Constants/scaling.h"
#include "Constants/symbol_values.h"
#include "DataTypes/common_types.h"
#include "DataTypes/stormm_vector_types.h"
#include "Math/rounding.h"
#include "Numerics/numeric_enumerators.h"
#include "Numerics/split_fixed_precision.h"
#include "Potential/cellgrid.h"
#include "Potential/energy_enumerators.h"
#include "Synthesis/valence_workunit.h"
#include "Topology/atomgraph_abstracts.h"
#include "Topology/atomgraph_enumerators.h"
#include "Trajectory/trajectory_enumerators.h"
#include "hpc_valence_potential.h"

namespace stormm {
namespace energy {

using constants::large_block_size;
using constants::medium_block_size;
using constants::small_block_size;
using constants::twice_warp_bits_mask_int;
using constants::twice_warp_size_int;
using constants::warp_size_int;
using constants::warp_bits;
using constants::warp_bits_mask_int;
using numerics::chooseAccumulationMethod;
using numerics::getEnumerationName;
using stmath::roundUp;
using symbols::asymptotic_to_one_f;
using symbols::asymptotic_to_one_lf;
using symbols::boltzmann_constant_f;
using symbols::gafs_to_kcal_f;
using symbols::inverse_one_minus_asymptote_f;
using symbols::inverse_one_minus_asymptote_lf;
using symbols::inverse_twopi_f;
using symbols::kcal_to_gafs_f;
using symbols::near_to_one_f;
using symbols::near_to_one_lf;
using symbols::pi;
using symbols::pi_f;
using symbols::twopi;
using symbols::twopi_f;
using synthesis::maximum_valence_work_unit_atoms;
using synthesis::half_valence_work_unit_atoms;
using synthesis::quarter_valence_work_unit_atoms;
using synthesis::eighth_valence_work_unit_atoms;
using synthesis::VwuAbstractMap;
using synthesis::vwu_abstract_length;
using trajectory::ThermostatKind;
using trajectory::ThermostatPartition;
using topology::TorsionKind;
using topology::VirtualSiteKind;

#include "Accelerator/syncwarp.cui"
#include "Math/rounding.cui"
#include "Math/vector_formulas.cui"
#include "Numerics/accumulation.cui"
#include "Trajectory/thermostat_utilities.cui"
#include "valence_util.cui"

// Double-precision floating point definitions
#define TCALC double
#  define VALENCE_BLOCK_MULTIPLICITY  2
#  define TCALC2 double2
#  define TCALC3 double3
#  define TCALC4 double4
#  define LLCONV_FUNC __double2ll_rn
#  define SPLITCONV_FUNC doubleToInt95
#  define SPLIT_TYPE int95_t
#  define SQRT_FUNC sqrt
#  define CBRT_FUNC cbrt
#  define ACOS_FUNC acos
#  define COS_FUNC  cos
#  define SIN_FUNC  sin
#  define ABS_FUNC  fabs
#  define MIX_FUNC  computeRestraintMixtureD
#  define SPLIT_FORCE_ACCUMULATION

// Define PME-compatible variants of kernels invoking double-precision arithmetic, beginning with
// single neighbor list cell grids.  TCALC is inherent to the valence kernel, but the coordinate
// type TCOORD and associated TACC may be defined independently.  Begin with double-precision
// coordinates and 95-bit force accumulation in the neighbor list cell grids.
#  define PME_COMPATIBLE
#  define UPDATE_ATOMS
#  define COMPUTE_FORCE
#  define TCOORD double
#  define TACC llint
#  define TCOORD4 double4
#  define TCOORD_IS_LONG
#    define VALENCE_KERNEL_THREAD_COUNT 192
#      define KERNEL_NAME kdsdPmeValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef VALENCE_KERNEL_THREAD_COUNT
#    define VALENCE_KERNEL_THREAD_COUNT 192
#      define COMPUTE_ENERGY
#        define KERNEL_NAME kdsdPmeValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef  COMPUTE_ENERGY
#    undef VALENCE_KERNEL_THREAD_COUNT
#    define CLASH_FORGIVENESS
#      define VALENCE_KERNEL_THREAD_COUNT 192
#        define KERNEL_NAME kdsdPmeValenceAtomUpdateNonClash
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define COMPUTE_ENERGY
#        define VALENCE_KERNEL_THREAD_COUNT 192
#          define KERNEL_NAME kdsdPmeValenceEnergyAtomUpdateNonClash
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef  COMPUTE_ENERGY
#    undef CLASH_FORGIVENESS
#  undef TCOORD_IS_LONG
#  undef TCOORD4
#  undef TACC
#  undef TCOORD
  
// Define additional PME-compatible kernels for use with float coordinates and int63 accumulation
// in the neighbor list's non-bonded forces.
#  define TCOORD float
#  define TACC int
#  define TCOORD4 float4
#    define VALENCE_KERNEL_THREAD_COUNT 192
#      define KERNEL_NAME kdsfPmeValenceAtomUpdate
#        include "valence_potential.cui"
#      undef KERNEL_NAME
#    undef VALENCE_KERNEL_THREAD_COUNT
#    define VALENCE_KERNEL_THREAD_COUNT 192
#      define COMPUTE_ENERGY
#        define KERNEL_NAME kdsfPmeValenceEnergyAtomUpdate
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef  COMPUTE_ENERGY
#    undef VALENCE_KERNEL_THREAD_COUNT
#    define CLASH_FORGIVENESS
#      define VALENCE_KERNEL_THREAD_COUNT 192
#        define KERNEL_NAME kdsfPmeValenceAtomUpdateNonClash
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define COMPUTE_ENERGY
#        define VALENCE_KERNEL_THREAD_COUNT 192
#          define KERNEL_NAME kdsfPmeValenceEnergyAtomUpdateNonClash
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef  COMPUTE_ENERGY
#    undef CLASH_FORGIVENESS
#  undef TCOORD4
#  undef TACC
#  undef TCOORD
#  undef COMPUTE_FORCE
#  undef UPDATE_ATOMS
#  undef PME_COMPATIBLE

// Clear double-precision floating point definitions
#  undef VALENCE_BLOCK_MULTIPLICITY
#  undef TCALC2
#  undef TCALC3
#  undef TCALC4
#  undef LLCONV_FUNC
#  undef SPLITCONV_FUNC
#  undef SPLIT_TYPE
#  undef SQRT_FUNC
#  undef CBRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef MIX_FUNC
#  undef SPLIT_FORCE_ACCUMULATION
#undef TCALC

//-------------------------------------------------------------------------------------------------
#ifdef STORMM_USE_CUDA
extern hipFuncAttributes
queryValenceKernelRequirementsDPME(const EvaluateEnergy eval_nrg,
                                   const ClashResponse collision_handling,
                                   const PrecisionModel neighbor_prec) {
  
  // The kernel manager will have information about the GPU to use--look at the work units from
  // the perspective of overall occupancy on the GPU.
  hipFuncAttributes result;
  hipError_t cfa = hipErrorInvalidValue;
  switch (collision_handling) {
  case ClashResponse::NONE:
    switch (eval_nrg) {
    case EvaluateEnergy::YES:
      switch (neighbor_prec) {
      case PrecisionModel::DOUBLE:
        cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsdPmeValenceEnergyAtomUpdate));
        break;
      case PrecisionModel::SINGLE:
        cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsfPmeValenceEnergyAtomUpdate));
        break;
      }
      break;
    case EvaluateEnergy::NO:
      switch (neighbor_prec) {
      case PrecisionModel::DOUBLE:
        cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsdPmeValenceAtomUpdate));
        break;
      case PrecisionModel::SINGLE:
        cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsfPmeValenceAtomUpdate));
        break;
      }
      break;
    }
    break;
  case ClashResponse::FORGIVE:
    switch (eval_nrg) {
    case EvaluateEnergy::YES:
      switch (neighbor_prec) {
      case PrecisionModel::DOUBLE:
        cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsdPmeValenceEnergyAtomUpdateNonClash));
        break;
      case PrecisionModel::SINGLE:
        cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsfPmeValenceEnergyAtomUpdateNonClash));
        break;
      }
      break;
    case EvaluateEnergy::NO:
      switch (neighbor_prec) {
      case PrecisionModel::DOUBLE:
        cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsdPmeValenceAtomUpdateNonClash));
        break;
      case PrecisionModel::SINGLE:
        cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kdsfPmeValenceAtomUpdateNonClash));
        break;
      }
      break;
    }
    break;
  }
  
  // Check for errors
  if (cfa != hipSuccess) {

    // Construct the appropriate error message
    std::string error_message("Error obtaining attributes for kernel kds");
    switch (neighbor_prec) {
    case PrecisionModel::DOUBLE:
      error_message += "d";
      break;
    case PrecisionModel::SINGLE:
      error_message += "f";
      break;
    }
    error_message += "PmeValence";
    switch (eval_nrg) {
    case EvaluateEnergy::YES:
      error_message += "EnergyAtomUpdate";
      break;
    case EvaluateEnergy::NO:
      error_message += "AtomUpdate";
      break;
    }
    error_message += ".";

    // Report the error
    rtErr(error_message, "queryValenceKernelRequirementsDPME");
  }
  
  return result;
}
#endif

//-------------------------------------------------------------------------------------------------
extern void launchValence(const SyValenceKit<double> &poly_vk,
                          const SyRestraintKit<double, double2, double4> &poly_rk,
                          const CellGridReader<double, llint, double, double4> &cgr,
                          MMControlKit<double> *ctrl, PsSynthesisWriter *poly_psw,
                          const SyAtomUpdateKit<double, double2, double4> &poly_auk,
                          ThermostatWriter<double> *tstw, ScoreCardWriter *scw,
                          CacheResourceKit<double> *gmem_r, const EvaluateForce eval_force,
                          const EvaluateEnergy eval_energy, const VwuGoal purpose, const int2 bt,
                          const double clash_distance, const double clash_ratio) {
  switch (purpose) {
  case VwuGoal::ACCUMULATE:
    launchValence(poly_vk, poly_rk, ctrl, poly_psw, poly_auk, tstw, scw, gmem_r, eval_force,
                  eval_energy, purpose, bt, clash_distance, clash_ratio);
    break;
  case VwuGoal::MOVE_PARTICLES:
    if (clash_distance >= 1.0e-6 || clash_ratio >= 1.0e-6) {
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kdsdPmeValenceEnergyAtomUpdateNonClash<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                               cgr, clash_distance, clash_ratio,
                                                               poly_auk, *tstw, *scw, *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kdsdPmeValenceAtomUpdateNonClash<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, cgr,
                                                         clash_distance, clash_ratio, poly_auk,
                                                         *tstw, *gmem_r);
        break;
      }
    }
    else {
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kdsdPmeValenceEnergyAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, cgr,
                                                       poly_auk, *tstw, *scw, *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kdsdPmeValenceAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, cgr,
                                                 poly_auk, *tstw, *gmem_r);
        break;
      }
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
extern void launchValence(const SyValenceKit<double> &poly_vk,
                          const SyRestraintKit<double, double2, double4> &poly_rk,
                          const CellGridReader<float, int, float, float4> &cgr,
                          MMControlKit<double> *ctrl, PsSynthesisWriter *poly_psw,
                          const SyAtomUpdateKit<double, double2, double4> &poly_auk,
                          ThermostatWriter<double> *tstw, ScoreCardWriter *scw,
                          CacheResourceKit<double> *gmem_r, const EvaluateForce eval_force,
                          const EvaluateEnergy eval_energy, const VwuGoal purpose, const int2 bt,
                          const double clash_distance, const double clash_ratio) {
  switch (purpose) {
  case VwuGoal::ACCUMULATE:
    launchValence(poly_vk, poly_rk, ctrl, poly_psw, poly_auk, tstw, scw, gmem_r, eval_force,
                  eval_energy, purpose, bt, clash_distance, clash_ratio);
    break;
  case VwuGoal::MOVE_PARTICLES:
    if (clash_distance >= 1.0e-6 || clash_ratio >= 1.0e-6) {
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kdsfPmeValenceEnergyAtomUpdateNonClash<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                               cgr, clash_distance, clash_ratio,
                                                               poly_auk, *tstw, *scw, *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kdsfPmeValenceAtomUpdateNonClash<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, cgr,
                                                         clash_distance, clash_ratio, poly_auk,
                                                         *tstw, *gmem_r);
        break;
      }
    }
    else {
      switch (eval_energy) {
      case EvaluateEnergy::YES:
        kdsfPmeValenceEnergyAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, cgr,
                                                       poly_auk, *tstw, *scw, *gmem_r);
        break;
      case EvaluateEnergy::NO:
        kdsfPmeValenceAtomUpdate<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, cgr,
                                                 poly_auk, *tstw, *gmem_r);
        break;
      }
    }
    break;
  }
}

} // namespace energy
} // namespace stormm
