#include "hip/hip_runtime.h"
// -*-c++-*-
#include "copyright.h"
#include "Accelerator/ptx_macros.h"
#include "Accelerator/gpu_details.h"
#include "Constants/hpc_bounds.h"
#include "Constants/scaling.h"
#include "Constants/symbol_values.h"
#include "DataTypes/common_types.h"
#include "DataTypes/stormm_vector_types.h"
#include "Math/rounding.h"
#include "Numerics/numeric_enumerators.h"
#include "Numerics/split_fixed_precision.h"
#include "Potential/cellgrid.h"
#include "Potential/energy_enumerators.h"
#include "Synthesis/valence_workunit.h"
#include "Topology/atomgraph_abstracts.h"
#include "Topology/atomgraph_enumerators.h"
#include "Trajectory/trajectory_enumerators.h"
#include "hpc_valence_potential.h"

namespace stormm {
namespace energy {

using constants::large_block_size;
using constants::medium_block_size;
using constants::small_block_size;
using constants::twice_warp_bits_mask_int;
using constants::twice_warp_size_int;
using constants::warp_size_int;
using constants::warp_bits;
using constants::warp_bits_mask_int;
using numerics::chooseAccumulationMethod;
using numerics::getEnumerationName;
using stmath::roundUp;
using symbols::asymptotic_to_one_f;
using symbols::asymptotic_to_one_lf;
using symbols::boltzmann_constant_f;
using symbols::gafs_to_kcal_f;
using symbols::inverse_one_minus_asymptote_f;
using symbols::inverse_one_minus_asymptote_lf;
using symbols::inverse_twopi_f;
using symbols::kcal_to_gafs_f;
using symbols::near_to_one_f;
using symbols::near_to_one_lf;
using symbols::pi;
using symbols::pi_f;
using symbols::twopi;
using symbols::twopi_f;
using synthesis::maximum_valence_work_unit_atoms;
using synthesis::half_valence_work_unit_atoms;
using synthesis::quarter_valence_work_unit_atoms;
using synthesis::eighth_valence_work_unit_atoms;
using synthesis::VwuAbstractMap;
using synthesis::vwu_abstract_length;
using trajectory::ThermostatKind;
using trajectory::ThermostatPartition;
using topology::TorsionKind;
using topology::VirtualSiteKind;

#include "Accelerator/syncwarp.cui"
#include "Math/rounding.cui"
#include "Math/vector_formulas.cui"
#include "Numerics/accumulation.cui"
#include "Trajectory/thermostat_utilities.cui"
#include "valence_util.cui"

// Single-precision floating point definitions
#define TCALC float
#  define TCALC2 float2
#  define TCALC3 float3
#  define TCALC4 float4
#  define LLCONV_FUNC __float2ll_rn
#  define SPLITCONV_FUNC floatToInt63
#  define SPLIT_TYPE int2
#  define SQRT_FUNC sqrtf
#  define CBRT_FUNC cbrtf
#  define ACOS_FUNC acosf
#  define COS_FUNC  cosf
#  define SIN_FUNC  sinf
#  define ABS_FUNC  fabsf
#  define MIX_FUNC  computeRestraintMixtureF
#  define TCALC_IS_SINGLE

// Compile the standard kernels with all combinations of energy and force accumulation methods.
#  define COMPUTE_FORCE
#    define SPLIT_FORCE_ACCUMULATION
#      define VALENCE_KERNEL_THREAD_COUNT 128
#        define VALENCE_BLOCK_MULTIPLICITY 8
#          define KERNEL_NAME kfsValenceForceAccumulationMD
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_BLOCK_MULTIPLICITY
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define UPDATE_ATOMS
#        define VALENCE_KERNEL_THREAD_COUNT 128
#          define VALENCE_BLOCK_MULTIPLICITY 6
#            define KERNEL_NAME kfsValenceAtomUpdateMD
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_BLOCK_MULTIPLICITY
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef UPDATE_ATOMS
#      define COMPUTE_ENERGY
#        define VALENCE_KERNEL_THREAD_COUNT 128
#          define VALENCE_BLOCK_MULTIPLICITY 7
#            define KERNEL_NAME kfsValenceForceEnergyAccumulationMD
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_BLOCK_MULTIPLICITY
#        undef VALENCE_KERNEL_THREAD_COUNT
#        define UPDATE_ATOMS
#          define VALENCE_KERNEL_THREAD_COUNT 128
#            define VALENCE_BLOCK_MULTIPLICITY 5
#              define KERNEL_NAME kfsValenceEnergyAtomUpdateMD
#                include "valence_potential.cui"
#              undef KERNEL_NAME
#            undef VALENCE_BLOCK_MULTIPLICITY
#          undef VALENCE_KERNEL_THREAD_COUNT
#        undef UPDATE_ATOMS
#      undef COMPUTE_ENERGY
#    undef SPLIT_FORCE_ACCUMULATION
#    define VALENCE_KERNEL_THREAD_COUNT 128
#      define VALENCE_BLOCK_MULTIPLICITY 8
#        define KERNEL_NAME kfValenceForceAccumulationMD
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef VALENCE_BLOCK_MULTIPLICITY
#    undef VALENCE_KERNEL_THREAD_COUNT
#    define UPDATE_ATOMS
#      define VALENCE_KERNEL_THREAD_COUNT 128
#        define VALENCE_BLOCK_MULTIPLICITY 6
#          define KERNEL_NAME kfValenceAtomUpdateMD
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_BLOCK_MULTIPLICITY
#      undef VALENCE_KERNEL_THREAD_COUNT
#    undef UPDATE_ATOMS
#    define COMPUTE_ENERGY
#      define VALENCE_KERNEL_THREAD_COUNT 128
#        define VALENCE_BLOCK_MULTIPLICITY 7
#          define KERNEL_NAME kfValenceForceEnergyAccumulationMD
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_BLOCK_MULTIPLICITY
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define UPDATE_ATOMS
#        define VALENCE_KERNEL_THREAD_COUNT 128
#          define VALENCE_BLOCK_MULTIPLICITY 5
#            define KERNEL_NAME kfValenceEnergyAtomUpdateMD
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_BLOCK_MULTIPLICITY
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef UPDATE_ATOMS
#    undef COMPUTE_ENERGY
#  undef COMPUTE_FORCE
#  define COMPUTE_ENERGY
#    define VALENCE_KERNEL_THREAD_COUNT 128
#      define VALENCE_BLOCK_MULTIPLICITY 8
#        define KERNEL_NAME kfValenceEnergyAccumulationMD
#          include "valence_potential.cui"
#        undef KERNEL_NAME
#      undef VALENCE_BLOCK_MULTIPLICITY
#    undef VALENCE_KERNEL_THREAD_COUNT
#  undef COMPUTE_ENERGY

// Make new kernels with a clash forgiveness check.
#  define CLASH_FORGIVENESS
#    define COMPUTE_FORCE
#      define SPLIT_FORCE_ACCUMULATION
#        define VALENCE_KERNEL_THREAD_COUNT 128
#          define VALENCE_BLOCK_MULTIPLICITY 8
#            define KERNEL_NAME kfsValenceForceAccumulationNonClashMD
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_BLOCK_MULTIPLICITY
#        undef VALENCE_KERNEL_THREAD_COUNT
#        define UPDATE_ATOMS
#          define VALENCE_KERNEL_THREAD_COUNT 128
#            define VALENCE_BLOCK_MULTIPLICITY 6
#              define KERNEL_NAME kfsValenceAtomUpdateNonClashMD
#                include "valence_potential.cui"
#              undef KERNEL_NAME
#            undef VALENCE_BLOCK_MULTIPLICITY
#          undef VALENCE_KERNEL_THREAD_COUNT
#        undef UPDATE_ATOMS
#        define COMPUTE_ENERGY
#          define VALENCE_KERNEL_THREAD_COUNT 128
#            define VALENCE_BLOCK_MULTIPLICITY 6
#              define KERNEL_NAME kfsValenceForceEnergyAccumulationNonClashMD
#                include "valence_potential.cui"
#              undef KERNEL_NAME
#            undef VALENCE_BLOCK_MULTIPLICITY
#          undef VALENCE_KERNEL_THREAD_COUNT
#          define UPDATE_ATOMS
#            define VALENCE_KERNEL_THREAD_COUNT 128
#              define VALENCE_BLOCK_MULTIPLICITY 5
#                define KERNEL_NAME kfsValenceEnergyAtomUpdateNonClashMD
#                  include "valence_potential.cui"
#                undef KERNEL_NAME
#              undef VALENCE_BLOCK_MULTIPLICITY
#            undef VALENCE_KERNEL_THREAD_COUNT
#          undef UPDATE_ATOMS
#        undef COMPUTE_ENERGY
#      undef SPLIT_FORCE_ACCUMULATION
#      define VALENCE_KERNEL_THREAD_COUNT 128
#        define VALENCE_BLOCK_MULTIPLICITY 8
#          define KERNEL_NAME kfValenceForceAccumulationNonClashMD
#            include "valence_potential.cui"
#          undef KERNEL_NAME  
#        undef VALENCE_BLOCK_MULTIPLICITY
#      undef VALENCE_KERNEL_THREAD_COUNT
#      define UPDATE_ATOMS
#        define VALENCE_KERNEL_THREAD_COUNT 128
#          define VALENCE_BLOCK_MULTIPLICITY 6
#            define KERNEL_NAME kfValenceAtomUpdateNonClashMD
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_BLOCK_MULTIPLICITY
#        undef VALENCE_KERNEL_THREAD_COUNT
#      undef UPDATE_ATOMS
#      define COMPUTE_ENERGY
#        define VALENCE_KERNEL_THREAD_COUNT 96
#          define VALENCE_BLOCK_MULTIPLICITY 8
#            define KERNEL_NAME kfValenceForceEnergyAccumulationNonClashMD
#              include "valence_potential.cui"
#            undef KERNEL_NAME
#          undef VALENCE_BLOCK_MULTIPLICITY
#        undef VALENCE_KERNEL_THREAD_COUNT
#        define UPDATE_ATOMS
#          define VALENCE_KERNEL_THREAD_COUNT 128
#            define VALENCE_BLOCK_MULTIPLICITY 5
#              define KERNEL_NAME kfValenceEnergyAtomUpdateNonClashMD
#                include "valence_potential.cui"
#              undef KERNEL_NAME
#            undef VALENCE_BLOCK_MULTIPLICITY
#          undef VALENCE_KERNEL_THREAD_COUNT
#        undef UPDATE_ATOMS
#      undef COMPUTE_ENERGY
#    undef COMPUTE_FORCE
#    define COMPUTE_ENERGY
#      define VALENCE_KERNEL_THREAD_COUNT 128
#        define VALENCE_BLOCK_MULTIPLICITY 8
#          define KERNEL_NAME kfValenceEnergyAccumulationNonClashMD
#            include "valence_potential.cui"
#          undef KERNEL_NAME
#        undef VALENCE_BLOCK_MULTIPLICITY
#      undef VALENCE_KERNEL_THREAD_COUNT
#    undef COMPUTE_ENERGY
#  undef CLASH_FORGIVENES

// Clear single-precision floating point definitions
#  undef TCALC2
#  undef TCALC3
#  undef TCALC4
#  undef LLCONV_FUNC
#  undef SPLITCONV_FUNC
#  undef SPLIT_TYPE
#  undef SQRT_FUNC
#  undef CBRT_FUNC
#  undef ACOS_FUNC
#  undef COS_FUNC
#  undef SIN_FUNC
#  undef ABS_FUNC
#  undef MIX_FUNC
#  undef TCALC_IS_SINGLE
#undef TCALC

//-------------------------------------------------------------------------------------------------
#ifdef STORMM_USE_CUDA
extern hipFuncAttributes
queryValenceKernelRequirementsMD(const EvaluateForce eval_frc, const EvaluateEnergy eval_nrg,
                                 const AccumulationMethod acc_meth, const VwuGoal purpose,
                                 const ClashResponse collision_handling) {

  
  // The kernel manager will have information about the GPU to use--look at the work units from
  // the perspective of overall occupancy on the GPU.
  hipFuncAttributes result;
  hipError_t cfa = hipErrorInvalidValue;
  switch (collision_handling) {
  case ClashResponse::NONE:
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceEnergyAccumulationMD));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceEnergyAtomUpdateMD));
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceEnergyAccumulationMD));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAtomUpdateMD));
            break;
          }
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceAccumulationMD));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceAtomUpdateMD));
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceAccumulationMD));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceAtomUpdateMD));
            break;
          }
          break;
        }
        break;
      }
      break;
    case EvaluateForce::NO:
      cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAccumulationMD));
      break;
    }
    break;
  case ClashResponse::FORGIVE:
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceEnergyAccumulationNonClashMD));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceEnergyAtomUpdateNonClashMD));
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceEnergyAccumulationNonClashMD));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAtomUpdateNonClashMD));
            break;
          }
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (acc_meth) {
        case AccumulationMethod::SPLIT:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceForceAccumulationNonClashMD));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfsValenceAtomUpdateNonClashMD));
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (purpose) {
          case VwuGoal::ACCUMULATE:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceForceAccumulationNonClashMD));
            break;
          case VwuGoal::MOVE_PARTICLES:
            cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceAtomUpdateNonClashMD));
            break;
          }
          break;
        }
        break;
      }
      break;
    case EvaluateForce::NO:
      cfa = hipFuncGetAttributes(&result, reinterpret_cast<const void*>(kfValenceEnergyAccumulationNonClashMD));
      break;
    }
    break;
  }
  
  // Check for errors
  if (cfa != hipSuccess) {

    // Construct the appropriate error message
    std::string error_message("Error obtaining attributes for kernel k");
    switch (acc_meth) {
    case AccumulationMethod::SPLIT:
      error_message += "fs";
      break;
    case AccumulationMethod::WHOLE:
      error_message += "f";
      break;
    case AccumulationMethod::AUTOMATIC:
      rtErr("Kernels do not accept " + getEnumerationName(acc_meth) + " accumulation.",
            "queryValenceKernelRequirements");
      break;
    }
    error_message += "Valence";
    switch (eval_frc) {
    case EvaluateForce::YES:
      switch (eval_nrg) {
      case EvaluateEnergy::YES:
        switch (purpose) {
        case VwuGoal::ACCUMULATE:
          error_message += "ForceEnergyAccumulation";
          break;
        case VwuGoal::MOVE_PARTICLES:
          error_message += "EnergyAtomUpdate";
          break;
        }
        break;
      case EvaluateEnergy::NO:
        switch (purpose) {
        case VwuGoal::ACCUMULATE:
          error_message += "ForceAccumulation";
          break;
        case VwuGoal::MOVE_PARTICLES:
          error_message += "AtomUpdate";
          break;
        }
        break;
      }
      break;
    case EvaluateForce::NO:
      error_message += "EnergyAccumulation";
      break;
    }
    error_message += "MD.";

    // Report the error
    rtErr(error_message, "queryValenceKernelRequirementsMD");
  }
  
  return result;
}
#endif // STORMM_USE_CUDA

//-------------------------------------------------------------------------------------------------
extern void launchValenceMD(const SyValenceKit<float> &poly_vk,
                            const SyRestraintKit<float, float2, float4> &poly_rk,
                            MMControlKit<float> *ctrl, PsSynthesisWriter *poly_psw,
                            const SyAtomUpdateKit<float, float2, float4> &poly_auk,
                            ThermostatWriter<float> *tstw, ScoreCardWriter *scw,
                            CacheResourceKit<float> *gmem_r, const EvaluateForce eval_force,
                            const EvaluateEnergy eval_energy, const VwuGoal purpose,
                            const AccumulationMethod refined_force_sum, const int2 bt,
                            const float clash_distance, const float clash_ratio) {
  
  // Rather than a switch over cases of the ClashResponse enumerator, just use the nonzero values
  // of either parameter to indicate that clash damping has been requested.
  if (clash_distance >= 1.0e-6 || clash_ratio >= 1.0e-6) {
    switch (purpose) {
    case VwuGoal::ACCUMULATE:
    
      // When the goal is to accumulate energies, forces, or both, the force accumulation method
      // becomes a critical detail when choosing the kernel.
      switch (eval_force) {
      case EvaluateForce::YES:
        switch (refined_force_sum) {
        case AccumulationMethod::SPLIT:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            kfsValenceForceEnergyAccumulationNonClashMD<<<bt.x,
                                                          bt.y>>>(poly_vk, poly_rk, *ctrl,
                                                                  *poly_psw, clash_distance,
                                                                  clash_ratio, *scw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            kfsValenceForceAccumulationNonClashMD<<<bt.x,
                                                    bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                            clash_distance, clash_ratio,
                                                            *gmem_r);
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            kfValenceForceEnergyAccumulationNonClashMD<<<bt.x,
                                                         bt.y>>>(poly_vk, poly_rk, *ctrl,
                                                                 *poly_psw, clash_distance,
                                                                 clash_ratio, *scw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            kfValenceForceAccumulationNonClashMD<<<bt.x,
                                                   bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                           clash_distance, clash_ratio, *gmem_r);
            break;
          }
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        break;
      case EvaluateForce::NO:
        kfValenceEnergyAccumulationNonClashMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                              clash_distance, clash_ratio, *scw,
                                                              *gmem_r);
        break;
      }
      break;
    case VwuGoal::MOVE_PARTICLES:
    
      // When the goal is to move particles, evaluating the force is obligatory, but the manner in
      // which forces are accumulated is still important.  Whether to accumulate energies while
      // evaluating forces and moving the particles remains a consideration in choosing the proper
      // kernel.
      switch (refined_force_sum) {
      case AccumulationMethod::SPLIT:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfsValenceEnergyAtomUpdateNonClashMD<<<bt.x,
                                                 bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                         clash_distance, clash_ratio, poly_auk,
                                                         *tstw, *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfsValenceAtomUpdateNonClashMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                         clash_distance, clash_ratio, poly_auk,
                                                         *tstw, *gmem_r);
          break;
        }
        break;
      case AccumulationMethod::WHOLE:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfValenceEnergyAtomUpdateNonClashMD<<<bt.x,
                                                bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                        clash_distance, clash_ratio, poly_auk,
                                                        *tstw, *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfValenceAtomUpdateNonClashMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                        clash_distance, clash_ratio, poly_auk,
                                                        *tstw,*gmem_r);
          break;
        }
        break;
      case AccumulationMethod::AUTOMATIC:
        break;
      }
      break;
    }
  }
  else {
    switch (purpose) {
    case VwuGoal::ACCUMULATE:

      // See above for the rationale on whether forces or energies are evaluated in each context.
      switch (eval_force) {
      case EvaluateForce::YES:
        switch (refined_force_sum) {
        case AccumulationMethod::SPLIT:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            kfsValenceForceEnergyAccumulationMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl,
                                                                *poly_psw, *scw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            kfsValenceForceAccumulationMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                          *gmem_r);
            break;
          }
          break;
        case AccumulationMethod::WHOLE:
          switch (eval_energy) {
          case EvaluateEnergy::YES:
            kfValenceForceEnergyAccumulationMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl,
                                                               *poly_psw, *scw, *gmem_r);
            break;
          case EvaluateEnergy::NO:
            kfValenceForceAccumulationMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                         *gmem_r);
            break;
          }
          break;
        case AccumulationMethod::AUTOMATIC:
          break;
        }
        break;
      case EvaluateForce::NO:
        kfValenceEnergyAccumulationMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, *scw,
                                                      *gmem_r);
        break;
      }
      break;
    case VwuGoal::MOVE_PARTICLES:
    
      // See above for the rationale on the choice of each kernel.
      switch (refined_force_sum) {
      case AccumulationMethod::SPLIT:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfsValenceEnergyAtomUpdateMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                       poly_auk, *tstw, *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfsValenceAtomUpdateMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk,
                                                 *tstw, *gmem_r);
          break;
        }
        break;
      case AccumulationMethod::WHOLE:
        switch (eval_energy) {
        case EvaluateEnergy::YES:
          kfValenceEnergyAtomUpdateMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw,
                                                      poly_auk, *tstw, *scw, *gmem_r);
          break;
        case EvaluateEnergy::NO:
          kfValenceAtomUpdateMD<<<bt.x, bt.y>>>(poly_vk, poly_rk, *ctrl, *poly_psw, poly_auk,
                                                *tstw, *gmem_r);
          break;
        }
        break;
      case AccumulationMethod::AUTOMATIC:
        break;
      }
      break;
    }
  }
}

} // namespace energy
} // namespace stormm
