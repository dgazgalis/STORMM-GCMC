#include "hip/hip_runtime.h"
// -*-c++-*-
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <nvml.h>
#include <string>
#include <vector>
#include "../../src/Accelerator/hpc_config.cuh"
#include "../../src/Accelerator/kernel_manager.h"
#include "../../src/Constants/fixed_precision.h"
#include "../../src/Constants/scaling.h"
#include "../../src/FileManagement/file_listing.h"
#include "../../src/MolecularMechanics/mm_controls.h"
#include "../../src/Namelists/nml_files.h"
#include "../../src/Parsing/textfile.h"
#include "../../src/Potential/cacheresource.h"
#include "../../src/Potential/hpc_valence_potential.cuh"
#include "../../src/Potential/hpc_nonbonded_potential.cuh"
#include "../../src/Potential/valence_potential.h"
#include "../../src/Synthesis/phasespace_synthesis.h"
#include "../../src/Synthesis/systemcache.h"
#include "../../src/Synthesis/atomgraph_synthesis.h"
#include "../../src/Synthesis/static_mask_synthesis.h"
#include "../../src/Synthesis/synthesis_abstracts.h"
#include "../../src/Synthesis/nonbonded_workunit.h"
#include "../../src/Synthesis/valence_workunit.h"
#include "../../src/Topology/atomgraph.h"
#include "../../src/UnitTesting/unit_test.h"
#include "../../src/UnitTesting/stopwatch.h"


using namespace omni::card;
using namespace omni::constants;
using namespace omni::errors;
using namespace omni::diskutil;
using namespace omni::energy;
using namespace omni::math;
using namespace omni::mm;
using namespace omni::numerics;
using namespace omni::parse;
using namespace omni::restraints;
using namespace omni::synthesis;
using namespace omni::testing;
using namespace omni::topology;
using namespace omni::trajectory;

//-------------------------------------------------------------------------------------------------
// Get a SystemCache object containing all topologies and coordinates in a pair of directories.
//
// Arguments:
//   topol_path:  A series of strings that will be joined into the topology directory name
//   coord_path:  A series of strings that will be joined into the coordinate directory name
//   oe:          Contains critical shell variables such as the $OMNI source path where the
//                named directories are expected to reside
//-------------------------------------------------------------------------------------------------
SystemCache directorySweep(const std::vector<std::string> &topol_path,
                           const std::vector<std::string> &coord_path, const TestEnvironment &oe) {
  
  // Collect coordinates and topologies
  const char osc = osSeparator();
  std::string buffer("&files\n  -p ");
  buffer += oe.getOmniSourcePath() + osc + "benchmark";
  for (size_t i = 0; i < topol_path.size(); i++) {
    buffer += osc + topol_path[i];
  }
  buffer += "\n  -c ";
  buffer += oe.getOmniSourcePath() + osc + "benchmark";
  for (size_t i = 0; i < topol_path.size(); i++) {
    buffer += osc + coord_path[i];
  }
  buffer += "\n&end\n";
  const TextFile tf(buffer, TextOrigin::RAM);
  int start_line = 0;
  FilesControls fcon(tf, &start_line);
  return SystemCache(fcon, ExceptionResponse::SILENT, MapRotatableGroups::NO);
}

//-------------------------------------------------------------------------------------------------
// Replicate a single topology and coordinate system many times, then run kernels to obtain
// timings.
//
// Arguments:
//   ag:       The topology to replicate
//   ps:       The coordinates to replicate
//   nrep:     The number of replicas to make
//   mmctrl:   Step counter and progress counters for all work units
//   gpu:      Details of the GPU to use in calculations
//   timer:    Object to record the timings
//   desc:     Description of the system to run (this will factor into the timings section names)
//-------------------------------------------------------------------------------------------------
void replicaProcessing(AtomGraph *ag, const PhaseSpace &ps, const int nrep,
                       MolecularMechanicsControls *mmctrl, const GpuDetails &gpu,
                       StopWatch *timer, const PrecisionModel prec, const EvaluateForce eval_frc,
                       const EvaluateEnergy eval_nrg, const ForceAccumulationMethod acc_meth,
                       const VwuGoal purpose) {
  std::vector<AtomGraph*> ag_vec(1, ag);
  std::vector<PhaseSpace> ps_vec(1, ps);
  std::vector<int> ag_idx(nrep, 0);
  AtomGraphSynthesis poly_ag(ag_vec, ag_idx, ExceptionResponse::SILENT,
                             maximum_valence_work_unit_atoms);
  PhaseSpaceSynthesis poly_ps(ps_vec, ag_vec, ag_idx);
  StaticExclusionMaskSynthesis poly_se(ag_vec, ag_idx);
  SeMaskSynthesisReader poly_ser = poly_se.data();
  poly_ag.loadNonbondedWorkUnits(poly_se);
  KernelManager launcher(gpu, poly_ag);
  ScoreCard sc(nrep, 1, 32);
  const int2 valence_lp = launcher.getValenceKernelDims(prec, eval_frc, eval_nrg, acc_meth,
                                                        purpose);
  const int2 nonbond_lp = launcher.getNonbondedKernelDims(prec, NbwuKind::TILE_GROUPS, eval_frc,
                                                          eval_nrg, acc_meth);
  CacheResource valence_tb_space(valence_lp.x, maximum_valence_work_unit_atoms);
  CacheResource nonbond_tb_space(nonbond_lp.x, small_block_max_atoms);
  mmctrl->primeWorkUnitCounters(launcher, prec, poly_ag);
  
  // Upload the critical components
  poly_ag.upload();
  poly_se.upload();
  poly_ps.upload();

  // Some common variables for either branch
  const std::string valk_name = valenceKernelKey(prec, eval_frc, eval_nrg, acc_meth, purpose);
  const std::string nnbk_name = nonbondedKernelKey(prec, NbwuKind::TILE_GROUPS, eval_frc,
                                                   eval_nrg, acc_meth);
  const int sys_val_time = timer->addCategory(getBaseName(ag->getFileName()) + " on " +
                                              valk_name + " (" + std::to_string(nrep) + ")");
  const int sys_nb_time = timer->addCategory(getBaseName(ag->getFileName()) + " on " +
                                             nnbk_name + " (" + std::to_string(nrep) + ")");
  
  // Obtain abstracts outside the inner loop, in case this is a significant contributor to the
  // run time.  Forces will only be initialized once, and thereafter calculated repeatedly to test
  // only the run time of the one kernel.
  const HybridTargetLevel devc_tier = HybridTargetLevel::DEVICE;
  switch (prec) {
  case PrecisionModel::DOUBLE:
    {
      const SyValenceKit<double> poly_vk = poly_ag.getDoublePrecisionValenceKit(devc_tier);
      const SyNonbondedKit<double> poly_nbk = poly_ag.getDoublePrecisionNonbondedKit(devc_tier);
      const SyRestraintKit<double,
                           double2,
                           double4> poly_rk = poly_ag.getDoublePrecisionRestraintKit(devc_tier);
      MMControlKit<double> ctrl = mmctrl->dpData(devc_tier);
      PsSynthesisWriter poly_psw = poly_ps.data(devc_tier);
      ScoreCardWriter scw = sc.data(devc_tier);
      CacheResourceKit<double> gmem_rval = valence_tb_space.dpData(devc_tier);
      CacheResourceKit<double> gmem_rnnb = nonbond_tb_space.dpData(devc_tier);
      poly_ps.initializeForces(gpu, devc_tier);
      timer->assignTime(0);

      // Test the valence kernel
      for (int i = 0; i < 1000; i++) {
        ctrl.step += 1;
        launchValenceDp(poly_vk, poly_rk, &ctrl, &poly_psw, &scw, &gmem_rval, eval_frc, eval_nrg,
                        purpose, launcher);
      }
      hipDeviceSynchronize();
      timer->assignTime(sys_val_time);

      // Test the non-bonded kernel
      poly_ps.initializeForces(gpu, devc_tier);
      timer->assignTime(0);
      for (int i = 0; i < 1000; i++) {
        ctrl.step += 1;
        launchNonbondedTileGroupsDp(poly_nbk, poly_ser, &ctrl, &poly_psw, &scw, &gmem_rnnb,
                                    eval_frc, eval_nrg, launcher);
      }
      hipDeviceSynchronize();
      timer->assignTime(sys_nb_time);
    }
    break;
  case PrecisionModel::SINGLE:
    {
      const SyValenceKit<float> poly_vk = poly_ag.getSinglePrecisionValenceKit(devc_tier);
      const SyNonbondedKit<float> poly_nbk = poly_ag.getSinglePrecisionNonbondedKit(devc_tier);
      const SyRestraintKit<float,
                           float2,
                           float4> poly_rk = poly_ag.getSinglePrecisionRestraintKit(devc_tier);
      MMControlKit<float> ctrl = mmctrl->spData(devc_tier);
      PsSynthesisWriter poly_psw = poly_ps.data(devc_tier);
      ScoreCardWriter scw = sc.data(devc_tier);
      CacheResourceKit<float> gmem_rval = valence_tb_space.spData(devc_tier);
      CacheResourceKit<float> gmem_rnnb = nonbond_tb_space.spData(devc_tier);
      poly_ps.initializeForces(gpu, devc_tier);      
      timer->assignTime(0);

      // Test the valence kernel
      for (int i = 0; i < 1000; i++) {
        ctrl.step += 1;
        launchValenceSp(poly_vk, poly_rk, &ctrl, &poly_psw, &scw, &gmem_rval, eval_frc, eval_nrg,
                        purpose, acc_meth, launcher);
      }
      hipDeviceSynchronize();
      timer->assignTime(sys_val_time);

      // Test the non-bonded kernel
      poly_ps.initializeForces(gpu, devc_tier);      
      timer->assignTime(0);
      for (int i = 0; i < 1000; i++) {
        ctrl.step += 1;
        launchNonbondedTileGroupsSp(poly_nbk, poly_ser, &ctrl, &poly_psw, &scw, &gmem_rnnb,
                                    eval_frc, eval_nrg, acc_meth, launcher);
      }
      hipDeviceSynchronize();
      timer->assignTime(sys_nb_time);
    }
    break;
  }
}

//-------------------------------------------------------------------------------------------------
// main
//-------------------------------------------------------------------------------------------------
int main(const int argc, const char* argv[]) {

  // Some baseline initialization
  TestEnvironment oe(argc, argv);
  StopWatch timer;
  HpcConfig gpu_config(ExceptionResponse::WARN);
  std::vector<int> my_gpus = gpu_config.getGpuDevice(1);
  GpuDetails gpu = gpu_config.getGpuInfo(my_gpus[0]);

  // Configure the relevant kernels for this executable.
  valenceKernelSetup();

  // Read dipeptides
  const std::vector<std::string> dipeptide_topols = { "Topologies", "Dipeptides",
                                                      ".*_ff1.*SB.top" };
  const std::vector<std::string> dipeptide_coords = { "Coordinates", "Dipeptides",
                                                      ".*_ff1.*SB.inpcrd"};
  SystemCache dipeptide_sc = directorySweep(dipeptide_topols, dipeptide_coords, oe);
  MolecularMechanicsControls mmctrl;
  
  // Loop over the dipeptides one at a time, make syntheses of each of them individually, and
  // test kernels.
  const int ndipeptides = dipeptide_sc.getSystemCount();
  const std::vector<int> batch_multiplier = { 1, 3, 5, 10, 20, 40 };
  for (int i = 0; i < ndipeptides; i++) {
    AtomGraph *iag_ptr = dipeptide_sc.getSystemTopologyPointer(i);
    for (int j = 0; j < 6; j++) {
      const int ncopy = gpu.getSMPCount() * batch_multiplier[j];
      replicaProcessing(iag_ptr, dipeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                        &timer, PrecisionModel::SINGLE, EvaluateForce::YES, EvaluateEnergy::NO,
                        ForceAccumulationMethod::SPLIT, VwuGoal::ACCUMULATE);
      replicaProcessing(iag_ptr, dipeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                        &timer, PrecisionModel::SINGLE, EvaluateForce::YES, EvaluateEnergy::YES,
                        ForceAccumulationMethod::SPLIT, VwuGoal::ACCUMULATE);
      replicaProcessing(iag_ptr, dipeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                        &timer, PrecisionModel::SINGLE, EvaluateForce::YES, EvaluateEnergy::NO,
                        ForceAccumulationMethod::WHOLE, VwuGoal::ACCUMULATE);
      replicaProcessing(iag_ptr, dipeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                        &timer, PrecisionModel::SINGLE, EvaluateForce::YES, EvaluateEnergy::YES,
                        ForceAccumulationMethod::WHOLE, VwuGoal::ACCUMULATE);
      
      // Only do double-precision calculations for low replica numbers--this can be strenuous on
      // many architectures, particularly in the non-bonded kernel.
      if (ncopy < 10) {
        replicaProcessing(iag_ptr, dipeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                          &timer, PrecisionModel::DOUBLE, EvaluateForce::YES, EvaluateEnergy::NO,
                          ForceAccumulationMethod::SPLIT, VwuGoal::ACCUMULATE);
        replicaProcessing(iag_ptr, dipeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                          &timer, PrecisionModel::DOUBLE, EvaluateForce::YES, EvaluateEnergy::YES,
                          ForceAccumulationMethod::SPLIT, VwuGoal::ACCUMULATE);
      }
    }
  }

  // Read dipeptides
  const std::vector<std::string> tripeptide_topols = { "Topologies", "Tripeptides",
                                                       ".*_ff1.*SB.top" };
  const std::vector<std::string> tripeptide_coords = { "Coordinates", "Tripeptides",
                                                       ".*_ff1.*SB.inpcrd"};
  SystemCache tripeptide_sc = directorySweep(tripeptide_topols, tripeptide_coords, oe);
  
  // Loop over the tripeptides one at a time, make syntheses of each of them individually, and
  // test kernels.
  const int ntripeptides = tripeptide_sc.getSystemCount();
  for (int i = 0; i < ntripeptides; i++) {
    AtomGraph *iag_ptr = tripeptide_sc.getSystemTopologyPointer(i);
    for (int j = 0; j < 6; j++) {
      const int ncopy = gpu.getSMPCount() * batch_multiplier[j];
      replicaProcessing(iag_ptr, tripeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                        &timer, PrecisionModel::SINGLE, EvaluateForce::YES, EvaluateEnergy::NO,
                        ForceAccumulationMethod::SPLIT, VwuGoal::ACCUMULATE);
      replicaProcessing(iag_ptr, tripeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                        &timer, PrecisionModel::SINGLE, EvaluateForce::YES, EvaluateEnergy::YES,
                        ForceAccumulationMethod::SPLIT, VwuGoal::ACCUMULATE);
      replicaProcessing(iag_ptr, tripeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                        &timer, PrecisionModel::SINGLE, EvaluateForce::YES, EvaluateEnergy::NO,
                        ForceAccumulationMethod::WHOLE, VwuGoal::ACCUMULATE);
      replicaProcessing(iag_ptr, tripeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                        &timer, PrecisionModel::SINGLE, EvaluateForce::YES, EvaluateEnergy::YES,
                        ForceAccumulationMethod::WHOLE, VwuGoal::ACCUMULATE);
      if (ncopy < 10) {
        replicaProcessing(iag_ptr, tripeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                          &timer, PrecisionModel::DOUBLE, EvaluateForce::YES, EvaluateEnergy::NO,
                          ForceAccumulationMethod::SPLIT, VwuGoal::ACCUMULATE);
        replicaProcessing(iag_ptr, tripeptide_sc.getCoordinateReference(i), ncopy, &mmctrl, gpu,
                          &timer, PrecisionModel::DOUBLE, EvaluateForce::YES, EvaluateEnergy::YES,
                          ForceAccumulationMethod::SPLIT, VwuGoal::ACCUMULATE);
      }
    }
  }

  // Summary evaluation
  if (oe.getDisplayTimingsOrder()) {
    timer.assignTime(0);
    timer.printResults();
  }
  printTestSummary(oe.getVerbosity());
}
