// -*-c++-*-
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <nvml.h>
#include "../../src/Accelerator/hpc_config.cuh"
#include "../../src/Constants/fixed_precision.h"
#include "../../src/Constants/scaling.h"
#include "../../src/FileManagement/file_listing.h"
#include "../../src/Math/rounding.h"
#include "../../src/MolecularMechanics/mm_evaluation.h"
#include "../../src/Namelists/nml_files.h"
#include "../../src/Parsing/textfile.h"
#include "../../src/Potential/cacheresource.h"
#include "../../src/Potential/hpc_valence_potential.cuh"
#include "../../src/Potential/valence_potential.h"
#include "../../src/Reporting/error_format.h"
#include "../../src/Restraints/restraint_apparatus.h"
#include "../../src/Synthesis/phasespace_synthesis.h"
#include "../../src/Synthesis/systemcache.h"
#include "../../src/Synthesis/atomgraph_synthesis.h"
#include "../../src/Synthesis/synthesis_abstracts.h"
#include "../../src/Synthesis/valence_workunit.h"
#include "../../src/Topology/atomgraph.h"
#include "../../src/Topology/atomgraph_abstracts.h"
#include "../../src/Trajectory/phasespace.h"
#include "../../src/UnitTesting/unit_test.h"
#include "../../src/UnitTesting/stopwatch.h"
#include "assemble_restraints.h"

using namespace omni::card;
using namespace omni::constants;
using namespace omni::errors;
using namespace omni::diskutil;
using namespace omni::energy;
using namespace omni::math;
using namespace omni::mm;
using namespace omni::numerics;
using namespace omni::parse;
using namespace omni::restraints;
using namespace omni::synthesis;
using namespace omni::testing;
using namespace omni::topology;
using namespace omni::trajectory;

//-------------------------------------------------------------------------------------------------
// Compute forces due to valence interactions acting on a series of systems using topology and
// coordinate compilations.  Check the results against the accumulations for individual systems.
// This function assumes that the topology and coordinate compilations have already been uploaded
// to the device.
//
// Arguments:
//   poly_ps:        Coordinates for many systems
//   mmctrl:         Molecular mechanics progress counters
//   tb_space:       Thread block resources, pre-allocated on the GPU
//   poly_ag:        Topologies for many systems
//   facc_method:    Force accumulation method
//   prec:           Precision level at which to perform the calculations (may not be compatible
//                   with all force accumulation methods)
//   gpu:            Details of the GPU to use
//   mue_tol:        Tolerance for mean unsigned error in forces 
//   max_error_tol:  Tolerance for maximum unsigned error in forces 
//-------------------------------------------------------------------------------------------------
void checkCompilationForces(PhaseSpaceSynthesis *poly_ps, MolecularMechanicsControls *mmctrl,
                            CacheResource *tb_space, const AtomGraphSynthesis &poly_ag,
                            const ForceAccumulationMethod facc_method, const PrecisionLevel prec,
                            const GpuDetails &gpu, const double mue_tol,
                            const double max_error_tol, const TestPriority do_tests) {
  const int nsys = poly_ps->getSystemCount();
  std::vector<double> frc_mues(nsys);
  const std::vector<double> frc_mue_tolerance(nsys, mue_tol);
  std::vector<double> frc_max_errors(nsys);
  const std::vector<double> frc_max_error_tolerance(nsys, max_error_tol);
  ScoreCard sc(nsys, 1, 32);
  poly_ps->initializeForces(gpu, HybridTargetLevel::DEVICE);
  mmctrl->incrementStep();
  switch (prec) {
  case PrecisionLevel::SINGLE:
  case PrecisionLevel::SINGLE_PLUS:
    launchValenceSp(poly_ag, mmctrl, poly_ps, &sc, tb_space, EvaluateForce::YES,
                    EvaluateEnergy::NO, VwuGoal::ACCUMULATE, facc_method, gpu);
    break;
  case PrecisionLevel::DOUBLE:
    launchValenceDp(poly_ag, mmctrl, poly_ps, &sc, tb_space, EvaluateForce::YES,
                    EvaluateEnergy::NO, VwuGoal::ACCUMULATE, gpu);    
    break;
  }
  for (int i = 0; i < nsys; i++) {
    PhaseSpace host_result = poly_ps->exportSystem(i, HybridTargetLevel::HOST);
    PhaseSpace devc_result = poly_ps->exportSystem(i, HybridTargetLevel::DEVICE);
    host_result.initializeForces();
    ScoreCard isc(1, 1, 32);
    evalValeMM(&host_result, &isc, poly_ag.getSystemTopologyPointer(i), EvaluateForce::YES, 0);
    const TrajectoryKind frcid = TrajectoryKind::FORCES;
    const std::vector<double> devc_frc = devc_result.getInterlacedCoordinates(frcid);
    const std::vector<double> host_frc = host_result.getInterlacedCoordinates(frcid);
    frc_mues[i] = meanUnsignedError(devc_frc, host_frc);
    frc_max_errors[i] = maxAbsoluteDifference(devc_frc, host_frc);
  }
  check(frc_mues, RelationalOperator::LESS_THAN, frc_mue_tolerance, "Forces obtained by the "
        "valence interaction kernel, operating on systems without external restraints, exceed the "
        "tolerance for mean unsigned errors in their vector components.  Force accumulation "
        "method: " + getForceAccumulationMethodName(facc_method) + ".  Precision level in the "
        "calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
  check(frc_max_errors, RelationalOperator::LESS_THAN, frc_max_error_tolerance, "Forces obtained "
        "by the valence interaction kernel, operating on systems without external restraints, "
        "exceed the maximum allowed errors for forces acting on any one particle.  Force "
        "accumulation method: " + getForceAccumulationMethodName(facc_method) + ".  Precision "
        "level in the calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
}

//-------------------------------------------------------------------------------------------------
// Compute energies of a series of systems, using topology and coordinate compilations on the GPU,
// due to valence interactions.  Check the results against the accumulations for individual
// systems.  This function assumes that the topology and coordinate compilations have already been
// uploaded to the device.
//
// Arguments:
//-------------------------------------------------------------------------------------------------
void checkCompilationEnergies(PhaseSpaceSynthesis *poly_ps, MolecularMechanicsControls *mmctrl,
                              CacheResource *tb_space, const AtomGraphSynthesis &poly_ag,
                              const PrecisionLevel prec, const GpuDetails &gpu,
                              const double bond_tol, const double angl_tol, const double dihe_tol,
                              const double impr_tol, const double ubrd_tol, const double cimp_tol,
                              const double cmap_tol, const double lj14_tol, const double qq14_tol,
                              const double rstr_tol, const TestPriority do_tests) {
  const int nsys = poly_ps->getSystemCount();
  ScoreCard sc(nsys, 1, 32);
  poly_ps->initializeForces(gpu, HybridTargetLevel::DEVICE);
  mmctrl->incrementStep();
  switch (prec) {
  case PrecisionLevel::SINGLE:
  case PrecisionLevel::SINGLE_PLUS:
    launchValenceSp(poly_ag, mmctrl, poly_ps, &sc, tb_space, EvaluateForce::NO,
                    EvaluateEnergy::YES, VwuGoal::ACCUMULATE, ForceAccumulationMethod::SPLIT, gpu);
    break;
  case PrecisionLevel::DOUBLE:
    launchValenceDp(poly_ag, mmctrl, poly_ps, &sc, tb_space, EvaluateForce::NO,
                    EvaluateEnergy::YES, VwuGoal::ACCUMULATE, gpu);
    break;
  }
  sc.download();
  std::vector<double> cpu_bond(nsys), gpu_bond(nsys), cpu_angl(nsys), gpu_angl(nsys);
  std::vector<double> cpu_dihe(nsys), gpu_dihe(nsys), cpu_impr(nsys), gpu_impr(nsys);
  std::vector<double> cpu_ubrd(nsys), gpu_ubrd(nsys), cpu_cimp(nsys), gpu_cimp(nsys);
  std::vector<double> cpu_cmap(nsys), gpu_cmap(nsys), cpu_qq14(nsys), gpu_qq14(nsys);
  std::vector<double> cpu_lj14(nsys), gpu_lj14(nsys);
  for (int i = 0; i < nsys; i++) {
    PhaseSpace devc_result = poly_ps->exportSystem(i, HybridTargetLevel::DEVICE);
    PhaseSpace host_result = poly_ps->exportSystem(i, HybridTargetLevel::HOST);
    host_result.initializeForces();
    ScoreCard isc(1, 1, 32);
    evalValeMM(&host_result, &isc, poly_ag.getSystemTopologyPointer(i), EvaluateForce::NO, 0);
    gpu_bond[i] =  sc.reportInstantaneousStates(StateVariable::BOND, i);
    cpu_bond[i] = isc.reportInstantaneousStates(StateVariable::BOND, 0);
    gpu_angl[i] =  sc.reportInstantaneousStates(StateVariable::ANGLE, i);
    cpu_angl[i] = isc.reportInstantaneousStates(StateVariable::ANGLE, 0);
    gpu_dihe[i] =  sc.reportInstantaneousStates(StateVariable::PROPER_DIHEDRAL, i);
    cpu_dihe[i] = isc.reportInstantaneousStates(StateVariable::PROPER_DIHEDRAL, 0);
    gpu_impr[i] =  sc.reportInstantaneousStates(StateVariable::IMPROPER_DIHEDRAL, i);
    cpu_impr[i] = isc.reportInstantaneousStates(StateVariable::IMPROPER_DIHEDRAL, 0);
    gpu_ubrd[i] =  sc.reportInstantaneousStates(StateVariable::UREY_BRADLEY, i);
    cpu_ubrd[i] = isc.reportInstantaneousStates(StateVariable::UREY_BRADLEY, 0);
    gpu_cimp[i] =  sc.reportInstantaneousStates(StateVariable::CHARMM_IMPROPER, i);
    cpu_cimp[i] = isc.reportInstantaneousStates(StateVariable::CHARMM_IMPROPER, 0);
    gpu_cmap[i] =  sc.reportInstantaneousStates(StateVariable::CMAP, i);
    cpu_cmap[i] = isc.reportInstantaneousStates(StateVariable::CMAP, 0);
    gpu_qq14[i] =  sc.reportInstantaneousStates(StateVariable::ELECTROSTATIC_ONE_FOUR, i);
    cpu_qq14[i] = isc.reportInstantaneousStates(StateVariable::ELECTROSTATIC_ONE_FOUR, 0);
    gpu_lj14[i] =  sc.reportInstantaneousStates(StateVariable::VDW_ONE_FOUR, i);
    cpu_lj14[i] = isc.reportInstantaneousStates(StateVariable::VDW_ONE_FOUR, 0);
  }
  check(gpu_bond, RelationalOperator::EQUAL, Approx(cpu_bond).margin(bond_tol), "Bond energies "
        "computed on the CPU and GPU do not agree.  Precision level in the calculation: " +
        getPrecisionLevelName(prec) + ".", do_tests);
  check(gpu_angl, RelationalOperator::EQUAL, Approx(cpu_angl).margin(angl_tol), "Angle energies "
        "computed on the CPU and GPU do not agree.  Precision level in the calculation: " +
        getPrecisionLevelName(prec) + ".", do_tests);
  check(gpu_dihe, RelationalOperator::EQUAL, Approx(cpu_dihe).margin(dihe_tol), "Proper "
        "dihedral energies computed on the CPU and GPU do not agree.  Precision level in the "
        "calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
  check(gpu_impr, RelationalOperator::EQUAL, Approx(cpu_impr).margin(impr_tol), "Improper "
        "dihedral energies computed on the CPU and GPU do not agree.  Precision level in the "
        "calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
  check(gpu_ubrd, RelationalOperator::EQUAL, Approx(cpu_ubrd).margin(ubrd_tol), "Urey-Bradley "
        "energies computed on the CPU and GPU do not agree.  Precision level in the "
        "calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
  check(gpu_cimp, RelationalOperator::EQUAL, Approx(cpu_cimp).margin(cimp_tol), "CHARMM "
        "improper dihedral energies computed on the CPU and GPU do not agree.  Precision level in "
        "the calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
  check(gpu_cmap, RelationalOperator::EQUAL, Approx(cpu_cmap).margin(cmap_tol), "CMAP "
        "energies computed on the CPU and GPU do not agree.  Precision level in the "
        "calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
  check(gpu_qq14, RelationalOperator::EQUAL, Approx(cpu_qq14).margin(qq14_tol), "Electrostatic "
        "1:4 energies computed on the CPU and GPU do not agree.  Precision level in the "
        "calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
  check(gpu_lj14, RelationalOperator::EQUAL, Approx(cpu_lj14).margin(lj14_tol), "Lennard-Jones "
        "1:4 energies computed on the CPU and GPU do not agree.  Precision level in the "
        "calculation: " + getPrecisionLevelName(prec) + ".", do_tests);
}

//-------------------------------------------------------------------------------------------------
// main
//-------------------------------------------------------------------------------------------------
int main(const int argc, const char* argv[]) {

  // Some baseline initialization
  TestEnvironment oe(argc, argv);
  StopWatch timer;
  
  // Section 1
  section("Coordinate compilation and staging");
  
  // Section 2
  section("Topology compilation and staging");

  // Get the GPU specs.  Set of parameters for the work units and launch grids.
  HpcConfig gpu_config(ExceptionResponse::WARN);
  std::vector<int> my_gpus = gpu_config.getGpuDevice(1);
  GpuDetails gpu = gpu_config.getGpuInfo(my_gpus[0]);
  int nblocks = gpu.getSMPCount();
  int nthreads = gpu.getMaxThreadsPerBlock();
  int max_vwu_atoms = maximum_valence_work_unit_atoms;
  if (gpu.getArchMajor() == 6 && gpu.getArchMinor() == 1) {
    nblocks *= 2;
    nthreads /= 2;
    max_vwu_atoms /= 2;
  }

  // Configure the relevant kernels for this executable.
  valenceKernelSetup();
  
  // Collect coordinates and topologies
  const char osc = osSeparator();
  std::string buffer("&files\n  -p ");
  buffer += oe.getOmniSourcePath() + osc + "test" + osc + "Namelists" + osc + "topol" + osc +
            ".*.top\n  -c ";
  buffer += oe.getOmniSourcePath() + osc + "test" + osc + "Namelists" + osc + "coord" + osc +
            ".*.inpcrd\n&end\n";
  const TextFile tf(buffer, TextOrigin::RAM);
  int start_line = 0;
  FilesControls fcon(tf, &start_line);
  const SystemCache sysc(fcon, ExceptionResponse::SILENT, MapRotatableGroups::NO, &timer);

  // Form the syntheses of topologies and coordinates
  section(1);
  const int nsys = sysc.getSystemCount();
  const TestPriority do_tests = (nsys > 0) ? TestPriority::CRITICAL : TestPriority::ABORT;
  if (nsys == 0) {
    rtWarn("No system topology and coordinate pairs were detected.  Subsequent tests will be "
           "skipped.", "test_hpc_synthesis");
  }
  std::vector<int> topology_indices(nsys, 0);
  for (int i = 0; i < nsys; i++) {
    topology_indices[i] = i;
  }
  AtomGraphSynthesis poly_ag(sysc.getSystemTopologyPointerCC(), topology_indices,
                             ExceptionResponse::WARN, max_vwu_atoms, &timer);
  PhaseSpaceSynthesis poly_ps(sysc);
  check(poly_ag.getSystemCount(), RelationalOperator::EQUAL, poly_ps.getSystemCount(),
        "PhaseSpaceSynthesis and AtomGraphSynthesis objects formed from the same SystemCache have "
        "different numbers of systems inside of them.", do_tests);

  // Upload the compiled systems and check the results
  PsSynthesisWriter psw = poly_ps.data();
  poly_ag.upload();
  poly_ps.upload();
  std::vector<double> gpu_charges = poly_ag.getPartialCharges<double>(HybridTargetLevel::DEVICE);
  int padded_atom_count = 0;
  for (int i = 0; i < nsys; i++) {
    if (i == nsys - 1) {
      padded_atom_count += poly_ag.getSystemTopologyPointer(i)->getAtomCount();
    }
    else {
      padded_atom_count += roundUp(poly_ag.getSystemTopologyPointer(i)->getAtomCount(),
                                   warp_size_int);
    }
  }
  std::vector<double> rbt_charges(padded_atom_count, 0.0);
  int atom_offset = 0;
  for (int i = 0; i < nsys; i++) {
    const AtomGraph *iag_ptr = poly_ag.getSystemTopologyPointer(i);
    std::vector<double> ichg = iag_ptr->getPartialCharge<double>();
    const int natom = iag_ptr->getAtomCount();
    for (int j = 0; j < natom; j++) {
      rbt_charges[atom_offset + j] = ichg[j];
    }
    const int padded_natom = roundUp(natom, warp_size_int);
    if (i < nsys - 1) {
      for (int j = natom; j < padded_natom; j++) {
        gpu_charges[atom_offset + j] = 0.0;
      }
    }
    atom_offset += padded_natom;
  }
  check(gpu_charges, RelationalOperator::EQUAL, rbt_charges, "Charges pulled from the GPU in an "
        "AtomGraphSynthesis object do not meet expectations.", do_tests);

  // Allocate resources for various kernels
  CacheResource tb_space(nblocks, max_vwu_atoms);
  MolecularMechanicsControls mmctrl;
  mmctrl.primeWorkUnitCounters(gpu, poly_ag);
  ScoreCard sc(nsys, 1, 32);

  // Launch the valence evaluation kernel for small systems with only bonds, angles, dihedrals,
  // and 1:4 attenuated interactions.
  checkCompilationForces(&poly_ps, &mmctrl, &tb_space, poly_ag, ForceAccumulationMethod::WHOLE,
                         PrecisionLevel::DOUBLE, gpu, 3.5e-6, 2.0e-5, do_tests);
  checkCompilationForces(&poly_ps, &mmctrl, &tb_space, poly_ag, ForceAccumulationMethod::SPLIT,
                         PrecisionLevel::SINGLE, gpu, 3.5e-5, 2.0e-4, do_tests);
  checkCompilationForces(&poly_ps, &mmctrl, &tb_space, poly_ag, ForceAccumulationMethod::WHOLE,
                         PrecisionLevel::SINGLE, gpu, 3.5e-5, 2.0e-4, do_tests);
  checkCompilationEnergies(&poly_ps, &mmctrl, &tb_space, poly_ag, PrecisionLevel::DOUBLE,
                           gpu, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6,
                           1.0e-6, 1.0e-6, do_tests);
  checkCompilationEnergies(&poly_ps, &mmctrl, &tb_space, poly_ag, PrecisionLevel::SINGLE,
                           gpu, 1.5e-5, 1.5e-5, 5.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 6.0e-6,
                           2.2e-5, 1.0e-6, do_tests);
  
  // Create a set of larger systems, now involving CMAPs and other CHARMM force field terms
  const std::string topology_base = oe.getOmniSourcePath() + osc + "test" + osc + "Topology";
  const std::string trpi_top_name = topology_base + osc + "trpcage.top";
  const std::string dhfr_top_name = topology_base + osc + "dhfr_cmap.top";
  const std::string alad_top_name = topology_base + osc + "ala_dipeptide.top";
  const std::string coordinate_base = oe.getOmniSourcePath() + osc + "test" + osc + "Trajectory";
  const std::string trpi_crd_name = coordinate_base + osc + "trpcage.inpcrd";
  const std::string dhfr_crd_name = coordinate_base + osc + "dhfr_cmap.inpcrd";
  const std::string alad_crd_name = coordinate_base + osc + "ala_dipeptide.inpcrd";  
  const bool files_exist = (getDrivePathType(trpi_top_name) == DrivePathType::FILE &&
                            getDrivePathType(dhfr_top_name) == DrivePathType::FILE &&
                            getDrivePathType(alad_top_name) == DrivePathType::FILE &&
                            getDrivePathType(trpi_crd_name) == DrivePathType::FILE &&
                            getDrivePathType(dhfr_crd_name) == DrivePathType::FILE &&
                            getDrivePathType(alad_crd_name) == DrivePathType::FILE);
  AtomGraph trpi_ag, dhfr_ag, alad_ag;
  PhaseSpace trpi_ps, dhfr_ps, alad_ps;
  if (files_exist) {
    trpi_ag.buildFromPrmtop(trpi_top_name, ExceptionResponse::SILENT);
    dhfr_ag.buildFromPrmtop(dhfr_top_name, ExceptionResponse::SILENT);
    alad_ag.buildFromPrmtop(alad_top_name, ExceptionResponse::SILENT);
    trpi_ps.buildFromFile(trpi_crd_name);
    dhfr_ps.buildFromFile(dhfr_crd_name);
    alad_ps.buildFromFile(alad_crd_name);
  }
  else {
    rtWarn("Files for several systems in implicit solvent were not found.  Check the "
           "${OMNI_SOURCE} environment variable for validity.  Subsequent tests will be skipped.");
  }

  // Read some larger topologies, with CHARMM CMAP and other force field terms
  const std::vector<AtomGraph*> bigger_tops = { &trpi_ag, &dhfr_ag, &alad_ag };
  const std::vector<PhaseSpace> bigger_crds = { trpi_ps, dhfr_ps, alad_ps };
  PhaseSpaceSynthesis big_poly_ps(bigger_crds, bigger_tops);
  const std::vector<int> big_top_indices = { 0, 1, 2 };
  AtomGraphSynthesis big_poly_ag(bigger_tops, big_top_indices, ExceptionResponse::SILENT,
                                 max_vwu_atoms, &timer);
  big_poly_ag.upload();
  big_poly_ps.upload();
  timer.assignTime(0);
  checkCompilationForces(&big_poly_ps, &mmctrl, &tb_space, big_poly_ag,
                         ForceAccumulationMethod::WHOLE, PrecisionLevel::DOUBLE, gpu, 3.5e-6,
                         2.0e-5, do_tests);
  checkCompilationForces(&big_poly_ps, &mmctrl, &tb_space, big_poly_ag,
                         ForceAccumulationMethod::SPLIT, PrecisionLevel::SINGLE, gpu, 7.5e-5,
                         3.0e-3, do_tests);
  checkCompilationForces(&big_poly_ps, &mmctrl, &tb_space, big_poly_ag,
                         ForceAccumulationMethod::WHOLE, PrecisionLevel::SINGLE, gpu, 7.5e-5,
                         3.0e-3, do_tests);
  checkCompilationEnergies(&big_poly_ps, &mmctrl, &tb_space, big_poly_ag, PrecisionLevel::DOUBLE,
                           gpu, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 6.0e-6, 1.0e-6,
                           1.0e-6, 1.0e-6, do_tests);
  checkCompilationEnergies(&big_poly_ps, &mmctrl, &tb_space, big_poly_ag, PrecisionLevel::SINGLE,
                           gpu, 1.5e-4, 2.2e-5, 9.0e-5, 1.5e-5, 6.0e-5, 3.0e-5, 6.0e-6, 7.5e-5,
                           2.2e-4, 1.0e-6, do_tests);  

  // Read some topologies with virtual sites.  First, test the forces that appear to act on the
  // virtual sites.  Add restraints to these ligands.
  const std::string brbz_top_name = topology_base + osc + "bromobenzene_iso.top";
  const std::string lig1_top_name = topology_base + osc + "stereo_L1.top";
  const std::string lig2_top_name = topology_base + osc + "symmetry_L1.top";
  const std::string brbz_crd_name = coordinate_base + osc + "bromobenzene_iso.inpcrd";
  const std::string lig1_crd_name = coordinate_base + osc + "stereo_L1.inpcrd";
  const std::string lig2_crd_name = coordinate_base + osc + "symmetry_L1.inpcrd";
  const bool ligands_exist = (getDrivePathType(brbz_top_name) == DrivePathType::FILE &&
                              getDrivePathType(lig1_top_name) == DrivePathType::FILE &&
                              getDrivePathType(lig2_top_name) == DrivePathType::FILE &&
                              getDrivePathType(brbz_crd_name) == DrivePathType::FILE &&
                              getDrivePathType(lig1_crd_name) == DrivePathType::FILE &&
                              getDrivePathType(lig2_crd_name) == DrivePathType::FILE);
  AtomGraph brbz_ag, lig1_ag, lig2_ag;
  PhaseSpace brbz_ps, lig1_ps, lig2_ps;
  if (ligands_exist) {
    brbz_ag.buildFromPrmtop(brbz_top_name);
    lig1_ag.buildFromPrmtop(lig1_top_name);
    lig2_ag.buildFromPrmtop(lig2_top_name);
    brbz_ps.buildFromFile(brbz_crd_name);
    lig1_ps.buildFromFile(lig1_crd_name);
    lig2_ps.buildFromFile(lig2_crd_name);
  }
  RestraintApparatus brbz_ra = assembleRestraints(&brbz_ag, brbz_ps);
  RestraintApparatus lig1_ra = assembleRestraints(&lig1_ag, lig1_ps);
  RestraintApparatus lig2_ra = assembleRestraints(&lig2_ag, lig2_ps);
  const std::vector<AtomGraph*> ligand_ag_list = { &brbz_ag, &lig1_ag, &lig2_ag };
  const std::vector<PhaseSpace> ligand_ps_list = {  brbz_ps,  lig1_ps,  lig2_ps };
  const std::vector<RestraintApparatus*> ligand_ra_list = { &brbz_ra, &lig1_ra, &lig2_ra };
  const std::vector<int> ligand_tiling = { 0, 1, 2, 0, 0, 0, 1, 1, 1, 2, 2, 2, 2, 1, 0 };
  PhaseSpaceSynthesis ligand_poly_ps(ligand_ps_list, ligand_ag_list, ligand_tiling);
  AtomGraphSynthesis ligand_poly_ag(ligand_ag_list, //ligand_ra_list, ligand_tiling,
                                    ligand_tiling,
                                    ExceptionResponse::WARN, max_vwu_atoms, &timer);
  ligand_poly_ag.upload();
  ligand_poly_ps.upload();
  timer.assignTime(0);
  checkCompilationForces(&ligand_poly_ps, &mmctrl, &tb_space, ligand_poly_ag,
                         ForceAccumulationMethod::WHOLE, PrecisionLevel::DOUBLE, gpu, 3.5e-6,
                         2.0e-5, do_tests);
  checkCompilationForces(&ligand_poly_ps, &mmctrl, &tb_space, ligand_poly_ag,
                         ForceAccumulationMethod::SPLIT, PrecisionLevel::SINGLE, gpu, 7.5e-5,
                         3.0e-3, do_tests);
  checkCompilationForces(&ligand_poly_ps, &mmctrl, &tb_space, ligand_poly_ag,
                         ForceAccumulationMethod::WHOLE, PrecisionLevel::SINGLE, gpu, 7.5e-5,
                         3.0e-3, do_tests);
  checkCompilationEnergies(&ligand_poly_ps, &mmctrl, &tb_space, ligand_poly_ag,
                           PrecisionLevel::DOUBLE, gpu, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, 1.0e-6,
                           1.0e-6, 6.0e-6, 1.0e-6, 1.0e-6, 1.0e-6, do_tests);
  checkCompilationEnergies(&ligand_poly_ps, &mmctrl, &tb_space, ligand_poly_ag,
                           PrecisionLevel::SINGLE, gpu, 1.5e-4, 2.2e-5, 9.0e-5, 1.5e-5, 6.0e-5,
                           3.0e-5, 6.0e-6, 7.5e-5, 2.2e-4, 1.0e-6, do_tests);
  
#if 0
  for (int len = 4; len < 36; len += 4) {
    const std::vector<AtomGraph*> bigger_tops(len, &trpi_ag);
    const std::vector<PhaseSpace> bigger_crds(len, trpi_ps);
    std::vector<int> trpi_indices(len);
    for (int i = 0; i < len; i++) {
      trpi_indices[i] = i;
    }
    PhaseSpaceSynthesis big_poly_ps(bigger_crds, bigger_tops);
    AtomGraphSynthesis big_poly_ag(bigger_tops, trpi_indices, ExceptionResponse::SILENT,
                                   max_vwu_atoms, &timer);
    big_poly_ag.upload();
    big_poly_ps.upload();
    timer.assignTime(0);
    const int i_timings = timer.addCategory("GPU VWU evaluation " + std::to_string(len));
    const int cpu_timings = timer.addCategory("CPU VWU evaluation " + std::to_string(len));
    for (int i = 0; i < 10; i++) {
      for (int j = 0; j < 1000; j++) {
        mmctrl.incrementStep();
        launchValenceSp(big_poly_ag, &mmctrl, &big_poly_ps, &sc, &tb_space, EvaluateForce::YES,
                        EvaluateEnergy::YES, VwuGoal::ACCUMULATE, ForceAccumulationMethod::SPLIT,
                        gpu);
      }
      hipDeviceSynchronize();
      timer.assignTime(i_timings);
    }
  }
#endif

  // Summary evaluation
  if (oe.getDisplayTimingsOrder()) {
    timer.assignTime(0);
    timer.printResults();
  }
  printTestSummary(oe.getVerbosity());

  return 0;
}
