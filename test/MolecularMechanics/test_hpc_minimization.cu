#include "hip/hip_runtime.h"
// -*-c++-*-
#include "../../src/Accelerator/hybrid.h"
#include "../../src/Accelerator/gpu_details.h"
#include "../../src/Accelerator/kernel_manager.h"
#include "../../src/Accelerator/hpc_config.cuh"
#include "../../src/Constants/behavior.h"
#include "../../src/FileManagement/file_listing.h"
#include "../../src/Math/reduction_abstracts.h"
#include "../../src/Math/reduction_bridge.h"
#include "../../src/Math/hpc_reduction.cuh"
#include "../../src/MolecularMechanics/mm_controls.h"
#include "../../src/MolecularMechanics/mm_evaluation.h"
#include "../../src/Potential/cacheresource.h"
#include "../../src/Potential/hpc_nonbonded_potential.cuh"
#include "../../src/Potential/hpc_valence_potential.cuh"
#include "../../src/Potential/scorecard.h"
#include "../../src/Reporting/error_format.h"
#include "../../src/Synthesis/atomgraph_synthesis.h"
#include "../../src/Synthesis/phasespace_synthesis.h"
#include "../../src/Synthesis/static_mask_synthesis.h"
#include "../../src/Synthesis/synthesis_abstracts.h"
#include "../../src/Synthesis/nonbonded_workunit.h"
#include "../../src/Synthesis/valence_workunit.h"
#include "../../src/Topology/atomgraph.h"
#include "../../src/Trajectory/phasespace.h"
#include "../../src/UnitTesting/stopwatch.h"
#include "../../src/UnitTesting/test_environment.h"
#include "../../src/UnitTesting/unit_test.h"

using namespace omni::card;
using namespace omni::constants;
using namespace omni::diskutil;
using namespace omni::energy;
using namespace omni::errors;
using namespace omni::math;
using namespace omni::mm;
using namespace omni::synthesis;
using namespace omni::testing;
using namespace omni::topology;
using namespace omni::trajectory;

//-------------------------------------------------------------------------------------------------
// main
//-------------------------------------------------------------------------------------------------
int main(const int argc, const char* argv[]) {
  TestEnvironment oe(argc, argv);
  StopWatch timer;
  HpcConfig gpu_config(ExceptionResponse::WARN);
  const std::vector<int> my_gpus = gpu_config.getGpuDevice(1);
  GpuDetails gpu = gpu_config.getGpuInfo(my_gpus[0]);
  
  // Section 1
  section("Minimize a collection of drug molecules and dipeptides");

  // Read small molecules and compile them into a synthesis
  const char osc = osSeparator();
  const std::string base_top_name  = oe.getOmniSourcePath() + osc + "test" + osc + "Topology";
  const std::string base_crd_name  = oe.getOmniSourcePath() + osc + "test" + osc + "Trajectory";
  const std::string alad_top_name = base_top_name + osc + "ala_dipeptide.top";
  const std::string alad_crd_name = base_crd_name + osc + "ala_dipeptide.inpcrd";
  const std::string brbz_top_name = base_top_name + osc + "bromobenzene_iso.top";
  const std::string brbz_crd_name = base_crd_name + osc + "bromobenzene_iso.inpcrd";
  const std::string lig1_top_name = base_top_name + osc + "stereo_L1.top";
  const std::string lig1_crd_name = base_crd_name + osc + "stereo_L1.inpcrd";
  const std::string lig2_top_name = base_top_name + osc + "symmetry_L1.top";
  const std::string lig2_crd_name = base_crd_name + osc + "symmetry_L1.inpcrd";
  const std::vector<std::string> all_top = { alad_top_name, brbz_top_name, lig1_top_name,
                                             lig2_top_name };
  const std::vector<std::string> all_crd = { alad_crd_name, brbz_crd_name, lig1_crd_name,
                                             lig2_crd_name };
  const int small_mol_count = all_top.size();
  bool files_exist = true;
  for (int i = 0; i < small_mol_count; i++) {
    files_exist = (getDrivePathType(all_top[i]) == DrivePathType::FILE &&
                   getDrivePathType(all_crd[i]) == DrivePathType::FILE && files_exist);
  }
  std::vector<AtomGraph> small_mol_ag;
  std::vector<AtomGraph*> small_mol_ag_ptr;
  std::vector<PhaseSpace> small_mol_ps;
  if (files_exist) {
    small_mol_ag.reserve(small_mol_count);
    small_mol_ps.reserve(small_mol_count);
    small_mol_ag_ptr.resize(small_mol_count);
    for (int i = 0; i < small_mol_count; i++) {
      small_mol_ag.emplace_back(all_top[i], ExceptionResponse::SILENT);
      small_mol_ps.emplace_back(all_crd[i]);
      small_mol_ag_ptr[i] = &small_mol_ag[i];
    }
  }
  else {
    small_mol_ag.resize(small_mol_count);
    small_mol_ag.resize(small_mol_count);
    rtWarn("Topology and coordinate files for a number of small molecules and dipeptides were not "
           "found.  Check the ${OMNI_SOURCE} environment variable, currently set to " +
           oe.getOmniSourcePath() + ", for validity.  Subsequent tests will be skipped.",
           "test_hpc_minimization");
  }
  std::vector<int> small_mol_id = { 0, 1, 2, 3, 0, 1, 2, 3, 0, 3, 1, 2, 2, 1, 3, 0 };
  small_mol_id.resize(1024);
  for (int i = 0; i < 64; i++) {
    for (int j = 0; j < 16; j++) {
      small_mol_id[(16 * i) + j] = small_mol_id[j];
    }
  }
  AtomGraphSynthesis small_poly_ag(small_mol_ag_ptr, small_mol_id, ExceptionResponse::WARN, gpu,
                                   &timer);
  StaticExclusionMaskSynthesis small_poly_se(small_poly_ag.getTopologyPointers(), small_mol_id);
  small_poly_ag.loadNonbondedWorkUnits(small_poly_se);
  PhaseSpaceSynthesis small_poly_ps(small_mol_ps, small_mol_ag_ptr, small_mol_id);
  
  // Create the minimization instructions
  MinimizeControls mincon;
  mincon.setTotalCycles(1000);
  
  // Create a molecular mechanics control object based on the minimization operations
  MolecularMechanicsControls mmctrl(mincon);

  // Track energies in the systems
  ScoreCard sc(small_mol_id.size(), mincon.getTotalCycles(), 32);

  // Obtain kernel launch parameters for the workload
  KernelManager launcher(gpu, small_poly_ag);
  
  // Lay out GPU cache resources
  const int2 vale_lp = launcher.getValenceKernelDims(PrecisionModel::SINGLE, EvaluateForce::YES,
                                                     EvaluateEnergy::YES,
                                                     ForceAccumulationMethod::SPLIT,
                                                     VwuGoal::ACCUMULATE);
  const int2 nonb_lp = launcher.getNonbondedKernelDims(PrecisionModel::SINGLE,
                                                       NbwuKind::TILE_GROUPS, EvaluateForce::YES,
                                                       EvaluateEnergy::YES,
                                                       ForceAccumulationMethod::SPLIT);
  CacheResource valence_tb_reserve(vale_lp.x, maximum_valence_work_unit_atoms);
  CacheResource nonbond_tb_reserve(nonb_lp.x, small_block_max_atoms);

  // Upload the synthesis and prime the pumps
  small_poly_ag.upload();
  small_poly_ps.upload();
  small_poly_se.upload();
  mmctrl.primeWorkUnitCounters(launcher, PrecisionModel::SINGLE, small_poly_ag);
  
  // Obtain the appropriate abstracts
  const HybridTargetLevel tier = HybridTargetLevel::DEVICE;
  const SyValenceKit<float> small_poly_vk = small_poly_ag.getSinglePrecisionValenceKit(tier);
  const SyNonbondedKit<float> small_poly_nbk = small_poly_ag.getSinglePrecisionNonbondedKit(tier);
  const SeMaskSynthesisReader small_poly_ser = small_poly_se.data(tier);
  const SyRestraintKit<float, float2, float4> small_poly_rk =
    small_poly_ag.getSinglePrecisionRestraintKit(tier);
  const NbwuKind nb_work_type = small_poly_ag.getNonbondedWorkType();
  MMControlKit<float> ctrl = mmctrl.spData(tier);
  PsSynthesisWriter small_poly_psw = small_poly_ps.data(tier);
  ScoreCardWriter scw = sc.data(tier);
  CacheResourceKit vale_tbk = valence_tb_reserve.spData(tier);
  CacheResourceKit nonb_tbk = nonbond_tb_reserve.spData(tier);
  ReductionKit small_poly_redk(small_poly_ag, tier);
  ReductionBridge small_poly_rbg(small_poly_ag.getReductionWorkUnitCount());
  ConjGradSubstrate cgsbs(&small_poly_ps, &small_poly_rbg, tier);
  
  // Run minimizations
  const int min_timings = timer.addCategory("Minimization of small molecules");
  timer.assignTime(0);
  for (int i = 0; i < mincon.getTotalCycles(); i++) {
    small_poly_ps.initializeForces(gpu, HybridTargetLevel::DEVICE);
    launchNonbonded(nb_work_type, small_poly_nbk, small_poly_ser, &ctrl, &small_poly_psw,
                    &scw, &nonb_tbk, EvaluateForce::YES, EvaluateEnergy::YES,
                    ForceAccumulationMethod::SPLIT, nonb_lp);
    launchValence(small_poly_vk, small_poly_rk, &ctrl, &small_poly_psw,
                  &scw, &vale_tbk, EvaluateForce::YES, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                  ForceAccumulationMethod::SPLIT, vale_lp);
    if (i == 0) {
      small_poly_ps.primeConjugateGradient(gpu, tier);
    }
    launchConjugateGradientSp(small_poly_redk, &cgsbs, &ctrl, launcher);

    // CHECK
    for (int j = 0; j < 1024; j += 173) {
      PhaseSpace chkj_ps = small_poly_ps.exportSystem(j, HybridTargetLevel::DEVICE);
      const std::vector<double> gpu_frc = chkj_ps.getInterlacedCoordinates(TrajectoryKind::FORCES);
      chkj_ps.initializeForces();
      ScoreCard tmp_sc(1, 1, 32);
      StaticExclusionMask chkj_se(small_poly_ag.getSystemTopologyPointer(j));
      evalNonbValeMM(&chkj_ps, &tmp_sc, small_poly_ag.getSystemTopologyPointer(j), chkj_se,
                     EvaluateForce::YES, 0);
      const std::vector<double> cpu_frc = chkj_ps.getInterlacedCoordinates(TrajectoryKind::FORCES);
      printf("System %4d:\n", j);
      for (int k = 0; k < chkj_ps.getAtomCount(); k++) {
        printf("  %9.4lf %9.4lf %9.4lf    %9.4lf %9.4lf %9.4lf\n", cpu_frc[3 * k],
               cpu_frc[(3 * k) + 1], cpu_frc[(3 * k) + 2], gpu_frc[3 * k], gpu_frc[(3 * k) + 1],
               gpu_frc[(3 * k) + 2]);
      }
      printf("\n");
    }
    if (i == 2) {
      exit(1);
    }
    // END CHECK
    
    ctrl.step += 1;
  }
  hipDeviceSynchronize();
  timer.assignTime(min_timings);
  
  // Summary evaluation
  if (oe.getDisplayTimingsOrder()) {
    timer.assignTime(0);
    timer.printResults();
  }
  printTestSummary(oe.getVerbosity());

  return 0;
}
