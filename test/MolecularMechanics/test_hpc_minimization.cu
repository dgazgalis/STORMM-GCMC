#include "hip/hip_runtime.h"
// -*-c++-*-
#include "../../src/Accelerator/hybrid.h"
#include "../../src/Accelerator/gpu_details.h"
#include "../../src/Accelerator/kernel_manager.h"
#include "../../src/Accelerator/hpc_config.cuh"
#include "../../src/Constants/behavior.h"
#include "../../src/FileManagement/file_listing.h"
#include "../../src/Math/reduction_abstracts.h"
#include "../../src/Math/reduction_bridge.h"
#include "../../src/Math/reduction_enumerators.h"
#include "../../src/Math/rounding.h"
#include "../../src/Math/hpc_reduction.h"
#include "../../src/MolecularMechanics/hpc_minimization.h"
#include "../../src/MolecularMechanics/line_minimization.h"
#include "../../src/MolecularMechanics/mm_controls.h"
#include "../../src/MolecularMechanics/mm_evaluation.h"
#include "../../src/Potential/cacheresource.h"
#include "../../src/Potential/hpc_nonbonded_potential.h"
#include "../../src/Potential/hpc_valence_potential.h"
#include "../../src/Potential/scorecard.h"
#include "../../src/Reporting/error_format.h"
#include "../../src/Synthesis/atomgraph_synthesis.h"
#include "../../src/Synthesis/phasespace_synthesis.h"
#include "../../src/Synthesis/static_mask_synthesis.h"
#include "../../src/Synthesis/synthesis_abstracts.h"
#include "../../src/Synthesis/nonbonded_workunit.h"
#include "../../src/Synthesis/valence_workunit.h"
#include "../../src/Topology/atomgraph.h"
#include "../../src/Trajectory/phasespace.h"
#include "../../src/UnitTesting/stopwatch.h"
#include "../../src/UnitTesting/test_environment.h"
#include "../../src/UnitTesting/unit_test.h"

using namespace stormm::card;
using namespace stormm::constants;
using namespace stormm::diskutil;
using namespace stormm::energy;
using namespace stormm::errors;
using namespace stormm::math;
using namespace stormm::mm;
using namespace stormm::synthesis;
using namespace stormm::testing;
using namespace stormm::topology;
using namespace stormm::trajectory;

//-------------------------------------------------------------------------------------------------
// Check that all forces on equivalent systems are equal, to within a reasonable tolerance.  It is
// assumed that systems described by the same topology are identical--no restraints, or at least
// no unique restraints, are present.
//
// Arguments:
//   poly_ps:  Compilation of coordinates and forces
//   poly_ag:  Compilation of topologies for all systems
//   err_msg:  Error to display if forces between equivalent systems do not match up
//   c_limit:  Tolerance for coordinates to disagree before being deemed inconsistent
//   f_limit:  Tolerance for forces to disagree before being deemed inconsistent
//   step_no:  Number of the minimization step
//-------------------------------------------------------------------------------------------------
bool checkConsistency(const PhaseSpaceSynthesis &poly_ps, const AtomGraphSynthesis &poly_ag,
                      const std::string &err_msg, const double c_limit, const double f_limit,
                      const int step_no) {
  int n_coord_mismatch = 0;
  int n_force_mismatch = 0;
  int n_syscrd_mismatch = 0;
  int n_sysfrc_mismatch = 0;
  PsSynthesisReader poly_psr = poly_ps.data();
  const std::vector<int> ag_indices = poly_ag.getTopologyIndices();
  std::vector<bool> covered(poly_psr.system_count, false);
  std::vector<bool> syscrd_ok(poly_psr.system_count, true);
  std::vector<bool> sysfrc_ok(poly_psr.system_count, true);
  for (int i = 0; i < poly_psr.system_count; i++) {
    if (covered[i]) {
      continue;
    }
    covered[i] = true;
    const int iag_no = ag_indices[i];
    for (int j = i + 1; j < poly_psr.system_count; j++) {
      if (covered[j] == false && ag_indices[j] == iag_no) {
        covered[j] = true;
        int itrack = poly_psr.atom_starts[i];
        const int jtrack_lim = poly_psr.atom_starts[j] + poly_psr.atom_counts[j];
        for (int jtrack = poly_psr.atom_starts[j]; jtrack < jtrack_lim; jtrack++) {
          if (fabs(poly_psr.xcrd[itrack] - poly_psr.xcrd[jtrack]) > f_limit ||
              fabs(poly_psr.ycrd[itrack] - poly_psr.ycrd[jtrack]) > f_limit ||
              fabs(poly_psr.zcrd[itrack] - poly_psr.zcrd[jtrack]) > f_limit) {
            n_coord_mismatch++;
            if (syscrd_ok[j]) {
              n_syscrd_mismatch++;
              syscrd_ok[j] = false;
            }
          }
          if (fabs(poly_psr.xfrc[itrack] - poly_psr.xfrc[jtrack]) > f_limit ||
              fabs(poly_psr.yfrc[itrack] - poly_psr.yfrc[jtrack]) > f_limit ||
              fabs(poly_psr.zfrc[itrack] - poly_psr.zfrc[jtrack]) > f_limit) {
            n_force_mismatch++;
            if (sysfrc_ok[j]) {
              n_sysfrc_mismatch++;
              sysfrc_ok[j] = false;
            }
          }
          itrack++;
        }
      }
    }
  }
  if (n_coord_mismatch > 0) {
    rtWarn("A total of " + std::to_string(n_coord_mismatch) + " atoms' coordinates were "
           "inconsistent among the first instance of a system and subsequent replicas with the "
           "same topology.  In all, " + std::to_string(n_syscrd_mismatch) + " systems displayed "
           "errors.  Checked at: " + err_msg + ", step " + std::to_string(step_no) + ".");
  }
  if (n_force_mismatch > 0) {
    rtWarn("A total of " + std::to_string(n_force_mismatch) + " atoms' forces were inconsistent "
           "among the first instance of a system and subsequent replicas with the same topology.  "
           "In all, " + std::to_string(n_sysfrc_mismatch) + " systems displayed errors.  Checked "
           "at: " + err_msg + ", step " + std::to_string(step_no) + ".");
  }
  return (n_coord_mismatch == 0 && n_force_mismatch == 0);
}

//-------------------------------------------------------------------------------------------------
// Set forces and coordinates for all systems to be exactly the same.
//
// Arguments:
//   poly_ps:  Compilation of coordinates and forces
//   poly_ag:  Compilation of topologies for all systems
//-------------------------------------------------------------------------------------------------
void mandateEquality(PhaseSpaceSynthesis *poly_ps, const AtomGraphSynthesis &poly_ag) {
  PsSynthesisWriter poly_psr = poly_ps->data();
  const std::vector<int> ag_indices = poly_ag.getTopologyIndices();
  std::vector<bool> covered(poly_psr.system_count, false);
  for (int i = 0; i < poly_psr.system_count; i++) {
    if (covered[i]) {
      continue;
    }
    covered[i] = true;
    const int iag_no = ag_indices[i];
    for (int j = i + 1; j < poly_psr.system_count; j++) {
      if (ag_indices[j] == iag_no) {
        covered[j] = true;
        int itrack = poly_psr.atom_starts[i];
        const int jtrack_lim = poly_psr.atom_starts[j] + poly_psr.atom_counts[j];
        for (int jtrack = poly_psr.atom_starts[j]; jtrack < jtrack_lim; jtrack++) {
          poly_psr.xfrc[jtrack] = poly_psr.xfrc[itrack];
          poly_psr.yfrc[jtrack] = poly_psr.yfrc[itrack];
          poly_psr.zfrc[jtrack] = poly_psr.zfrc[itrack];
          itrack++;
        }
      }
    }
  }
}

//-------------------------------------------------------------------------------------------------
// Perform energy minimization with meticulous checks to ensure that the process is consistent and
// reproducible.
//
// Arguments:
//   ag_ptr_vec:          Vector of pointers to unique topologies for systems in ps_vec
//   ps_vec:              Coordinates for all systems, to be replicated in the resulting synthesis
//   mol_id_vec:          Indication of how to replicate various structures described in ps_vec
//   gpu:                 Details of the GPU available
//   do_tests:            Indicate whether tests are possible to run
//   oe:                  Contains the name of the STORMM source path from shell variables as well
//                        as information on whether to write snapshot files or do the comparisons
//   psnap:               Instructions as to whether to begin printing a new snapshot file or
//                        append to an existing one, if snapshots are to be written
//   snap_name:           Name of the snapshot file for final energies of systems
//   prec:                Precision model for arithmetic and fixed-precision representations
//   gpos_bits:           Fixed-precision bits after the decimal in the positional representation
//   frc_bits:            Fixed-precision bits after the decimal in force accumulation
//   maxcyc:              Maximum number of minimization cycles
//   enforce_same_track:  Flag to have the systems explicitly set to keep on the same track if
//                        they start to diverge very slightly
//   check_mm:            Flag to have molecular mechanics energies and forces of the final states
//                        checked
//   frc_tol:             Tolerance for force comparisons
//   nrg_tol:             Tolerance for energy comparisons
//   test_name:           Name given to the collection of tests 
//   timer:               Optional profiling tool, if of interest for a brief performance check
//-------------------------------------------------------------------------------------------------
void metaMinimization(const std::vector<AtomGraph*> &ag_ptr_vec,
                      const std::vector<PhaseSpace> &ps_vec, const std::vector<int> &mol_id_vec,
                      const GpuDetails &gpu, const TestPriority do_tests,
                      const TestEnvironment &oe,
                      const PrintSituation psnap = PrintSituation::OVERWRITE, 
                      const std::string &snap_name = std::string(""),
                      const std::string &var_name = std::string(""),
                      const PrecisionModel prec = PrecisionModel::DOUBLE, const int gpos_bits = 40,
                      const int frc_bits = 40, const int maxcyc = 500,
                      const bool enforce_same_track = true, const bool check_mm = true,
                      const double frc_tol = 1.0e-6, const double nrg_tol = 1.0e-6,
                      const std::string &test_name = std::string(""), StopWatch *timer = nullptr) {
  AtomGraphSynthesis poly_ag(ag_ptr_vec, mol_id_vec, ExceptionResponse::SILENT, gpu, timer);
  StaticExclusionMaskSynthesis poly_se(poly_ag.getTopologyPointers(), mol_id_vec);
  poly_ag.loadNonbondedWorkUnits(poly_se);
  PhaseSpaceSynthesis poly_ps(ps_vec, ag_ptr_vec, mol_id_vec, gpos_bits, 24, 40, frc_bits);
  
  // Create the minimization instructions
  MinimizeControls mincon;
  mincon.setTotalCycles(maxcyc);
  mincon.setSteepestDescentCycles(maxcyc / 10);
  
  // Create separate molecular mechanics control objects based on the minimization operations
  // for each of the ways that the valence kernel gets subdivided. 
  MolecularMechanicsControls mmctrl_fe(mincon);
  MolecularMechanicsControls mmctrl_xe(mincon);

  // Track energies in the systems
  ScoreCard sc(mol_id_vec.size(), mincon.getTotalCycles(), 32);

  // Obtain kernel launch parameters for the workload
  KernelManager launcher(gpu, poly_ag);
  
  // Lay out GPU cache resources
  const int2 vale_fe_lp = launcher.getValenceKernelDims(prec, EvaluateForce::YES,
                                                        EvaluateEnergy::YES,
                                                        ForceAccumulationMethod::SPLIT,
                                                        VwuGoal::ACCUMULATE);
  const int2 vale_xe_lp = launcher.getValenceKernelDims(prec, EvaluateForce::NO,
                                                        EvaluateEnergy::YES,
                                                        ForceAccumulationMethod::SPLIT,
                                                        VwuGoal::ACCUMULATE);
  const int2 nonb_lp = launcher.getNonbondedKernelDims(prec,
                                                       NbwuKind::TILE_GROUPS,
                                                       EvaluateForce::YES, EvaluateEnergy::YES,
                                                       ForceAccumulationMethod::SPLIT);
  const int2 redu_lp = launcher.getReductionKernelDims(prec,
                                                       ReductionGoal::CONJUGATE_GRADIENT,
                                                       ReductionStage::ALL_REDUCE);
  CacheResource valence_fe_tb_reserve(vale_fe_lp.x, maximum_valence_work_unit_atoms);
  CacheResource valence_xe_tb_reserve(vale_xe_lp.x, maximum_valence_work_unit_atoms);
  CacheResource nonbond_tb_reserve(nonb_lp.x, small_block_max_atoms);
  
  // Upload the synthesis and prime the pumps
  poly_ag.upload();
  poly_ps.upload();
  poly_se.upload();
  mmctrl_fe.primeWorkUnitCounters(launcher, EvaluateForce::YES, EvaluateEnergy::YES, prec,
                                  poly_ag);
  mmctrl_xe.primeWorkUnitCounters(launcher, EvaluateForce::NO, EvaluateEnergy::YES, prec, poly_ag);
  
  // Obtain the appropriate abstracts
  const HybridTargetLevel devc = HybridTargetLevel::DEVICE;
  const SyValenceKit<double> d_poly_vk = poly_ag.getDoublePrecisionValenceKit(devc);
  const SyValenceKit<float>  f_poly_vk = poly_ag.getSinglePrecisionValenceKit(devc);
  const SyNonbondedKit<double> d_poly_nbk = poly_ag.getDoublePrecisionNonbondedKit(devc);
  const SyNonbondedKit<float>  f_poly_nbk = poly_ag.getSinglePrecisionNonbondedKit(devc);
  const SeMaskSynthesisReader poly_ser = poly_se.data(devc);
  const SyRestraintKit<double,
                       double2, double4> d_poly_rk = poly_ag.getDoublePrecisionRestraintKit(devc);
  const SyRestraintKit<float,
                       float2, float4> f_poly_rk = poly_ag.getSinglePrecisionRestraintKit(devc);
  const NbwuKind nb_work_type = poly_ag.getNonbondedWorkType();
  MMControlKit<double> d_ctrl_fe = mmctrl_fe.dpData(devc);
  MMControlKit<double> d_ctrl_xe = mmctrl_xe.dpData(devc);
  MMControlKit<float>  f_ctrl_fe = mmctrl_fe.spData(devc);
  MMControlKit<float>  f_ctrl_xe = mmctrl_xe.spData(devc);
  PsSynthesisWriter poly_psw = poly_ps.data(devc);
  ScoreCardWriter scw = sc.data(devc);
  CacheResourceKit<double> d_vale_fe_tbk = valence_fe_tb_reserve.dpData(devc);
  CacheResourceKit<double> d_vale_xe_tbk = valence_xe_tb_reserve.dpData(devc);
  CacheResourceKit<double> d_nonb_tbk = nonbond_tb_reserve.dpData(devc);
  CacheResourceKit<float> f_vale_fe_tbk = valence_fe_tb_reserve.spData(devc);
  CacheResourceKit<float> f_vale_xe_tbk = valence_xe_tb_reserve.spData(devc);
  CacheResourceKit<float> f_nonb_tbk = nonbond_tb_reserve.spData(devc);
  ReductionKit poly_redk(poly_ag, devc);
  ReductionBridge poly_rbg(poly_ag.getReductionWorkUnitCount());
  ConjGradSubstrate cgsbs(&poly_ps, &poly_rbg, devc);
  LineMinimization line_record(poly_ag.getSystemCount());
  line_record.primeMoveLengths(mmctrl_fe.getInitialMinimizationStep());
  LinMinWriter lmw = line_record.data(devc);

  // Run minimizations
  const int meta_timings = (timer == nullptr) ? 0 : timer->addCategory(test_name);
  if (timer != nullptr) {
    timer->assignTime(0);
  }
  const int n_mm_sample = roundUp(mincon.getTotalCycles(), 32) / 32;
  std::vector<double> cpu_total_e(n_mm_sample, 0.0);
  std::vector<double> gpu_total_e(n_mm_sample, 0.0);
  std::vector<double> force_mue(n_mm_sample, 0.0);
  int consistency_failures = 0;
  for (int i = 0; i < mincon.getTotalCycles(); i++) {
    
    // First stage of the cycle: compute forces and obtain the conjugate gradient move.
    poly_ps.initializeForces(gpu, devc);
    sc.initialize(devc, gpu);
    switch (prec) {
    case PrecisionModel::DOUBLE:
      launchNonbonded(nb_work_type, d_poly_nbk, poly_ser, &d_ctrl_fe, &poly_psw, &scw, &d_nonb_tbk,
                      EvaluateForce::YES, EvaluateEnergy::YES, nonb_lp);
      launchValence(d_poly_vk, d_poly_rk, &d_ctrl_fe, &poly_psw, &scw, &d_vale_fe_tbk,
                    EvaluateForce::YES, EvaluateEnergy::YES, VwuGoal::ACCUMULATE, vale_fe_lp);
      d_ctrl_fe.step += 1;
      break;
    case PrecisionModel::SINGLE:
      launchNonbonded(nb_work_type, f_poly_nbk, poly_ser, &f_ctrl_fe, &poly_psw, &scw, &f_nonb_tbk,
                      EvaluateForce::YES, EvaluateEnergy::YES, ForceAccumulationMethod::SPLIT,
                      nonb_lp);
      launchValence(f_poly_vk, f_poly_rk, &f_ctrl_fe, &poly_psw, &scw, &f_vale_fe_tbk,
                    EvaluateForce::YES, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                    ForceAccumulationMethod::SPLIT, vale_fe_lp);
      f_ctrl_fe.step += 1;
      break;
    }

    // Check the forces computed for a couple of systems.  This is somewhat redundant, but serves
    // as a sanity check in case other aspects of the energy minimization show problems.
    if (check_mm && (i & 0x1f) == 0) {
      const int jlim = (3 * i) + 1;
      const TrajectoryKind tforce = TrajectoryKind::FORCES; 
      for (int j = 3 * i; j < jlim; j++) {
        const int jmod = j % poly_ag.getSystemCount();
        PhaseSpace chkj_ps = poly_ps.exportSystem(jmod, devc);
        const std::vector<double> gpu_frc = chkj_ps.getInterlacedCoordinates(tforce);
        chkj_ps.initializeForces();
        ScoreCard tmp_sc(1, 1, 32);
        StaticExclusionMask chkj_se(poly_ag.getSystemTopologyPointer(jmod));
        evalNonbValeMM(&chkj_ps, &tmp_sc, poly_ag.getSystemTopologyPointer(jmod), chkj_se,
                       EvaluateForce::YES, 0);
        const std::vector<double> cpu_frc = chkj_ps.getInterlacedCoordinates(tforce);
        cpu_total_e[i / 32] = tmp_sc.reportTotalEnergy(0);
        gpu_total_e[i / 32] = sc.reportTotalEnergy(jmod, devc);
        force_mue[i / 32] = meanUnsignedError(cpu_frc, gpu_frc);
      }
    }
    
    // Download and check the forces for each system to verify consistency.  If the forces are
    // consistent enough, set them to be exactly consistent, and do the same with the coordinates,
    // to avoid miniscule roundoff errors that could otherwis creep in over hundreds of steps.
    if (enforce_same_track) {
      poly_ps.download();
      if (checkConsistency(poly_ps, poly_ag, "Force computation", 5.0e-7, 5.0e-7, i)) {
        mandateEquality(&poly_ps, poly_ag);
      }
      else {
        consistency_failures++;
      }
      poly_ps.upload();
    }

    // Perform the conjugate gradient transformation
    if (i == 0) {
      poly_ps.primeConjugateGradientCalculation(gpu, devc);
    }
    switch (prec) {
    case PrecisionModel::DOUBLE:
      launchConjugateGradient(poly_redk, &cgsbs, &d_ctrl_fe, redu_lp);
      break;
    case PrecisionModel::SINGLE:
      launchConjugateGradient(poly_redk, &cgsbs, &f_ctrl_fe, redu_lp);
      break;
    }
    
    // Download and check the conjugate gradient transformation.
    if (enforce_same_track) {
      poly_ps.download();
      if (checkConsistency(poly_ps, poly_ag, "Conjugate gradient transformation",
                           5.0e-7, 5.0e-7, i)) {
        mandateEquality(&poly_ps, poly_ag);
      }
      else {
        consistency_failures++;
      }
      poly_ps.upload();
    }
    
    // Second stage of the cycle: advance once along the line and recompute the energy.
    launchLineAdvance(prec, &poly_psw, poly_redk, scw, &lmw, 0, redu_lp);
    sc.initialize(devc, gpu);
    switch (prec) {
    case PrecisionModel::DOUBLE:
      launchNonbonded(nb_work_type, d_poly_nbk, poly_ser, &d_ctrl_xe, &poly_psw, &scw, &d_nonb_tbk,
                      EvaluateForce::NO, EvaluateEnergy::YES, nonb_lp);
      launchValence(d_poly_vk, d_poly_rk, &d_ctrl_xe, &poly_psw, &scw, &d_vale_xe_tbk,
                    EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE, vale_xe_lp);
      d_ctrl_xe.step += 1;
      break;
    case PrecisionModel::SINGLE:
      launchNonbonded(nb_work_type, f_poly_nbk, poly_ser, &f_ctrl_xe, &poly_psw, &scw, &f_nonb_tbk,
                      EvaluateForce::NO, EvaluateEnergy::YES, ForceAccumulationMethod::SPLIT,
                      nonb_lp);
      launchValence(f_poly_vk, f_poly_rk, &f_ctrl_xe, &poly_psw, &scw, &f_vale_xe_tbk,
                    EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                    ForceAccumulationMethod::SPLIT, vale_xe_lp);
      f_ctrl_xe.step += 1;
      break;
    }

    // Download and check the particle advancement.
    if (enforce_same_track) {
      poly_ps.download();
      if (checkConsistency(poly_ps, poly_ag, "Particle advance I", 1.0e5, 5.0e-7, i)) {
        mandateEquality(&poly_ps, poly_ag);
      }
      else {
        consistency_failures++;
      }
      poly_ps.upload();
    }
    
    // Third stage of the cycle: advance once more along the line and recompute the energy.
    launchLineAdvance(prec, &poly_psw, poly_redk, scw, &lmw, 1, redu_lp);
    sc.initialize(devc, gpu);
    switch (prec) {
    case PrecisionModel::DOUBLE:
      launchNonbonded(nb_work_type, d_poly_nbk, poly_ser, &d_ctrl_xe, &poly_psw, &scw, &d_nonb_tbk,
                      EvaluateForce::NO, EvaluateEnergy::YES, nonb_lp);
      launchValence(d_poly_vk, d_poly_rk, &d_ctrl_xe, &poly_psw, &scw, &d_vale_xe_tbk,
                    EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE, vale_xe_lp);
      d_ctrl_xe.step += 1;
      break;
    case PrecisionModel::SINGLE:
      launchNonbonded(nb_work_type, f_poly_nbk, poly_ser, &f_ctrl_xe, &poly_psw, &scw, &f_nonb_tbk,
                      EvaluateForce::NO, EvaluateEnergy::YES, ForceAccumulationMethod::SPLIT,
                      nonb_lp);
      launchValence(f_poly_vk, f_poly_rk, &f_ctrl_xe, &poly_psw, &scw, &f_vale_xe_tbk,
                    EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                    ForceAccumulationMethod::SPLIT, vale_xe_lp);
      f_ctrl_xe.step += 1;
      break;
    }

    // Download and check the particle advancement.
    if (enforce_same_track) {
      poly_ps.download();
      if (checkConsistency(poly_ps, poly_ag, "Particle advance II", 1.0e5, 5.0e-7, i)) {
        mandateEquality(&poly_ps, poly_ag);
      }
      else {
        consistency_failures++;
      }
      poly_ps.upload();
    }
    
    // Final stage of the cycle: advance a final time along the line and recompute the energy.
    launchLineAdvance(prec, &poly_psw, poly_redk, scw, &lmw, 2, redu_lp);
    sc.initialize(devc, gpu);
    switch (prec) {
    case PrecisionModel::DOUBLE:
      launchNonbonded(nb_work_type, d_poly_nbk, poly_ser, &d_ctrl_xe, &poly_psw, &scw, &d_nonb_tbk,
                      EvaluateForce::NO, EvaluateEnergy::YES, nonb_lp);
      launchValence(d_poly_vk, d_poly_rk, &d_ctrl_xe, &poly_psw, &scw, &d_vale_xe_tbk,
                    EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE, vale_xe_lp);
      d_ctrl_xe.step += 1;
      break;
    case PrecisionModel::SINGLE:
      launchNonbonded(nb_work_type, f_poly_nbk, poly_ser, &f_ctrl_xe, &poly_psw, &scw, &f_nonb_tbk,
                      EvaluateForce::NO, EvaluateEnergy::YES, ForceAccumulationMethod::SPLIT,
                      nonb_lp);
      launchValence(f_poly_vk, f_poly_rk, &f_ctrl_xe, &poly_psw, &scw, &f_vale_xe_tbk,
                    EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                    ForceAccumulationMethod::SPLIT, vale_xe_lp);
      f_ctrl_xe.step += 1;
      break;
    }

    // Download and check the particle advancement.
    if (enforce_same_track) {
      poly_ps.download();
      if (checkConsistency(poly_ps, poly_ag, "Particle advance III", 1.0e5, 5.0e-7, i)) {
        mandateEquality(&poly_ps, poly_ag);
      } 
      else {
        consistency_failures++;
      }
      poly_ps.upload();
    }
    
    // Fit a cubic polynomial to guess the best overall advancement, and place the system there.
    launchLineAdvance(prec, &poly_psw, poly_redk, scw, &lmw, 3, redu_lp);

    // Download and check the particle advancement.
    if (enforce_same_track) {
      poly_ps.download();
      if (checkConsistency(poly_ps, poly_ag, "Particle advance IV", 1.0e5, 5.0e-7, i)) {
        mandateEquality(&poly_ps, poly_ag);
      }
      else {
        consistency_failures++;
      }
      poly_ps.upload();
    }
  }
  hipDeviceSynchronize();
  if (timer != nullptr) {
    timer->assignTime(meta_timings);
  }
  if (enforce_same_track) {
    check(consistency_failures == 0, "Consistency failures occurred during the minimizations.  "
          "The results in identical systems diverged after approximately " +
          std::to_string((consistency_failures + 5) / 6) + " steps.", do_tests);
  }
  if (check_mm) {
    const int ave_atom_count = poly_ag.getAtomCount() / poly_ag.getSystemCount();
    check(cpu_total_e, RelationalOperator::EQUAL, Approx(gpu_total_e).margin(nrg_tol),
          "Energies of relaxed structures did not agree.  Average structure size: " +
          std::to_string(ave_atom_count) + ".  Precision level: " + getPrecisionModelName(prec) +
          ".  Test name: " + test_name + ".", do_tests);
    check(force_mue, RelationalOperator::EQUAL,
          Approx(std::vector<double>(n_mm_sample, 0.0)).margin(frc_tol), "Snapshots of forces "
          "taken during energy minimization on the GPU do not agree with their CPU-derived "
          "counterparts.  Average structure size: " + std::to_string(ave_atom_count) +
          ".  Precision level: " + getPrecisionModelName(prec) + ".  Test name: " + test_name +
          ".", do_tests);
  }
  
  // Verify that the energies for all systems meet the expected values
  if (check_mm) {
    const bool snap_exists = (getDrivePathType(snap_name) == DrivePathType::FILE);
    if (snap_exists == false) {
      rtWarn("The snapshot file " + snap_name + " could not be found.  Check the ${STORMM_SOURCE} "
             "environment variable, currently set to " + oe.getStormmSourcePath() + ", for "
             "validity.  Subsequent tests will be skipped.", "test_hpc_minimization");
    }
    const TestPriority do_snps = (snap_exists &&
                                  do_tests == TestPriority::CRITICAL) ? TestPriority::CRITICAL :
                                                                        TestPriority::ABORT;
    const std::vector<double> final_e = sc.reportTotalEnergies(devc);
    const std::string test_var = var_name + ((prec == PrecisionModel::DOUBLE) ? "d" : "f");
    snapshot(snap_name, polyNumericVector(final_e), test_var, 1.0e-6, "Final energies of "
             "energy-minimized structures did not reach their expected values.  Test: " +
             test_name + ".  Precision model: " + getPrecisionModelName(prec) + ".",
             oe.takeSnapshot(), 1.0e-8, NumberFormat::STANDARD_REAL, psnap, do_snps);
  }
}

//-------------------------------------------------------------------------------------------------
// Read a series of topologies and coordinate files, then process them into a tiled array of
// structures for energy minimization.
//
// Arguments:
//   top_names:  Names of topology files to seek out and read
//   crd_names:  Names of coordinate files to seek out and read
//   tile_list:  Indices of structures to add to the synthesis
//   n_tiles:    The number of times to repeat the tile list when making the synthesis
//   test_name:  Name given to this group of tests
//   oe:         Contains the name of the STORMM source path from shell variables
//   gpu:        Details of the GPU in use
//   test_name:  Name given to this test 
//   psnap:      Instructions as to whether to begin printing a new snapshot file or append to an
//               existing one, if snapshots are to be written
//   snap_name:  Name of the snapshot file for final energies of systems
//   timer:      Time tracking object for optional performance analysis
//-------------------------------------------------------------------------------------------------
void testCompilation(const std::vector<std::string> &top_names,
                     const std::vector<std::string> &crd_names, const std::vector<int> tile_list,
                     const int n_tiles, const double frc_tol, const double nrg_tol,
                     const TestEnvironment &oe, const GpuDetails &gpu,
                     const std::string &test_name, const PrintSituation psnap,
                     const std::string &snap_name, const std::string &var_name, StopWatch *timer) {
  const int mol_count = top_names.size();
  if (crd_names.size() != top_names.size()) {
    rtErr("A total of " + std::to_string(top_names.size()) + " topologies and " +
          std::to_string(crd_names.size()) + " coordinate files were provided.  The counts must "
          "match.", "test_hpc_minimization", "testCompilation");
  }
  bool files_exist = true;
  for (int i = 0; i < mol_count; i++) {
    files_exist = (getDrivePathType(top_names[i]) == DrivePathType::FILE &&
                   getDrivePathType(crd_names[i]) == DrivePathType::FILE && files_exist);
  }
  std::vector<AtomGraph> mol_ag;
  std::vector<AtomGraph*> mol_ag_ptr;
  std::vector<PhaseSpace> mol_ps;
  if (files_exist) {
    mol_ag.reserve(mol_count);
    mol_ps.reserve(mol_count);
    mol_ag_ptr.resize(mol_count);
    for (int i = 0; i < mol_count; i++) {
      mol_ag.emplace_back(top_names[i], ExceptionResponse::SILENT);
      mol_ps.emplace_back(crd_names[i]);
      mol_ag_ptr[i] = &mol_ag[i];
    }
  }
  else {
    mol_ag.resize(mol_count);
    mol_ag.resize(mol_count);
    rtWarn("Topology and coordinate files for a number of small molecules and dipeptides were not "
           "found.  Check the ${STORMM_SOURCE} environment variable, currently set to " +
           oe.getStormmSourcePath() + ", for validity.  Subsequent tests will be skipped.",
           "test_hpc_minimization");
  }
  const TestPriority do_tests = (files_exist) ? TestPriority::CRITICAL : TestPriority::ABORT;
  const int tlen = tile_list.size();
  const int total_mol = tlen * n_tiles;
  std::vector<int> mol_id(total_mol);
  std::vector<int> d_nrg_target(total_mol);
  std::vector<int> f_nrg_target(total_mol);

  // The test name determines the content of the target energy vector.  Codify the test name.
  for (int i = 0; i < n_tiles; i++) {
    for (int j = 0; j < tlen; j++) {
      mol_id[(tlen * i) + j] = tile_list[j];
    }
  }
  const PrintSituation x_psnap = (psnap == PrintSituation::OVERWRITE) ? PrintSituation::OVERWRITE :
                                                                        PrintSituation::APPEND;
  metaMinimization(mol_ag_ptr, mol_ps, mol_id, gpu, do_tests, oe, x_psnap, snap_name, var_name,
                   PrecisionModel::DOUBLE, 40, 40, 100, false, true, frc_tol, nrg_tol,
                   test_name + " (fp64)", timer);
  metaMinimization(mol_ag_ptr, mol_ps, mol_id, gpu, do_tests, oe, PrintSituation::APPEND,
                   snap_name, var_name, PrecisionModel::SINGLE, 28, 24, 500, false, true,
                   10.0 * frc_tol, 10.0 * nrg_tol, test_name + " (fp32)", timer);
}

//-------------------------------------------------------------------------------------------------
// main
//-------------------------------------------------------------------------------------------------
int main(const int argc, const char* argv[]) {
  TestEnvironment oe(argc, argv);
  StopWatch timer;
  HpcConfig gpu_config(ExceptionResponse::WARN);
  const std::vector<int> my_gpus = gpu_config.getGpuDevice(1);
  GpuDetails gpu = gpu_config.getGpuInfo(my_gpus[0]);

  // Kernel __shared__ memory configuration
  reductionKernelSetup();
  minimizationKernelSetup();
  
  // Section 1
  section("Minimize a collection of drug molecules and dipeptides");

  // Read small molecules and compile them into a synthesis
  const char osc = osSeparator();
  const std::string base_top_name  = oe.getStormmSourcePath() + osc + "test" + osc + "Topology";
  const std::string base_crd_name  = oe.getStormmSourcePath() + osc + "test" + osc + "Trajectory";
  const std::string alad_top_name = base_top_name + osc + "ala_dipeptide.top";
  const std::string alad_crd_name = base_crd_name + osc + "ala_dipeptide.inpcrd";
  const std::string brbz_top_name = base_top_name + osc + "bromobenzene_iso.top";
  const std::string brbz_crd_name = base_crd_name + osc + "bromobenzene_iso.inpcrd";
  const std::string lig1_top_name = base_top_name + osc + "stereo_L1.top";
  const std::string lig1_crd_name = base_crd_name + osc + "stereo_L1.inpcrd";
  const std::string lig2_top_name = base_top_name + osc + "symmetry_L1.top";
  const std::string lig2_crd_name = base_crd_name + osc + "symmetry_L1.inpcrd";
  const std::string trpi_top_name = base_top_name + osc + "trpcage.top";
  const std::string trpi_crd_name = base_crd_name + osc + "trpcage.inpcrd";
  const std::string dhfr_top_name = base_top_name + osc + "dhfr_cmap.top";
  const std::string dhfr_crd_name = base_crd_name + osc + "dhfr_cmap.inpcrd";
  const std::vector<std::string> lig_top = { alad_top_name, brbz_top_name, lig1_top_name,
                                             lig2_top_name };
  const std::vector<std::string> lig_crd = { alad_crd_name, brbz_crd_name, lig1_crd_name,
                                             lig2_crd_name };
  const std::vector<std::string> pro_top = { trpi_top_name, dhfr_top_name };
  const std::vector<std::string> pro_crd = { trpi_crd_name, dhfr_crd_name };
  const std::string snap_name = oe.getStormmSourcePath() + osc + "test" + osc +
                                "MolecularMechanics" + osc + "min_energy.m";
  
  // Run small molecule tests
  testCompilation(lig_top, lig_crd, { 0, 1, 2, 3, 0, 1, 2, 3, 0, 3, 1, 2, 2, 1, 3, 0 },
                  256, 1.0e-5, 1.0e-5, oe, gpu, "Small molecules", PrintSituation::OVERWRITE,
                  snap_name, "small_mol_", &timer);

  // Run tests on small proteins
  testCompilation(pro_top, pro_crd, { 0, 1, 0, 1, 1, 1, 0, 0 }, 3, 1.0e-5, 1.0e-3, oe, gpu,
                  "Folded proteins", PrintSituation::APPEND, snap_name, "folded_pro_", &timer);

  // Run tests on small proteins
  testCompilation(pro_top, pro_crd, { 0, 0, 0, 0, 0, 0, 0, 0 }, 8, 1.0e-5, 6.0e-5, oe, gpu,
                  "Trp-cage only", PrintSituation::APPEND, snap_name, "trp_cage_", &timer);

  // Run tests on small proteins
  testCompilation(pro_top, pro_crd, { 1, 1, 1, 1, 1, 1, 1, 1 }, 1, 1.0e-5, 6.0e-3, oe, gpu,
                  "DHFR only", PrintSituation::APPEND, snap_name, "dhfr_", &timer);
  
  // Summary evaluation
  if (oe.getDisplayTimingsOrder()) {
    timer.assignTime(0);
    timer.printResults();
  }
  printTestSummary(oe.getVerbosity());
  
  return 0;
}
