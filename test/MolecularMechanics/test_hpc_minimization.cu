#include "hip/hip_runtime.h"
// -*-c++-*-
#include "../../src/Accelerator/hybrid.h"
#include "../../src/Accelerator/gpu_details.h"
#include "../../src/Accelerator/kernel_manager.h"
#include "../../src/Accelerator/hpc_config.cuh"
#include "../../src/Constants/behavior.h"
#include "../../src/FileManagement/file_listing.h"
#include "../../src/Math/reduction_abstracts.h"
#include "../../src/Math/reduction_bridge.h"
#include "../../src/Math/reduction_enumerators.h"
#include "../../src/Math/hpc_reduction.h"
#include "../../src/MolecularMechanics/hpc_minimization.h"
#include "../../src/MolecularMechanics/line_minimization.h"
#include "../../src/MolecularMechanics/mm_controls.h"
#include "../../src/MolecularMechanics/mm_evaluation.h"
#include "../../src/Potential/cacheresource.h"
#include "../../src/Potential/hpc_nonbonded_potential.h"
#include "../../src/Potential/hpc_valence_potential.h"
#include "../../src/Potential/scorecard.h"
#include "../../src/Reporting/error_format.h"
#include "../../src/Synthesis/atomgraph_synthesis.h"
#include "../../src/Synthesis/phasespace_synthesis.h"
#include "../../src/Synthesis/static_mask_synthesis.h"
#include "../../src/Synthesis/synthesis_abstracts.h"
#include "../../src/Synthesis/nonbonded_workunit.h"
#include "../../src/Synthesis/valence_workunit.h"
#include "../../src/Topology/atomgraph.h"
#include "../../src/Trajectory/phasespace.h"
#include "../../src/UnitTesting/stopwatch.h"
#include "../../src/UnitTesting/test_environment.h"
#include "../../src/UnitTesting/unit_test.h"

using namespace omni::card;
using namespace omni::constants;
using namespace omni::diskutil;
using namespace omni::energy;
using namespace omni::errors;
using namespace omni::math;
using namespace omni::mm;
using namespace omni::synthesis;
using namespace omni::testing;
using namespace omni::topology;
using namespace omni::trajectory;

//-------------------------------------------------------------------------------------------------
// Check that all forces on equivalent systems are equal, to within a reasonable tolerance.  It is
// assumed that systems described by the same topology are identical--no restraints, or at least
// no unique restraints, are present.
//
// Arguments:
//   poly_ps:  Compilation of coordinates and forces
//   poly_ag:  Compilation of topologies for all systems
//   err_msg:  Error to display if forces between equivalent systems do not match up
//   c_limit:  Tolerance for coordinates to disagree before being deemed inconsistent
//   f_limit:  Tolerance for forces to disagree before being deemed inconsistent
//   step_no:  Number of the minimization step
//-------------------------------------------------------------------------------------------------
bool checkConsistency(const PhaseSpaceSynthesis &poly_ps, const AtomGraphSynthesis &poly_ag,
                      const std::string &err_msg, const double c_limit, const double f_limit,
                      const int step_no) {
  int n_coord_mismatch = 0;
  int n_force_mismatch = 0;
  int n_syscrd_mismatch = 0;
  int n_sysfrc_mismatch = 0;
  PsSynthesisReader poly_psr = poly_ps.data();
  const std::vector<int> ag_indices = poly_ag.getTopologyIndices();
  std::vector<bool> covered(poly_psr.system_count, false);
  std::vector<bool> syscrd_ok(poly_psr.system_count, true);
  std::vector<bool> sysfrc_ok(poly_psr.system_count, true);
  for (int i = 0; i < poly_psr.system_count; i++) {
    if (covered[i]) {
      continue;
    }
    covered[i] = true;
    const int iag_no = ag_indices[i];
    for (int j = i + 1; j < poly_psr.system_count; j++) {
      if (covered[j] == false && ag_indices[j] == iag_no) {
        covered[j] = true;
        int itrack = poly_psr.atom_starts[i];
        const int jtrack_lim = poly_psr.atom_starts[j] + poly_psr.atom_counts[j];
        for (int jtrack = poly_psr.atom_starts[j]; jtrack < jtrack_lim; jtrack++) {
          if (fabs(poly_psr.xcrd[itrack] - poly_psr.xcrd[jtrack]) > f_limit ||
              fabs(poly_psr.ycrd[itrack] - poly_psr.ycrd[jtrack]) > f_limit ||
              fabs(poly_psr.zcrd[itrack] - poly_psr.zcrd[jtrack]) > f_limit) {
            n_coord_mismatch++;
            if (syscrd_ok[j]) {
              n_syscrd_mismatch++;
              syscrd_ok[j] = false;
            }
          }
          if (fabs(poly_psr.xfrc[itrack] - poly_psr.xfrc[jtrack]) > f_limit ||
              fabs(poly_psr.yfrc[itrack] - poly_psr.yfrc[jtrack]) > f_limit ||
              fabs(poly_psr.zfrc[itrack] - poly_psr.zfrc[jtrack]) > f_limit) {
            n_force_mismatch++;
            if (sysfrc_ok[j]) {
              n_sysfrc_mismatch++;
              sysfrc_ok[j] = false;
            }
          }
          itrack++;
        }
      }
    }
  }
  if (n_coord_mismatch > 0) {
    rtWarn("A total of " + std::to_string(n_coord_mismatch) + " atoms' coordinates were "
           "inconsistent among the first instance of a system and subsequent replicas with the "
           "same topology.  In all, " + std::to_string(n_syscrd_mismatch) + " systems displayed "
           "errors.  Checked at: " + err_msg + ", step " + std::to_string(step_no) + ".");
  }
  if (n_force_mismatch > 0) {
    rtWarn("A total of " + std::to_string(n_force_mismatch) + " atoms' forces were inconsistent "
           "among the first instance of a system and subsequent replicas with the same topology.  "
           "In all, " + std::to_string(n_sysfrc_mismatch) + " systems displayed errors.  Checked "
           "at: " + err_msg + ", step " + std::to_string(step_no) + ".");
  }
  return (n_coord_mismatch == 0 && n_force_mismatch == 0);
}

//-------------------------------------------------------------------------------------------------
// main
//-------------------------------------------------------------------------------------------------
int main(const int argc, const char* argv[]) {
  TestEnvironment oe(argc, argv);
  StopWatch timer;
  HpcConfig gpu_config(ExceptionResponse::WARN);
  const std::vector<int> my_gpus = gpu_config.getGpuDevice(1);
  GpuDetails gpu = gpu_config.getGpuInfo(my_gpus[0]);
  
  // Section 1
  section("Minimize a collection of drug molecules and dipeptides");

  // Read small molecules and compile them into a synthesis
  const char osc = osSeparator();
  const std::string base_top_name  = oe.getOmniSourcePath() + osc + "test" + osc + "Topology";
  const std::string base_crd_name  = oe.getOmniSourcePath() + osc + "test" + osc + "Trajectory";
  const std::string alad_top_name = base_top_name + osc + "ala_dipeptide.top";
  const std::string alad_crd_name = base_crd_name + osc + "ala_dipeptide.inpcrd";
  const std::string brbz_top_name = base_top_name + osc + "bromobenzene_iso.top";
  const std::string brbz_crd_name = base_crd_name + osc + "bromobenzene_iso.inpcrd";
  const std::string lig1_top_name = base_top_name + osc + "stereo_L1.top";
  const std::string lig1_crd_name = base_crd_name + osc + "stereo_L1.inpcrd";
  const std::string lig2_top_name = base_top_name + osc + "symmetry_L1.top";
  const std::string lig2_crd_name = base_crd_name + osc + "symmetry_L1.inpcrd";
  const std::vector<std::string> all_top = { alad_top_name, brbz_top_name, lig1_top_name,
                                             lig2_top_name };
  const std::vector<std::string> all_crd = { alad_crd_name, brbz_crd_name, lig1_crd_name,
                                             lig2_crd_name };
  const int small_mol_count = all_top.size();
  bool files_exist = true;
  for (int i = 0; i < small_mol_count; i++) {
    files_exist = (getDrivePathType(all_top[i]) == DrivePathType::FILE &&
                   getDrivePathType(all_crd[i]) == DrivePathType::FILE && files_exist);
  }
  std::vector<AtomGraph> small_mol_ag;
  std::vector<AtomGraph*> small_mol_ag_ptr;
  std::vector<PhaseSpace> small_mol_ps;
  if (files_exist) {
    small_mol_ag.reserve(small_mol_count);
    small_mol_ps.reserve(small_mol_count);
    small_mol_ag_ptr.resize(small_mol_count);
    for (int i = 0; i < small_mol_count; i++) {
      small_mol_ag.emplace_back(all_top[i], ExceptionResponse::SILENT);
      small_mol_ps.emplace_back(all_crd[i]);
      small_mol_ag_ptr[i] = &small_mol_ag[i];
    }
  }
  else {
    small_mol_ag.resize(small_mol_count);
    small_mol_ag.resize(small_mol_count);
    rtWarn("Topology and coordinate files for a number of small molecules and dipeptides were not "
           "found.  Check the ${OMNI_SOURCE} environment variable, currently set to " +
           oe.getOmniSourcePath() + ", for validity.  Subsequent tests will be skipped.",
           "test_hpc_minimization");
  }
  //const TestPriority do_tests = (files_exist) ? TestPriority::CRITICAL : TestPriority::ABORT;
  std::vector<int> small_mol_id = { 0, 1, 2, 3, 0, 1, 2, 3, 0, 3, 1, 2, 2, 1, 3, 0 };
  small_mol_id.resize(8192);
  for (int i = 0; i < 512; i++) {
    for (int j = 0; j < 16; j++) {
      small_mol_id[(16 * i) + j] = small_mol_id[j];
    }
  }
  AtomGraphSynthesis small_poly_ag(small_mol_ag_ptr, small_mol_id, ExceptionResponse::WARN, gpu,
                                   &timer);
  StaticExclusionMaskSynthesis small_poly_se(small_poly_ag.getTopologyPointers(), small_mol_id);
  small_poly_ag.loadNonbondedWorkUnits(small_poly_se);
  PhaseSpaceSynthesis small_poly_ps(small_mol_ps, small_mol_ag_ptr, small_mol_id, 40, 24, 40, 40);
  
  // Create the minimization instructions
  MinimizeControls mincon;
  mincon.setTotalCycles(500);
  
  // Create a molecular mechanics control object based on the minimization operations
  MolecularMechanicsControls mmctrl(mincon);

  // Track energies in the systems
  ScoreCard sc(small_mol_id.size(), mincon.getTotalCycles(), 32);

  // Obtain kernel launch parameters for the workload
  KernelManager launcher(gpu, small_poly_ag);
  
  // Lay out GPU cache resources
  const int2 vale_lp = launcher.getValenceKernelDims(PrecisionModel::DOUBLE, EvaluateForce::YES,
                                                     EvaluateEnergy::YES,
                                                     ForceAccumulationMethod::SPLIT,
                                                     VwuGoal::ACCUMULATE);
  const int2 nonb_lp = launcher.getNonbondedKernelDims(PrecisionModel::DOUBLE,
                                                       NbwuKind::TILE_GROUPS,
                                                       EvaluateForce::YES, EvaluateEnergy::YES,
                                                       ForceAccumulationMethod::SPLIT);
  const int2 redu_lp = launcher.getReductionKernelDims(PrecisionModel::DOUBLE,
                                                       ReductionGoal::CONJUGATE_GRADIENT,
                                                       ReductionStage::ALL_REDUCE);
  CacheResource valence_tb_reserve(vale_lp.x, maximum_valence_work_unit_atoms);
  CacheResource nonbond_tb_reserve(nonb_lp.x, small_block_max_atoms);

  // Upload the synthesis and prime the pumps
  small_poly_ag.upload();
  small_poly_ps.upload();
  small_poly_se.upload();
  mmctrl.primeWorkUnitCounters(launcher, PrecisionModel::DOUBLE, small_poly_ag);
  
  // Obtain the appropriate abstracts
  const HybridTargetLevel tier = HybridTargetLevel::DEVICE;
  const SyValenceKit<double> small_poly_vk = small_poly_ag.getDoublePrecisionValenceKit(tier);
  const SyNonbondedKit<double> small_poly_nbk = small_poly_ag.getDoublePrecisionNonbondedKit(tier);
  const SeMaskSynthesisReader small_poly_ser = small_poly_se.data(tier);
  const SyRestraintKit<double, double2, double4> small_poly_rk =
    small_poly_ag.getDoublePrecisionRestraintKit(tier);
  const NbwuKind nb_work_type = small_poly_ag.getNonbondedWorkType();
  MMControlKit<double> ctrl = mmctrl.dpData(tier);
  PsSynthesisWriter small_poly_psw = small_poly_ps.data(tier);
  ScoreCardWriter scw = sc.data(tier);
  CacheResourceKit vale_tbk = valence_tb_reserve.dpData(tier);
  CacheResourceKit nonb_tbk = nonbond_tb_reserve.dpData(tier);
  ReductionKit small_poly_redk(small_poly_ag, tier);
  ReductionBridge small_poly_rbg(small_poly_ag.getReductionWorkUnitCount());
  ConjGradSubstrate cgsbs(&small_poly_ps, &small_poly_rbg, tier);
  LineMinimization line_record(small_poly_ag.getSystemCount());
  line_record.primeMoveLengths(mmctrl.getInitialMinimizationStep());
  LinMinWriter lmw = line_record.data();
  
  // Run minimizations
  const int min_timings = timer.addCategory("Minimization of small molecules");
  timer.assignTime(0);
  for (int i = 0; i < mincon.getTotalCycles(); i++) {

    // First stage of the cycle: compute forces and obtain the conjugate gradient move.
    small_poly_ps.initializeForces(gpu, HybridTargetLevel::DEVICE);
    sc.initialize(HybridTargetLevel::DEVICE, gpu);
    launchNonbonded(nb_work_type, small_poly_nbk, small_poly_ser, &ctrl, &small_poly_psw,
                    &scw, &nonb_tbk, EvaluateForce::YES, EvaluateEnergy::YES, nonb_lp);
    launchValence(small_poly_vk, small_poly_rk, &ctrl, &small_poly_psw,
                  &scw, &vale_tbk, EvaluateForce::YES, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                  vale_lp);

    // Download and check the forces for each system to verify consistency.  If the forces are
    // consistent enough, set them to be exactly consistent, and do the same with the coordinates,
    // to avoid miniscule roundoff errors that could otherwis creep in over hundreds of steps.
    small_poly_ps.download();
    if (checkConsistency(small_poly_ps, small_poly_ag, "Force computation", 5.0e-7, 5.0e-7, i)) {
      //mandateForceEquality(&small_poly_ps, small_poly_ag);
    }
    small_poly_ps.upload();
    
    // CHECK
    if (i == 48) {
      int k = 0;
      for (int j = 0; j < 512; j++) {
        printf(" %12.6lf %12.6lf", sc.reportTotalEnergy((j * 16) + 2, HybridTargetLevel::DEVICE),
               sc.reportTotalEnergy((j * 16) + 6, HybridTargetLevel::DEVICE));
        k++;
        if (k == 8) {
          k = 0;
          printf("\n");
        }
      }
    }
    // END CHECK
    
    if (i == 0) {
      small_poly_ps.primeConjugateGradientCalculation(gpu, tier);
    }
    launchConjugateGradient(small_poly_redk, &cgsbs, &ctrl, redu_lp);

    // Download and check the conjugate gradient transformation.
    small_poly_ps.download();
    if (checkConsistency(small_poly_ps, small_poly_ag, "Conjugate gradient transformation",
                         5.0e-7, 5.0e-7, i)) {
      //mandateForceEquality(&small_poly_ps, small_poly_ag);
    }
    small_poly_ps.upload();

    // Second stage of the cycle: advance once along the line and recompute the energy.
    launchLineAdvance(PrecisionModel::DOUBLE, &small_poly_psw, small_poly_redk, scw,
                      &lmw, 0, redu_lp);
    ctrl.step += 1;
    sc.initialize(HybridTargetLevel::DEVICE, gpu);
    launchNonbonded(nb_work_type, small_poly_nbk, small_poly_ser, &ctrl, &small_poly_psw,
                    &scw, &nonb_tbk, EvaluateForce::NO, EvaluateEnergy::YES, nonb_lp);
    launchValence(small_poly_vk, small_poly_rk, &ctrl, &small_poly_psw,
                  &scw, &vale_tbk, EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                  vale_lp);

    // Download and check the particle advancement.
    small_poly_ps.download();
    if (checkConsistency(small_poly_ps, small_poly_ag, "Particle advance I", 1.0e5, 5.0e-7, i)) {
      //mandateForceEquality(&small_poly_ps, small_poly_ag);
    }
    small_poly_ps.upload();

    // Third stage of the cycle: advance once more along the line and recompute the energy.
    launchLineAdvance(PrecisionModel::DOUBLE, &small_poly_psw, small_poly_redk, scw,
                      &lmw, 1, redu_lp);
    ctrl.step += 1;
    sc.initialize(HybridTargetLevel::DEVICE, gpu);
    launchNonbonded(nb_work_type, small_poly_nbk, small_poly_ser, &ctrl, &small_poly_psw,
                    &scw, &nonb_tbk, EvaluateForce::NO, EvaluateEnergy::YES, nonb_lp);
    launchValence(small_poly_vk, small_poly_rk, &ctrl, &small_poly_psw,
                  &scw, &vale_tbk, EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                  vale_lp);

    // Download and check the particle advancement.
    small_poly_ps.download();
    if (checkConsistency(small_poly_ps, small_poly_ag, "Particle advance II", 1.0e5, 5.0e-7, i)) {
      //mandateForceEquality(&small_poly_ps, small_poly_ag);
    }
    small_poly_ps.upload();

    // Final stage of the cycle: advance a final time along the line, recompute the energy, fit
    // a cubic polynomial to guess the best overall advancement, and place the system there.
    launchLineAdvance(PrecisionModel::DOUBLE, &small_poly_psw, small_poly_redk, scw,
                      &lmw, 2, redu_lp);
    ctrl.step += 1;
    sc.initialize(HybridTargetLevel::DEVICE, gpu);
    launchNonbonded(nb_work_type, small_poly_nbk, small_poly_ser, &ctrl, &small_poly_psw,
                    &scw, &nonb_tbk, EvaluateForce::NO, EvaluateEnergy::YES, nonb_lp);
    launchValence(small_poly_vk, small_poly_rk, &ctrl, &small_poly_psw,
                  &scw, &vale_tbk, EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                  vale_lp);

    // Download and check the particle advancement.
    small_poly_ps.download();
    if (checkConsistency(small_poly_ps, small_poly_ag, "Particle advance III", 1.0e5, 5.0e-7, i)) {
      //mandateForceEquality(&small_poly_ps, small_poly_ag);
    }
    small_poly_ps.upload();

    launchLineAdvance(PrecisionModel::DOUBLE, &small_poly_psw, small_poly_redk, scw,
                      &lmw, 3, redu_lp);
    ctrl.step += 1;

    // Download and check the particle advancement.
    small_poly_ps.download();
    if (checkConsistency(small_poly_ps, small_poly_ag, "Particle advance IV", 1.0e5, 5.0e-7, i)) {
      //mandateForceEquality(&small_poly_ps, small_poly_ag);
    }
    small_poly_ps.upload();

    // CHECK
#if 0
    sc.initialize(HybridTargetLevel::DEVICE, gpu);
    launchNonbonded(nb_work_type, small_poly_nbk, small_poly_ser, &ctrl, &small_poly_psw,
                    &scw, &nonb_tbk, EvaluateForce::NO, EvaluateEnergy::YES, nonb_lp);
    launchValence(small_poly_vk, small_poly_rk, &ctrl, &small_poly_psw,
                  &scw, &vale_tbk, EvaluateForce::NO, EvaluateEnergy::YES, VwuGoal::ACCUMULATE,
                  vale_lp);
    for (int j = 15; j < 30; j += 173) {
      PhaseSpace chkj_ps = small_poly_ps.exportSystem(j, HybridTargetLevel::DEVICE);
      const std::vector<double> gpu_frc = chkj_ps.getInterlacedCoordinates(TrajectoryKind::FORCES);
      chkj_ps.initializeForces();
      ScoreCard tmp_sc(1, 1, 32);
      StaticExclusionMask chkj_se(small_poly_ag.getSystemTopologyPointer(j));
      evalNonbValeMM(&chkj_ps, &tmp_sc, small_poly_ag.getSystemTopologyPointer(j), chkj_se,
                     EvaluateForce::YES, 0);
      const std::vector<double> cpu_frc = chkj_ps.getInterlacedCoordinates(TrajectoryKind::FORCES);
      const double total_e = tmp_sc.reportTotalEnergy(0);
      printf("  %12.6lf\n", total_e);
    }
#endif
    // END CHECK
  }
  hipDeviceSynchronize();
  timer.assignTime(min_timings);
  
  // Summary evaluation
  if (oe.getDisplayTimingsOrder()) {
    timer.assignTime(0);
    timer.printResults();
  }
  printTestSummary(oe.getVerbosity());
  
  return 0;
}
